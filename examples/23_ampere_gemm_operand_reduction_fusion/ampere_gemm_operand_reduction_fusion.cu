/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
 * CUTLASS Example 23: GEMM 与操作数归约融合
 *
 * 本示例演示了一种高级融合技术，将 GEMM 计算与沿 K 维度的操作数归约相结合。
 * 这同时产生标准 GEMM 输出 C = alpha * A * B + beta * C 和一个归约向量（Mx1 或 1xN）。
 *
 * 核心特性：
 * =========
 * - 将归约操作与 GEMM 计算融合，避免额外的内核启动
 * - 在 GEMM 执行期间沿 K 维度归约 A 或 B 操作数
 * - 针对 Ampere 架构的 16x8x16 Tensor Core 操作优化
 * - 支持串行和并行 split-K 策略
 *
 * 性能优势：
 * =========
 * - 消除单独的归约内核启动开销
 * - 通过避免额外的内存读取降低带宽需求
 * - 归约在 GEMM 计算过程中进行，提供更好的数据局部性
 *
 * 应用场景：
 * =========
 * - 在执行矩阵乘法时计算行/列和
 * - 批归一化（Batch Normalization）计算
 * - 神经网络中的统计操作
 * - 注意力机制中的 softmax 归一化
 *
 * 实现细节：
 * =========
 * - 归约主要在 warp 级别进行（gemm/warp/mma_with_reduction_tensor_op.h）
 * - 最终归约在后处理阶段完成（epilogue/threadblock/epilogue_gemm_k_reduction.h）
 * - 使用 FP16/BF16 数据类型，在 Ampere SM80 架构上运行
 * - 通过双缓冲技术隐藏内存延迟
 */

#include <iostream>
#include <fstream>
#include <sstream>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm_with_k_reduction.h"
#include "cutlass/gemm/kernel/default_gemm_with_k_reduction.h"
#include "cutlass/reduction/device/reduce_split_k.h"
#include "cutlass/reduction/kernel/reduce_split_k.h"
#include "cutlass/reduction/thread/reduction_operators.h"
#include "cutlass/matrix_coord.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/device/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/device/convolution.h"

#include "helper.h"

// =====================================================================
// 数据类型配置
// =====================================================================
// 定义输入、输出张量和计算过程中使用的数据类型
using ElementAccumulator = float;                  // 累加器数据类型 - 使用 float 以获得更高精度
using ElementComputeEpilogue = ElementAccumulator; // 后处理计算数据类型 - 与累加器一致
using ElementInputA = cutlass::bfloat16_t;         // A 矩阵元素数据类型 - BF16 提高效率
using ElementInputB = cutlass::bfloat16_t;         // B 矩阵元素数据类型 - BF16 提高效率
using ElementOutput = cutlass::bfloat16_t;         // 输出矩阵元素数据类型 - BF16 与输入匹配

// =====================================================================
// 内存布局配置
// =====================================================================
// 定义矩阵在内存中的存储方式
using LayoutInputA = cutlass::layout::ColumnMajor;  // A 矩阵列主序存储（FORTRAN 风格）
using LayoutInputB = cutlass::layout::RowMajor;     // B 矩阵行主序存储（C 风格）
using LayoutOutput = cutlass::layout::ColumnMajor;  // 输出 C 矩阵列主序存储
// 归约操作输出向量的布局
using LayoutGemmKReduction = cutlass::layout::PitchLinear;  // 归约向量使用线性布局

// =====================================================================
// 硬件架构配置
// =====================================================================
// 选择使用 Tensor Core 还是常规 SIMT 核心
using MMAOp = cutlass::arch::OpClassTensorOp;  // 使用 Tensor Core 进行矩阵乘法

// 指定 CUDA SM 架构版本
using SmArch = cutlass::arch::Sm80;  // 目标 Ampere 架构（计算能力 8.0）

// =====================================================================
// 分层计算的 Tile 配置
// =====================================================================
// 定义线程块（Thread Block）计算的 tile 大小
// 形状：<M, N, K> = <128, 128, 32>
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;  // 每个线程块计算 128x128 的输出 tile

// 定义 Warp 计算的 tile 大小
// 形状：<M, N, K> = <64, 64, 32>
using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;  // 每个 warp 计算 64x64 的输出 tile

// 定义 MMA 操作的大小
// 形状：<M, N, K> = <16, 8, 16> - Ampere 的 mma.sync 指令形状
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;  // 单个 Tensor Core 指令计算 16x8 的输出

// =====================================================================
// 性能优化配置
// =====================================================================
// 定义线程块在 GPU 上的调度方式
// Swizzle 通过改变线程块调度模式来提高 L2 缓存局部性
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>;

// 流水线级数配置
// 更多的级数可以更好地隐藏延迟，但会使用更多共享内存
constexpr int NumStages = 4;  // 4 级流水线，为 Ampere 架构优化

// =====================================================================
// 归约和对齐配置
// =====================================================================
// 选择沿 K 维度归约 A 还是 B 操作数
// true: 归约 A 产生 Mx1 向量（A 的行和）
// false: 归约 B 产生 1xN 向量（B 的列和）
constexpr bool ReduceKForA = true;

// 向量化加载的内存对齐要求
// 8 个元素 = 128 位（BF16 每个元素 16 位）
constexpr int AlignmentA = 8;  // A 矩阵必须对齐到 8 个 BF16 元素
constexpr int AlignmentB = 8;  // B 矩阵必须对齐到 8 个 BF16 元素

// =====================================================================
// 后处理（Epilogue）配置
// =====================================================================
// 定义内核的后处理部分
// LinearCombination 执行：D = alpha * accumulator + beta * C
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        // 输出矩阵数据类型（BF16）
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // 向量宽度 = 128 位 / 16 位 = 8 个元素
                                                          // 决定每次内存事务的元素数
    ElementAccumulator,                                   // 累加器数据类型（float）
    ElementComputeEpilogue>;                             // 后处理计算数据类型（float）

// =====================================================================
// 带 K 维归约的主 GEMM 内核
// =====================================================================
// 这个特殊的 GEMM 内核在执行矩阵乘法的同时
// 沿 K 维度归约一个操作数
using Gemm = typename cutlass::gemm::device::GemmWithKReduction<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  ReduceKForA,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  AlignmentA,
  AlignmentB,
  cutlass::arch::OpMultiplyAdd,
  cutlass::ComplexTransform::kNone,
  cutlass::ComplexTransform::kNone
>;

// =====================================================================
// Split-K 归约配置
// =====================================================================
// 并行 split-k 情况下使用的归约内核
// Shape<4, 64> 表示每个归约 tile 包含 4 行和 64 列
using ReduceGemmSplitKShape = cutlass::MatrixShape<4, 64>;

using ReduceOp = cutlass::reduction::thread::ReduceAdd<
    ElementAccumulator,
    ElementOutput,
    EpilogueOp::kCount 
  >;

using ReduceGemmSplitKKernel = cutlass::reduction::kernel::ReduceSplitK<
    ReduceGemmSplitKShape,
    EpilogueOp,
    ReduceOp
  >;

using ReduceGemmSplitK = cutlass::reduction::device::ReduceSplitK<ReduceGemmSplitKKernel>;

using ReduceVectorSplitKShape = cutlass::MatrixShape<1, 256>;

// 定义内核的后处理部分，使用默认值
using DummyEpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        // Data type of output matrix.
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // The number of elements per vectorized.
                                                          // memory access. This becomes the vector width of
                                                          // math instructions in the epilogue too.
    ElementAccumulator,                                   // Data type of accumulator
    ElementComputeEpilogue,
    cutlass::epilogue::thread::ScaleType::Nothing>;

using ReduceVectorSplitKKernel = cutlass::reduction::kernel::ReduceSplitK<
    ReduceVectorSplitKShape,
    DummyEpilogueOp,
    ReduceOp
  >;

using ReduceVectorSplitK = cutlass::reduction::device::ReduceSplitK<ReduceVectorSplitKKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// =====================================================================
// 命令行接口
// =====================================================================
// 命令行选项解析结构体
struct Options {

  bool help;
  cutlass::gemm::GemmCoord problem_size;
  int split_k_slices;
  bool parallel_split_k;
  bool reference_check;
  bool measure_performance;
  int iterations;
  bool save_workspace;
  ElementComputeEpilogue alpha;
  ElementComputeEpilogue beta;
  bool benchmark;
  std::string tag;

  Options():
    help(false),
    problem_size(1024, 1024, 1024),
    split_k_slices(1),
    parallel_split_k(false),
    reference_check(true),
    measure_performance(false),
    iterations(20),
    save_workspace(false),
    alpha(-1),
    beta(-1),
    benchmark(false) { }

  // 验证问题大小是否与 CUTLASS 实现兼容
  bool valid() {

    // CUTLASS 对 BF16 元素（每个 16 位）使用 128 位向量加载
    // 这要求所有维度必须能被 8 个元素整除（128/16 = 8）
    // 未对齐的访问会导致性能下降或错误
    int const kAlignment = 8;

    if ((problem_size.m() % kAlignment) ||
        (problem_size.n() % kAlignment) ||
        (problem_size.k() % kAlignment)) {

      // 张量未对齐
      return false;
    }

    return true;
  }

  /// 更新输入和过滤器大小
  void update(
    cutlass::gemm::GemmCoord problem_size,
    int split_k_slices,
    bool parallel_split_k) {

    this->problem_size = problem_size;
    this->split_k_slices = split_k_slices;
    this->parallel_split_k = parallel_split_k;
  }

  // 解析命令行参数
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("parallel-split-k")) {
      parallel_split_k = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("split-k-slices", split_k_slices);

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);
  }

  /// 打印使用说明
  std::ostream & print_usage(std::ostream &out) const {

    out << "23_ampere_operand_gemm_reduction_fusion\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --m=<int>            GEMM M\n"
      << "  --n=<int>            GEMM N\n"
      << "  --k=<int>            GEMM K\n"
      << "  --split-k-slices=<int> Split K Slices\n"
      << "  --alpha=<float>      Epilogue scalar alpha\n"
      << "  --beta=<float>       Epilogue scalar beta\n\n"
      << "  --parallel-split-k   If set (true), use parallel split K\n"
      << "  --ref-check          If set (true), reference check on the host is computed\n"
      << "  --perf-check         If set (true), performance is measured.\n"
      << "  --benchmark          If set (true), performance benchmarking on several problem sizes.\n"
      << "  --iterations=<int>   Number of profiling iterations to perform.\n"
      << "  --save-workspace     If set, workspace is written to a text file.\n"
      << "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/23_ampere_gemm_operand_reduction_fusion/23_ampere_gemm_operand_reduction_fusion  --m=1024 --n=1024 --k=1024 \n\n";

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Result {
  double runtime_ms;
  cutlass::Status status;
  cutlass::Status reference_check;
  hipError_t error;

  Result(): 
    runtime_ms(0), 
    status(cutlass::Status::kSuccess),
    reference_check(cutlass::Status::kInvalid),
    error(hipSuccess) { }

  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";
    }

    out << "ID,M,N,K,SplitK-Slices,Parallel-SplitK,Runtime";

    return out;
  }

  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";
    }

    out 
      << "gemm_" << idx << ","
      << options.problem_size.m() << ","
      << options.problem_size.n() << ","
      << options.problem_size.k() << ","
      << options.split_k_slices << ","
      << options.parallel_split_k << ","
      << runtime_ms ;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

/// 运行一个基准测试
Result profile(Options const &options) {

  Result result;

  // 使用 CUTLASS 辅助函数初始化张量
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.problem_size.mk());
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.problem_size.kn());


  // 创建张量 C，维度为 M x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.problem_size.mn());

  // 创建张量 D 用于存储 CUTLASS 内核的输出
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.problem_size.mn());
  // 创建矩阵 D，维度为 M x N，用于存储参考内核的输出
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.problem_size.mn());

  int reduce_vector_length = ReduceKForA ? options.problem_size.m() : options.problem_size.n();

  cutlass::HostTensor<ElementOutput, LayoutGemmKReduction> tensor_reduction({reduce_vector_length, 1});
  cutlass::HostTensor<ElementOutput, LayoutGemmKReduction> tensor_ref_reduction({reduce_vector_length, 1});

  // 使用 CUTLASS 辅助函数在主机上填充输入和输出矩阵
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1997,
      ElementInputA(1),
      ElementInputA(-1),
      0);  // <- 在主机上用均匀分布的随机数据填充张量 A

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      2003,
      ElementInputB(1),
      ElementInputB(-1),
      0);  // <- 在主机上用均匀分布的随机数据填充张量 B

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      2017,
      ElementOutput(1),
      ElementOutput(-1),
      0);  // <- 在主机上用均匀分布的随机数据填充矩阵 C
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- 在主机上用零填充矩阵 D
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- 在主机上用零填充参考矩阵 D

  cutlass::reference::host::TensorFill(
      tensor_reduction.host_view());  // <- 在主机上用零填充归约向量
  cutlass::reference::host::TensorFill(
      tensor_ref_reduction.host_view());  // <- 在主机上用零填充参考归约向量

  // 将数据从主机复制到 GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();
  tensor_reduction.sync_device();

  // 初始化点积计算的 alpha 值
  ElementComputeEpilogue alpha = options.parallel_split_k ? ElementComputeEpilogue(1)
                                                          : ElementComputeEpilogue(options.alpha);
  ElementComputeEpilogue beta = options.parallel_split_k ? ElementComputeEpilogue(0)
                                                         : ElementComputeEpilogue(options.beta);

  cutlass::gemm::GemmUniversalMode mode = options.parallel_split_k ? 
                     cutlass::gemm::GemmUniversalMode::kGemmSplitKParallel :
                     cutlass::gemm::GemmUniversalMode::kGemm;

  int batch_count = options.split_k_slices;

  // 创建 GEMM 内核参数元组
  // 这将作为参数传递给实例化的 CUTLASS 内核
  typename Gemm::Arguments arguments(
    mode,
    options.problem_size,
    batch_count,
    {alpha, beta},
    tensor_a.device_ref().data(),              // <- 设备上张量 A 的引用
    tensor_b.device_ref().data(),              // <- 设备上张量 B 的引用
    tensor_c.device_ref().data(),              // <- 设备上矩阵 C 的引用
    tensor_d.device_ref().data(),              // <- 设备上矩阵 D 的引用
    tensor_reduction.device_ref().data(),      // <- 设备上归约张量的引用
    options.problem_size.m() * options.problem_size.k(),
    options.problem_size.n() * options.problem_size.k(),
    options.problem_size.m() * options.problem_size.n(),
    options.problem_size.m() * options.problem_size.n(),
    reduce_vector_length,
    tensor_a.layout().stride(0),
    tensor_b.layout().stride(0),
    tensor_c.layout().stride(0),
    tensor_d.layout().stride(0),
    tensor_reduction.layout().stride(0));

  // 根据模板实例化 CUTLASS 内核
  Gemm gemm_op;

  // 使用参数查询矩阵乘法计算所需的额外工作空间
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // 分配工作空间内存
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // 检查问题大小是否受支持
  result.status = gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  // 使用参数和工作空间指针初始化 CUTLASS 内核
  result.status = gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  // 启动已初始化的 CUTLASS 内核
  result.status = gemm_op();

  CUTLASS_CHECK(result.status);

  if (options.parallel_split_k && batch_count > 1) {
    // 归约 GEMM 结果

    ElementComputeEpilogue alpha = ElementComputeEpilogue(options.alpha);
    ElementComputeEpilogue beta = ElementComputeEpilogue(options.beta);

    int splitk_gemm_stride = options.problem_size.m();

    cutlass::layout::RowMajor splitk_gemm_layout(splitk_gemm_stride);

    void * workspace_gemm_ptr = workspace.get();
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> workspace_gemm_tensorref(static_cast<ElementOutput *>(workspace_gemm_ptr), splitk_gemm_layout);

    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_d_tensorref(tensor_d.device_ref().data(), splitk_gemm_layout);

    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_c_tensorref(tensor_c.device_ref().data(), splitk_gemm_layout);

    typename ReduceGemmSplitK::Arguments reduce_gemm_splitk_arguments{
      cutlass::MatrixCoord(options.problem_size.n(), options.problem_size.m()),
      batch_count,
      size_t(options.problem_size.m() * options.problem_size.n()),
      workspace_gemm_tensorref,
      tensor_d_tensorref,
      tensor_c_tensorref,
      {alpha, beta} 
    };

    ReduceGemmSplitK reduce_gemm_splitk_op;
   
    result.status = reduce_gemm_splitk_op.initialize(reduce_gemm_splitk_arguments); 
    CUTLASS_CHECK(result.status);

    result.status = reduce_gemm_splitk_op();
    CUTLASS_CHECK(result.status);

    // 归约 K 维向量
    cutlass::layout::RowMajor splitk_vector_layout(reduce_vector_length);
   
    ElementOutput *workspace_vector_ptr = static_cast<ElementOutput *>(workspace_gemm_ptr) + batch_count * options.problem_size.m() * options.problem_size.n();
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> workspace_vector_tensorref(workspace_vector_ptr, splitk_vector_layout);

    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_reduction_tensorref(tensor_reduction.device_ref().data(), splitk_vector_layout);

    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_nullptr_tensorref(nullptr, splitk_vector_layout);

    typename ReduceVectorSplitK::Arguments reduce_vector_splitk_arguments(
      cutlass::MatrixCoord(1, reduce_vector_length),
      batch_count,
      size_t(reduce_vector_length),
      workspace_vector_tensorref,
      tensor_reduction_tensorref,
      tensor_nullptr_tensorref,
      {1.0f, 0.0f});

    ReduceVectorSplitK reduce_vector_splitk_op;
   
    result.status = reduce_vector_splitk_op.initialize(reduce_vector_splitk_arguments); 
    CUTLASS_CHECK(result.status);

    result.status = reduce_vector_splitk_op();
    CUTLASS_CHECK(result.status);
  }

  //
  // 创建设备参考卷积内核的实例
  //
  if (options.reference_check) {
    // 启动设备参考内核来严格计算乘积 A * B
    cutlass::reference::device::Gemm<
        ElementInputA, 
        LayoutInputA, 
        ElementInputB, 
        LayoutInputB, 
        ElementOutput,
        LayoutOutput, 
        ElementComputeEpilogue, 
        ElementAccumulator> gemm_device;
  
    gemm_device
      (
        options.problem_size,
        ElementComputeEpilogue(options.alpha),
        tensor_a.device_ref(),
        tensor_b.device_ref(),
        ElementComputeEpilogue(options.beta),
        tensor_c.device_ref(), 
        tensor_ref_d.device_ref()
      );
  
    // 等待内核完成
    hipDeviceSynchronize();
  
    // 将 CUTLASS 和参考内核的输出数据复制到主机进行比较
    tensor_d.sync_host();
    tensor_ref_d.sync_host();
  
    tensor_reduction.sync_host();
  
    // 在主机代码中执行 K 维归约
    if (ReduceKForA) {
      for (int m = 0; m < options.problem_size.m(); ++m) {
        for (int k = 0; k < options.problem_size.k(); ++k) {
          tensor_ref_reduction.at({m, 0}) += 
            tensor_a.at(cutlass::MatrixCoord(m, k));
        }
      }
    } else {
      for (int k = 0; k < options.problem_size.k(); ++k) {
        for (int n = 0; n < options.problem_size.n(); ++n) {
          tensor_ref_reduction.at({n, 0}) += 
            tensor_b.at(cutlass::MatrixCoord(k, n));
        }
      }
    }
  
    // 检查 CUTLASS 内核和参考内核的输出是否相等
    bool pass = cutlass::reference::host::TensorEquals(tensor_d.host_view(),
                                                       tensor_ref_d.host_view());

    pass &= cutlass::reference::host::TensorEquals(tensor_ref_reduction.host_view(),
                                                   tensor_reduction.host_view());

    if (!pass) {
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    } else {
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  } else {
    result.reference_check = cutlass::Status::kInvalid;
  }

  if (options.save_workspace) {

    std::stringstream ss;

    ss << "23_ampere_gemm_operand_reduction_fusion"
      << options.problem_size.m() << "x" << options.problem_size.n() << "x" << options.problem_size.k()
      << ".dat";

    std::ofstream output_workspace(ss.str());

    output_workspace 
      << "A = \n" << tensor_a.host_view() << "\n\n"
      << "B = \n" << tensor_b.host_view() << "\n\n";

    if (options.reference_check) {
      output_workspace << "Reference D = \n" << tensor_ref_d.host_view() << "\n\n";
      output_workspace << "Reference reduction vector = \n" << tensor_ref_reduction.host_view() << "\n\n";
    }

    output_workspace << "Computed D = \n" << tensor_d.host_view() << std::endl;
    output_workspace << "Computed reduction vector = \n" << tensor_reduction.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }
 
  //
  // 性能测量
  //

  if (options.measure_performance) {

    hipEvent_t events[2];
    
    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // 在一系列卷积操作开始时记录事件
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 在设备上启动一系列隐式 GEMM 操作
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = gemm_op();
      CUTLASS_CHECK(result.status);
    }

    // 当卷积已启动时记录事件
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 等待设备上的工作完成
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 测量运行时间
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 打印平均运行时间和 GFLOPs
    result.runtime_ms = double(runtime_ms) / double(options.iterations);

    // 清理资源
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

int main(int argc, char const **args) {

  bool notSupported = false;

  // Ampere Tensor Core 操作通过 mma.sync 暴露，首次在 CUDA 11.0 中可用
  //
  // CUTLASS 必须使用 CUDA 11 工具包编译才能运行 Conv2dFprop 示例
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));

  if (!(props.major >= 8)) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    return 0;
  }

  Options options;
  
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // 基准测试多个层

    struct Benchmark {
      int m, n, k, split_k_slices, parallel_split_k;
    } problem_sizes[] = {
      {4096, 6144, 4096, 1, false},
    };

    Result::print_header(std::cout, options) << "\n";
 
    int idx = 1;

    for (auto const &problem_size : problem_sizes) {
      options.update({problem_size.m, problem_size.n, problem_size.k},
                     problem_size.split_k_slices, problem_size.parallel_split_k);

      Result result = profile(options);
      result.print(std::cout, idx, options) << "\n";

      ++idx;
    }
  } else { 

    // 执行一个问题大小
    if (!options.valid()) {
      std::cerr << "Invalid problem." << "\n";
      return -1;
    }

    Result result = profile(options);

    Result::print_header(std::cout, options) << "\n";
    result.print(std::cout, 1, options) << "\n";
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
