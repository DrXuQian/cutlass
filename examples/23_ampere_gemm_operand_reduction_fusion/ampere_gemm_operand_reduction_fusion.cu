/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
 * CUTLASS Example 23: GEMM 与操作数归约融合
 *
 * 本示例演示了一种高级融合技术，将 GEMM 计算与沿 K 维度的操作数归约相结合。
 * 这同时产生标准 GEMM 输出 C = alpha * A * B + beta * C 和一个归约向量（Mx1 或 1xN）。
 *
 * 核心特性：
 * =========
 * - 将归约操作与 GEMM 计算融合，避免额外的内核启动
 * - 在 GEMM 执行期间沿 K 维度归约 A 或 B 操作数
 * - 针对 Ampere 架构的 16x8x16 Tensor Core 操作优化
 * - 支持串行和并行 split-K 策略
 *
 * 性能优势：
 * =========
 * - 消除单独的归约内核启动开销
 * - 通过避免额外的内存读取降低带宽需求
 * - 归约在 GEMM 计算过程中进行，提供更好的数据局部性
 *
 * 应用场景：
 * =========
 * - 在执行矩阵乘法时计算行/列和
 * - 批归一化（Batch Normalization）计算
 * - 神经网络中的统计操作
 * - 注意力机制中的 softmax 归一化
 *
 * 实现细节：
 * =========
 * - 归约主要在 warp 级别进行（gemm/warp/mma_with_reduction_tensor_op.h）
 * - 最终归约在后处理阶段完成（epilogue/threadblock/epilogue_gemm_k_reduction.h）
 * - 使用 FP16/BF16 数据类型，在 Ampere SM80 架构上运行
 * - 通过双缓冲技术隐藏内存延迟
 */

// 标准 C++ 库
#include <iostream>
#include <fstream>
#include <sstream>

// CUTLASS 核心库
#include "cutlass/cutlass.h"
// 带 K 维归约的 GEMM 设备接口
#include "cutlass/gemm/device/gemm_with_k_reduction.h"
// 带 K 维归约的默认 GEMM 内核
#include "cutlass/gemm/kernel/default_gemm_with_k_reduction.h"
// Split-K 归约设备接口
#include "cutlass/reduction/device/reduce_split_k.h"
// Split-K 归约内核
#include "cutlass/reduction/kernel/reduce_split_k.h"
// 线程级归约操作符
#include "cutlass/reduction/thread/reduction_operators.h"
// 矩阵坐标辅助类
#include "cutlass/matrix_coord.h"

// CUTLASS 实用工具
#include "cutlass/util/command_line.h"        // 命令行参数解析
#include "cutlass/util/host_tensor.h"         // 主机端张量容器
#include "cutlass/util/tensor_view_io.h"      // 张量 I/O 操作
#include "cutlass/util/reference/device/gemm.h"  // 设备端参考 GEMM 实现
#include "cutlass/util/reference/host/tensor_compare.h"  // 张量比较工具
#include "cutlass/util/reference/host/tensor_copy.h"     // 张量复制工具
#include "cutlass/util/reference/host/tensor_fill.h"     // 张量填充工具
#include "cutlass/util/reference/device/convolution.h"   // 设备端卷积参考实现

// 辅助函数头文件
#include "helper.h"

// =====================================================================
// 数据类型配置
// =====================================================================
// 定义输入、输出张量和计算过程中使用的数据类型
using ElementAccumulator = float;                  // 累加器数据类型 - 使用 float 以获得更高精度
using ElementComputeEpilogue = ElementAccumulator; // 后处理计算数据类型 - 与累加器一致
using ElementInputA = cutlass::bfloat16_t;         // A 矩阵元素数据类型 - BF16 提高效率
using ElementInputB = cutlass::bfloat16_t;         // B 矩阵元素数据类型 - BF16 提高效率
using ElementOutput = cutlass::bfloat16_t;         // 输出矩阵元素数据类型 - BF16 与输入匹配

// =====================================================================
// 内存布局配置
// =====================================================================
// 定义矩阵在内存中的存储方式
using LayoutInputA = cutlass::layout::ColumnMajor;  // A 矩阵列主序存储（FORTRAN 风格）
using LayoutInputB = cutlass::layout::RowMajor;     // B 矩阵行主序存储（C 风格）
using LayoutOutput = cutlass::layout::ColumnMajor;  // 输出 C 矩阵列主序存储
// 归约操作输出向量的布局
using LayoutGemmKReduction = cutlass::layout::PitchLinear;  // 归约向量使用线性布局

// =====================================================================
// 硬件架构配置
// =====================================================================
// 选择使用 Tensor Core 还是常规 SIMT 核心
using MMAOp = cutlass::arch::OpClassTensorOp;  // 使用 Tensor Core 进行矩阵乘法

// 指定 CUDA SM 架构版本
using SmArch = cutlass::arch::Sm80;  // 目标 Ampere 架构（计算能力 8.0）

// =====================================================================
// 分层计算的 Tile 配置
// =====================================================================
// 定义线程块（Thread Block）计算的 tile 大小
// 形状：<M, N, K> = <128, 128, 32>
using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;  // 每个线程块计算 128x128 的输出 tile

// 定义 Warp 计算的 tile 大小
// 形状：<M, N, K> = <64, 64, 32>
using WarpShape = cutlass::gemm::GemmShape<64, 64, 32>;  // 每个 warp 计算 64x64 的输出 tile

// 定义 MMA 操作的大小
// 形状：<M, N, K> = <16, 8, 16> - Ampere 的 mma.sync 指令形状
using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;  // 单个 Tensor Core 指令计算 16x8 的输出

// =====================================================================
// 性能优化配置
// =====================================================================
// 定义线程块在 GPU 上的调度方式
// Swizzle 通过改变线程块调度模式来提高 L2 缓存局部性
using SwizzleThreadBlock = cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>;

// 流水线级数配置
// 更多的级数可以更好地隐藏延迟，但会使用更多共享内存
constexpr int NumStages = 4;  // 4 级流水线，为 Ampere 架构优化

// =====================================================================
// 归约和对齐配置
// =====================================================================
// 选择沿 K 维度归约 A 还是 B 操作数
// true: 归约 A 产生 Mx1 向量（A 的行和）
// false: 归约 B 产生 1xN 向量（B 的列和）
constexpr bool ReduceKForA = true;

// 向量化加载的内存对齐要求
// 8 个元素 = 128 位（BF16 每个元素 16 位）
constexpr int AlignmentA = 8;  // A 矩阵必须对齐到 8 个 BF16 元素
constexpr int AlignmentB = 8;  // B 矩阵必须对齐到 8 个 BF16 元素

// =====================================================================
// 后处理（Epilogue）配置
// =====================================================================
// 定义内核的后处理部分
// LinearCombination 执行：D = alpha * accumulator + beta * C
using EpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        // 输出矩阵数据类型（BF16）
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // 向量宽度 = 128 位 / 16 位 = 8 个元素
                                                          // 决定每次内存事务的元素数
    ElementAccumulator,                                   // 累加器数据类型（float）
    ElementComputeEpilogue>;                             // 后处理计算数据类型（float）

// =====================================================================
// 带 K 维归约的主 GEMM 内核
// =====================================================================
// 这个特殊的 GEMM 内核在执行矩阵乘法的同时
// 沿 K 维度归约一个操作数
using Gemm = typename cutlass::gemm::device::GemmWithKReduction<
  ElementInputA, LayoutInputA,
  ElementInputB, LayoutInputB,
  ElementOutput, LayoutOutput,
  ElementAccumulator,
  MMAOp,
  ReduceKForA,
  SmArch,
  ThreadblockShape,
  WarpShape,
  InstructionShape,
  EpilogueOp,
  SwizzleThreadBlock,
  NumStages,
  AlignmentA,
  AlignmentB,
  cutlass::arch::OpMultiplyAdd,
  cutlass::ComplexTransform::kNone,
  cutlass::ComplexTransform::kNone
>;

// =====================================================================
// Split-K 归约配置
// =====================================================================
// 并行 split-k 情况下使用的归约内核
// Shape<4, 64> 表示每个归约 tile 包含 4 行和 64 列
using ReduceGemmSplitKShape = cutlass::MatrixShape<4, 64>;

// 定义归约操作为加法
// 将多个累加器值相加得到最终输出
using ReduceOp = cutlass::reduction::thread::ReduceAdd<
    ElementAccumulator,  // 输入数据类型（累加器）
    ElementOutput,       // 输出数据类型
    EpilogueOp::kCount   // 向量元素个数
  >;

// Split-K GEMM 归约内核
// 用于合并多个 K 分片的部分结果
using ReduceGemmSplitKKernel = cutlass::reduction::kernel::ReduceSplitK<
    ReduceGemmSplitKShape,  // 归约 tile 形状
    EpilogueOp,             // 后处理操作
    ReduceOp                // 归约操作（加法）
  >;

// Split-K GEMM 归约设备接口
using ReduceGemmSplitK = cutlass::reduction::device::ReduceSplitK<ReduceGemmSplitKKernel>;

// Split-K 向量归约的 tile 形状
// Shape<1, 256> 表示归约 256 个元素的向量
using ReduceVectorSplitKShape = cutlass::MatrixShape<1, 256>;

// 定义向量归约的虚拟后处理操作
// 用于 Split-K 向量归约，不执行额外的缩放
using DummyEpilogueOp = cutlass::epilogue::thread::LinearCombination<
    ElementOutput,                                        // 输出矩阵数据类型
    128 / cutlass::sizeof_bits<ElementOutput>::value,     // 向量化访问的元素数量
                                                          // 128位/16位 = 8个元素
                                                          // 这也决定了后处理中数学指令的向量宽度
    ElementAccumulator,                                   // 累加器数据类型
    ElementComputeEpilogue,                              // 后处理计算数据类型
    cutlass::epilogue::thread::ScaleType::Nothing>;      // 不进行缩放

// Split-K 向量归约内核
// 用于合并多个 K 分片的向量结果
using ReduceVectorSplitKKernel = cutlass::reduction::kernel::ReduceSplitK<
    ReduceVectorSplitKShape,  // 向量归约 tile 形状
    DummyEpilogueOp,          // 虚拟后处理操作
    ReduceOp                  // 归约操作（加法）
  >;

// Split-K 向量归约设备接口
using ReduceVectorSplitK = cutlass::reduction::device::ReduceSplitK<ReduceVectorSplitKKernel>;

/////////////////////////////////////////////////////////////////////////////////////////////////

// =====================================================================
// 命令行接口
// =====================================================================
// 命令行选项解析结构体
struct Options {

  bool help;                                  // 显示帮助信息
  cutlass::gemm::GemmCoord problem_size;      // GEMM 问题大小 (M, N, K)
  int split_k_slices;                         // K 维度分片数量
  bool parallel_split_k;                      // 是否使用并行 split-K
  bool reference_check;                       // 是否进行正确性检查
  bool measure_performance;                   // 是否测量性能
  int iterations;                             // 性能测试迭代次数
  bool save_workspace;                        // 是否保存工作空间到文件
  ElementComputeEpilogue alpha;               // GEMM 缩放因子 alpha
  ElementComputeEpilogue beta;                // GEMM 缩放因子 beta
  bool benchmark;                             // 是否运行基准测试
  std::string tag;                            // 结果标签

  // 构造函数：设置默认参数值
  Options():
    help(false),
    problem_size(1024, 1024, 1024),  // 默认 1024x1024x1024 矩阵
    split_k_slices(1),                // 默认不分片
    parallel_split_k(false),          // 默认串行
    reference_check(true),            // 默认进行正确性检查
    measure_performance(false),       // 默认不测性能
    iterations(20),                   // 默认 20 次迭代
    save_workspace(false),            // 默认不保存工作空间
    alpha(-1),                        // 默认 alpha = -1
    beta(-1),                         // 默认 beta = -1
    benchmark(false) { }              // 默认不运行基准测试

  // 验证问题大小是否与 CUTLASS 实现兼容
  bool valid() {

    // CUTLASS 对 BF16 元素（每个 16 位）使用 128 位向量加载
    // 这要求所有维度必须能被 8 个元素整除（128/16 = 8）
    // 未对齐的访问会导致性能下降或错误
    int const kAlignment = 8;

    if ((problem_size.m() % kAlignment) ||
        (problem_size.n() % kAlignment) ||
        (problem_size.k() % kAlignment)) {

      // 张量未对齐
      return false;
    }

    return true;
  }

  /// 更新输入和过滤器大小
  void update(
    cutlass::gemm::GemmCoord problem_size,
    int split_k_slices,
    bool parallel_split_k) {

    this->problem_size = problem_size;
    this->split_k_slices = split_k_slices;
    this->parallel_split_k = parallel_split_k;
  }

  // 解析命令行参数
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    if (cmd.check_cmd_line_flag("parallel-split-k")) {
      parallel_split_k = true;
    }

    if (cmd.check_cmd_line_flag("ref-check")) {
      reference_check = true;
    }

    if (cmd.check_cmd_line_flag("perf-check")) {
      measure_performance = true;
    }

    if (cmd.check_cmd_line_flag("save-workspace")) {
      save_workspace = true;
    }

    if (cmd.check_cmd_line_flag("benchmark")) {
      benchmark = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());
    cmd.get_cmd_line_argument("split-k-slices", split_k_slices);

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);
    
    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("tag", tag);
  }

  /// 打印使用说明
  std::ostream & print_usage(std::ostream &out) const {

    out << "23_ampere_operand_gemm_reduction_fusion\n\n"
      << "Options:\n\n"
      << "  --help               If specified, displays this usage statement.\n\n"
      << "  --m=<int>            GEMM M\n"
      << "  --n=<int>            GEMM N\n"
      << "  --k=<int>            GEMM K\n"
      << "  --split-k-slices=<int> Split K Slices\n"
      << "  --alpha=<float>      Epilogue scalar alpha\n"
      << "  --beta=<float>       Epilogue scalar beta\n\n"
      << "  --parallel-split-k   If set (true), use parallel split K\n"
      << "  --ref-check          If set (true), reference check on the host is computed\n"
      << "  --perf-check         If set (true), performance is measured.\n"
      << "  --benchmark          If set (true), performance benchmarking on several problem sizes.\n"
      << "  --iterations=<int>   Number of profiling iterations to perform.\n"
      << "  --save-workspace     If set, workspace is written to a text file.\n"
      << "  --tag=<string>       String to replicate across the first column in the results table\n";

    out << "\n\nExamples:\n\n"
      << "$ ./examples/23_ampere_gemm_operand_reduction_fusion/23_ampere_gemm_operand_reduction_fusion  --m=1024 --n=1024 --k=1024 \n\n";

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// 结果结构体：保存运行结果和状态
struct Result {
  double runtime_ms;                   // 运行时间（毫秒）
  cutlass::Status status;               // CUTLASS 执行状态
  cutlass::Status reference_check;      // 正确性验证状态
  hipError_t error;                   // CUDA 错误代码

  // 构造函数：初始化所有成员
  Result():
    runtime_ms(0),                                     // 初始运行时间为 0
    status(cutlass::Status::kSuccess),                 // 初始状态为成功
    reference_check(cutlass::Status::kInvalid),        // 初始验证状态为无效
    error(hipSuccess) { }                             // 初始 CUDA 错误为成功

  // 打印结果表头
  static std::ostream & print_header(std::ostream &out, Options const &options) {

    if (!options.tag.empty()) {
      out << "Name,";  // 如果有标签，添加名称列
    }

    // 打印各列标题
    out << "ID,M,N,K,SplitK-Slices,Parallel-SplitK,Runtime";

    return out;
  }

  // 打印结果数据行
  std::ostream & print(std::ostream &out, int idx, Options const &options) {

    if (!options.tag.empty()) {
      out << options.tag << ",";  // 输出标签
    }

    // 输出测试编号和参数
    out
      << "gemm_" << idx << ","
      << options.problem_size.m() << ","
      << options.problem_size.n() << ","
      << options.problem_size.k() << ","
      << options.split_k_slices << ","
      << options.parallel_split_k << ","
      << runtime_ms ;

    return out;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

/// 运行一个基准测试
Result profile(Options const &options) {

  Result result;

  // 使用 CUTLASS 辅助函数初始化张量
  cutlass::HostTensor<ElementInputA, LayoutInputA> tensor_a(options.problem_size.mk());
  cutlass::HostTensor<ElementInputB, LayoutInputB> tensor_b(options.problem_size.kn());


  // 创建张量 C，维度为 M x N
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_c(options.problem_size.mn());

  // 创建张量 D 用于存储 CUTLASS 内核的输出
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_d(options.problem_size.mn());
  // 创建矩阵 D，维度为 M x N，用于存储参考内核的输出
  cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_ref_d(options.problem_size.mn());

  // 根据 ReduceKForA 配置确定归约向量的长度
  // true: 归约 A 矩阵，得到 M 长度的向量（每行的和）
  // false: 归约 B 矩阵，得到 N 长度的向量（每列的和）
  int reduce_vector_length = ReduceKForA ? options.problem_size.m() : options.problem_size.n();

  // 创建归约结果向量（大小为 reduce_vector_length x 1）
  cutlass::HostTensor<ElementOutput, LayoutGemmKReduction> tensor_reduction({reduce_vector_length, 1});
  // 创建参考归约结果向量，用于验证
  cutlass::HostTensor<ElementOutput, LayoutGemmKReduction> tensor_ref_reduction({reduce_vector_length, 1});

  // 使用 CUTLASS 辅助函数在主机上填充输入和输出矩阵
  cutlass::reference::host::TensorFillRandomUniform(
      tensor_a.host_view(),
      1997,
      ElementInputA(1),
      ElementInputA(-1),
      0);  // <- 在主机上用均匀分布的随机数据填充张量 A

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_b.host_view(),
      2003,
      ElementInputB(1),
      ElementInputB(-1),
      0);  // <- 在主机上用均匀分布的随机数据填充张量 B

  cutlass::reference::host::TensorFillRandomUniform(
      tensor_c.host_view(),
      2017,
      ElementOutput(1),
      ElementOutput(-1),
      0);  // <- 在主机上用均匀分布的随机数据填充矩阵 C
  cutlass::reference::host::TensorFill(
      tensor_d.host_view());  // <- 在主机上用零填充矩阵 D
  cutlass::reference::host::TensorFill(
      tensor_ref_d.host_view());  // <- 在主机上用零填充参考矩阵 D

  cutlass::reference::host::TensorFill(
      tensor_reduction.host_view());  // <- 在主机上用零填充归约向量
  cutlass::reference::host::TensorFill(
      tensor_ref_reduction.host_view());  // <- 在主机上用零填充参考归约向量

  // 将数据从主机复制到 GPU
  tensor_a.sync_device();
  tensor_b.sync_device();
  tensor_c.sync_device();
  tensor_d.sync_device();
  tensor_ref_d.sync_device();
  tensor_reduction.sync_device();

  // 根据 split-K 模式设置 alpha 和 beta 值
  // 并行 split-K: alpha=1, beta=0（第一次累加不需要额外缩放）
  // 串行模式: 使用用户指定的 alpha 和 beta
  ElementComputeEpilogue alpha = options.parallel_split_k ? ElementComputeEpilogue(1)
                                                          : ElementComputeEpilogue(options.alpha);
  ElementComputeEpilogue beta = options.parallel_split_k ? ElementComputeEpilogue(0)
                                                         : ElementComputeEpilogue(options.beta);

  // 设置 GEMM 运行模式
  // kGemmSplitKParallel: 并行 split-K，多个线程块处理同一个输出 tile
  // kGemm: 标准 GEMM 模式
  cutlass::gemm::GemmUniversalMode mode = options.parallel_split_k ?
                     cutlass::gemm::GemmUniversalMode::kGemmSplitKParallel :
                     cutlass::gemm::GemmUniversalMode::kGemm;

  // 设置批处理数量（即 K 维度的分片数）
  int batch_count = options.split_k_slices;

  // 创建 GEMM 内核参数元组
  // 这将作为参数传递给实例化的 CUTLASS 内核
  typename Gemm::Arguments arguments(
    mode,                                            // GEMM 运行模式
    options.problem_size,                            // 问题大小 (M, N, K)
    batch_count,                                     // K 维分片数
    {alpha, beta},                                   // 缩放因子
    tensor_a.device_ref().data(),              // <- 设备上张量 A 的引用
    tensor_b.device_ref().data(),              // <- 设备上张量 B 的引用
    tensor_c.device_ref().data(),              // <- 设备上矩阵 C 的引用
    tensor_d.device_ref().data(),              // <- 设备上矩阵 D 的引用
    tensor_reduction.device_ref().data(),      // <- 设备上归约张量的引用
    options.problem_size.m() * options.problem_size.k(),  // A 矩阵元素总数
    options.problem_size.n() * options.problem_size.k(),  // B 矩阵元素总数
    options.problem_size.m() * options.problem_size.n(),  // C 矩阵元素总数
    options.problem_size.m() * options.problem_size.n(),  // D 矩阵元素总数
    reduce_vector_length,                                 // 归约向量长度
    tensor_a.layout().stride(0),                         // A 矩阵 leading dimension
    tensor_b.layout().stride(0),                         // B 矩阵 leading dimension
    tensor_c.layout().stride(0),                         // C 矩阵 leading dimension
    tensor_d.layout().stride(0),                         // D 矩阵 leading dimension
    tensor_reduction.layout().stride(0));                // 归约向量 stride

  // 根据模板实例化 CUTLASS 内核
  Gemm gemm_op;

  // 使用参数查询矩阵乘法计算所需的额外工作空间
  size_t workspace_size = Gemm::get_workspace_size(arguments);

  // 分配工作空间内存
  cutlass::device_memory::allocation<uint8_t> workspace(workspace_size);

  // 检查问题大小是否受支持
  result.status = gemm_op.can_implement(arguments);
  CUTLASS_CHECK(result.status);

  // 使用参数和工作空间指针初始化 CUTLASS 内核
  result.status = gemm_op.initialize(arguments, workspace.get());
  CUTLASS_CHECK(result.status);

  // 启动已初始化的 CUTLASS 内核
  result.status = gemm_op();

  CUTLASS_CHECK(result.status);

  // 如果启用了并行 split-K 且分片数大于 1，需要执行归约
  if (options.parallel_split_k && batch_count > 1) {
    // 归约 GEMM 结果（合并多个 K 分片的部分结果）

    // 使用用户指定的 alpha 和 beta 进行最终归约
    ElementComputeEpilogue alpha = ElementComputeEpilogue(options.alpha);
    ElementComputeEpilogue beta = ElementComputeEpilogue(options.beta);

    // 设置 split-K GEMM 的 stride（列主序存储）
    int splitk_gemm_stride = options.problem_size.m();

    // 创建行主序布局，用于 split-K 归约
    cutlass::layout::RowMajor splitk_gemm_layout(splitk_gemm_stride);

    // 获取工作空间指针，用于存储中间结果
    void * workspace_gemm_ptr = workspace.get();
    // 创建工作空间的张量引用
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> workspace_gemm_tensorref(static_cast<ElementOutput *>(workspace_gemm_ptr), splitk_gemm_layout);

    // 创建输出矩阵 D 的张量引用
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_d_tensorref(tensor_d.device_ref().data(), splitk_gemm_layout);

    // 创建输入矩阵 C 的张量引用
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_c_tensorref(tensor_c.device_ref().data(), splitk_gemm_layout);

    // 创建 split-K GEMM 归约参数
    typename ReduceGemmSplitK::Arguments reduce_gemm_splitk_arguments{
      cutlass::MatrixCoord(options.problem_size.n(), options.problem_size.m()),  // 输出矩阵大小（转置）
      batch_count,                                                              // K 分片数
      size_t(options.problem_size.m() * options.problem_size.n()),              // 单个矩阵元素数
      workspace_gemm_tensorref,                                                 // 中间结果输入
      tensor_d_tensorref,                                                      // 最终输出
      tensor_c_tensorref,                                                      // C 矩阵输入
      {alpha, beta}                                                            // 缩放因子
    };

    // 创建并执行 split-K GEMM 归约操作
    ReduceGemmSplitK reduce_gemm_splitk_op;

    // 初始化归约操作
    result.status = reduce_gemm_splitk_op.initialize(reduce_gemm_splitk_arguments);
    CUTLASS_CHECK(result.status);

    // 执行归约
    result.status = reduce_gemm_splitk_op();
    CUTLASS_CHECK(result.status);

    // 归约 K 维向量（合并多个分片的向量结果）
    cutlass::layout::RowMajor splitk_vector_layout(reduce_vector_length);
   
    // 计算向量工作空间的起始位置（在 GEMM 工作空间之后）
    ElementOutput *workspace_vector_ptr = static_cast<ElementOutput *>(workspace_gemm_ptr) + batch_count * options.problem_size.m() * options.problem_size.n();
    // 创建向量工作空间的张量引用
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> workspace_vector_tensorref(workspace_vector_ptr, splitk_vector_layout);

    // 创建归约输出向量的张量引用
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_reduction_tensorref(tensor_reduction.device_ref().data(), splitk_vector_layout);

    // 创建空引用（不需要 C 向量输入）
    cutlass::TensorRef<ElementOutput, cutlass::layout::RowMajor> tensor_nullptr_tensorref(nullptr, splitk_vector_layout);

    // 创建 split-K 向量归约参数
    typename ReduceVectorSplitK::Arguments reduce_vector_splitk_arguments(
      cutlass::MatrixCoord(1, reduce_vector_length),   // 向量大小 (1 x length)
      batch_count,                                     // K 分片数
      size_t(reduce_vector_length),                    // 单个向量元素数
      workspace_vector_tensorref,                      // 中间结果输入
      tensor_reduction_tensorref,                      // 最终输出
      tensor_nullptr_tensorref,                        // 无需 C 向量
      {1.0f, 0.0f});                                   // alpha=1, beta=0（直接累加）

    // 创建并执行 split-K 向量归约操作
    ReduceVectorSplitK reduce_vector_splitk_op;

    // 初始化归约操作
    result.status = reduce_vector_splitk_op.initialize(reduce_vector_splitk_arguments);
    CUTLASS_CHECK(result.status);

    // 执行归约
    result.status = reduce_vector_splitk_op();
    CUTLASS_CHECK(result.status);
  }

  //
  // 创建设备参考内核的实例，用于验证结果正确性
  //
  if (options.reference_check) {
    // 启动设备参考内核来严格计算乘积 A * B
    // 这是一个标准的 GEMM 实现，用于对比验证
    cutlass::reference::device::Gemm<
        ElementInputA,                // A 矩阵元素类型
        LayoutInputA,                 // A 矩阵布局
        ElementInputB,                // B 矩阵元素类型
        LayoutInputB,                 // B 矩阵布局
        ElementOutput,                // 输出元素类型
        LayoutOutput,                 // 输出布局
        ElementComputeEpilogue,       // 后处理计算类型
        ElementAccumulator> gemm_device;  // 累加器类型
  
    // 执行参考 GEMM: D = alpha * A * B + beta * C
    gemm_device
      (
        options.problem_size,                    // 问题大小 (M, N, K)
        ElementComputeEpilogue(options.alpha),   // 缩放因子 alpha
        tensor_a.device_ref(),                   // A 矩阵
        tensor_b.device_ref(),                   // B 矩阵
        ElementComputeEpilogue(options.beta),    // 缩放因子 beta
        tensor_c.device_ref(),                   // C 矩阵
        tensor_ref_d.device_ref()                // 输出 D 矩阵
      );
  
    // 等待内核完成
    hipDeviceSynchronize();
  
    // 将 CUTLASS 和参考内核的输出数据复制到主机进行比较
    tensor_d.sync_host();
    tensor_ref_d.sync_host();
  
    tensor_reduction.sync_host();
  
    // 在主机代码中执行 K 维归约，作为参考结果
    if (ReduceKForA) {
      // 归约 A 矩阵：计算每一行的和
      for (int m = 0; m < options.problem_size.m(); ++m) {
        for (int k = 0; k < options.problem_size.k(); ++k) {
          tensor_ref_reduction.at({m, 0}) +=     // 累加到第 m 个元素
            tensor_a.at(cutlass::MatrixCoord(m, k));  // A[m][k]
        }
      }
    } else {
      // 归约 B 矩阵：计算每一列的和
      for (int k = 0; k < options.problem_size.k(); ++k) {
        for (int n = 0; n < options.problem_size.n(); ++n) {
          tensor_ref_reduction.at({n, 0}) +=     // 累加到第 n 个元素
            tensor_b.at(cutlass::MatrixCoord(k, n));  // B[k][n]
        }
      }
    }
  
    // 检查 CUTLASS 内核和参考内核的输出是否相等
    // 首先比较 GEMM 结果
    bool pass = cutlass::reference::host::TensorEquals(tensor_d.host_view(),
                                                       tensor_ref_d.host_view());

    // 然后比较归约向量结果
    pass &= cutlass::reference::host::TensorEquals(tensor_ref_reduction.host_view(),
                                                   tensor_reduction.host_view());

    if (!pass) {
      // 结果不匹配，设置错误状态
      result.reference_check = cutlass::Status::kErrorInternal;
      std::cout << "ERROR - results miscompared.\n";
    } else {
      // 结果匹配，验证通过
      result.reference_check = cutlass::Status::kSuccess;
      std::cout << "Passed.\n";
    }
  } else {
    // 未进行参考检查
    result.reference_check = cutlass::Status::kInvalid;
  }

  // 如果需要保存工作空间数据到文件
  if (options.save_workspace) {

    std::stringstream ss;

    // 构建输出文件名，包含问题大小信息
    ss << "23_ampere_gemm_operand_reduction_fusion"
      << options.problem_size.m() << "x" << options.problem_size.n() << "x" << options.problem_size.k()
      << ".dat";

    std::ofstream output_workspace(ss.str());

    // 输出输入矩阵 A 和 B
    output_workspace
      << "A = \n" << tensor_a.host_view() << "\n\n"
      << "B = \n" << tensor_b.host_view() << "\n\n";

    // 如果进行了参考检查，输出参考结果
    if (options.reference_check) {
      output_workspace << "Reference D = \n" << tensor_ref_d.host_view() << "\n\n";
      output_workspace << "Reference reduction vector = \n" << tensor_ref_reduction.host_view() << "\n\n";
    }

    // 输出计算结果
    output_workspace << "Computed D = \n" << tensor_d.host_view() << std::endl;
    output_workspace << "Computed reduction vector = \n" << tensor_reduction.host_view() << std::endl;

    std::cout << "Results written to '" << ss.str() << "'." << std::endl;
  }
 
  //
  // 性能测量
  //

  // 如果需要测量性能
  if (options.measure_performance) {

    // 创建 CUDA 事件用于计时
    hipEvent_t events[2];
    
    // 创建两个事件：开始和结束
    for (auto & event : events) {
      result.error = hipEventCreate(&event);
      if (result.error != hipSuccess) {
        std::cerr << "hipEventCreate() failed: " << hipGetErrorString(result.error) << std::endl;
        return result;
      }
    }

    // 在执行 GEMM 操作之前记录开始事件
    result.error = hipEventRecord(events[0]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 重复执行 GEMM 操作以获得平均性能
    for (int iteration = 0; iteration < options.iterations; ++iteration) {
      result.status = gemm_op();  // 执行一次 GEMM 操作
      CUTLASS_CHECK(result.status);
    }

    // 在所有 GEMM 操作启动后记录结束事件
    result.error = hipEventRecord(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventRecord() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 等待所有 GPU 工作完成
    result.error = hipEventSynchronize(events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "hipEventSynchronize() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 计算两个事件之间的经过时间（毫秒）
    float runtime_ms = 0;
    result.error = hipEventElapsedTime(&runtime_ms, events[0], events[1]);
    if (result.error != hipSuccess) {
      std::cerr << "cudaEventElapsed() failed: " << hipGetErrorString(result.error) << std::endl;
      return result;
    }

    // 计算平均运行时间（总时间 / 迭代次数）
    result.runtime_ms = double(runtime_ms) / double(options.iterations);

    // 销毁 CUDA 事件，释放资源
    for (auto event : events) {
      (void)hipEventDestroy(event);
    }
  }

  return result;
}

// 主函数：程序入口点
int main(int argc, char const **args) {

  bool notSupported = false;  // 标记是否支持当前硬件

  // Ampere Tensor Core 操作通过 mma.sync 暴露，首次在 CUDA 11.0 中可用
  //
  // CUTLASS 必须使用 CUDA 11 工具包编译才能运行 Conv2dFprop 示例
  if (!(__CUDACC_VER_MAJOR__ > 11 || (__CUDACC_VER_MAJOR__ == 11 && __CUDACC_VER_MINOR__ >= 0))) {
    std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  // 获取 GPU 设备属性
  hipDeviceProp_t props;
  CUDA_CHECK(hipGetDeviceProperties(&props, 0));  // 获取设备 0 的属性

  // 检查计算能力是否满足 Ampere 架构要求（SM 8.0 或更高）
  if (!(props.major >= 8)) {
    std::cerr << "Ampere Tensor Ops must be run on a machine with compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    // 如果不支持，返回 0（避免测试失败）
    return 0;
  }

  // 创建选项对象并解析命令行参数
  Options options;

  options.parse(argc, args);

  // 如果请求帮助，显示使用说明
  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (options.benchmark) {
    // 运行基准测试：测试多个预定义的问题大小

    // 定义基准测试的问题大小和配置
    struct Benchmark {
      int m, n, k, split_k_slices, parallel_split_k;
    } problem_sizes[] = {
      {4096, 6144, 4096, 1, false},  // 大型矩阵，无 split-K
    };

    // 打印结果表头
    Result::print_header(std::cout, options) << "\n";

    int idx = 1;

    // 遍历所有问题大小并运行测试
    for (auto const &problem_size : problem_sizes) {
      // 更新选项为当前问题大小
      options.update({problem_size.m, problem_size.n, problem_size.k},
                     problem_size.split_k_slices, problem_size.parallel_split_k);

      // 运行性能分析
      Result result = profile(options);
      // 打印结果
      result.print(std::cout, idx, options) << "\n";

      ++idx;
    }
  } else {

    // 执行单个问题大小测试
    if (!options.valid()) {
      // 检查问题大小是否有效（对齐要求等）
      std::cerr << "Invalid problem." << "\n";
      return -1;
    }

    // 运行性能分析
    Result result = profile(options);

    // 打印结果
    Result::print_header(std::cout, options) << "\n";
    result.print(std::cout, 1, options) << "\n";
  }

  return 0;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
