/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
 * CUTLASS Example 39: GEMM with Tensor Permutation
 *
 * This example demonstrates GEMM operations with fused tensor permutation/transposition
 * applied to input and/or output matrices. This fusion is crucial for efficient implementation
 * of complex tensor operations commonly found in modern deep learning workloads.
 *
 * FUSION OVERVIEW:
 * ================
 * The kernel performs: D_permuted = permute(alpha * A_permuted @ B_permuted + beta * C)
 * where permutation operations are applied as part of the memory access pattern,
 * avoiding separate kernels for tensor reshaping and transposition.
 *
 * PERMUTATION PATTERNS:
 * ====================
 * This example supports several categories of tensor permutations:
 *
 * 1. Normal GEMM Permutations:
 *    - Tensor4DPermute0213: [X, Y] → [X/S1, S1, S2, Y/S2] → permute([0,2,1,3]) → [X*S2/S1, Y*S1/S2]
 *    - Tensor5DPermute20314: [X, Y] → [X/T1, T1, T2, T3, Y/T2/T3] → permute([2,0,3,1,4]) → [X*T2/T1, Y*T1/T2]
 *
 * 2. Batched GEMM Permutations:
 *    - Tensor4DPermuteBMM0213: [B, X, Y] → [B/D1, D1, X, Y] → permute([0,2,1,3]) → [B/D1, X, Y*D1]
 *
 * PERFORMANCE BENEFITS:
 * ====================
 * 1. Memory Bandwidth Efficiency: Eliminates separate permutation kernels
 * 2. Cache Optimization: Improved spatial and temporal locality
 * 3. Kernel Launch Overhead: Single kernel vs. separate GEMM + permute launches
 * 4. Memory Footprint: Reduced intermediate storage requirements
 *
 * KEY ARCHITECTURAL FEATURES:
 * ===========================
 * - Flexible layout plugin system for custom permutation patterns
 * - Tensor Core acceleration with optimized memory access patterns
 * - Support for both normal and batched GEMM operations
 * - Configurable alignment for optimal memory coalescing
 * - Runtime tensor shape validation and constraint checking
 *
 * COMMON USE CASES:
 * =================
 * 1. Transformer Attention: Query/Key/Value matrix transpositions
 * 2. Convolution as GEMM: Im2col transformations with reshaping
 * 3. Tensor Contractions: Multi-dimensional matrix multiplications
 * 4. Data Layout Conversions: NCHW ↔ NHWC transformations
 * 5. Batch Processing: Efficient batched operations with reordering
 *
 * LAYOUT PLUGIN ARCHITECTURE:
 * ===========================
 * The permutation system uses a plugin architecture defined in:
 * include/cutlass/layout/permute.h
 *
 * Key components:
 * - Address computation functions: compute(col, row, stride, batch_idx)
 * - Dimension tracking: {col_permute, row_permute, stride_permute}
 * - Memory alignment optimization for permuted access patterns
 *
 * IMPLEMENTATION CONSTRAINTS:
 * ===========================
 * 1. Batch Stride Configuration:
 *    - Set batch_stride = 0 for BMM permutations
 *    - Use GemmUniversalMode::kBatched (not kArray) for batched operations
 *
 * 2. Memory Alignment Requirements:
 *    - Alignment = 1 when contiguous dimension is permuted
 *    - Alignment = 8 (or higher) when unit stride dimension is preserved
 *    - Row-major: [0,2,3,1] permutation requires Alignment = 1
 *    - Column-major: [1,0,2,3] permutation requires Alignment = 1
 *
 * 3. Performance Optimization:
 *    - Avoid permuting the unit stride dimension for best performance
 *    - Larger alignment values improve memory throughput
 *    - Consider memory access patterns in permutation design
 *
 * NUMERICAL PROPERTIES:
 * =====================
 * - Maintains identical numerical results to unpermuted GEMM operations
 * - Permutation affects only memory layout, not computation precision
 * - Deterministic results for fixed input ordering
 *
 * USAGE EXAMPLES:
 * ===============
 *   # Run batched GEMM with 96 batches and default permutations
 *   $ ./39_gemm_permute --batch-count=96
 *
 *   # Run with custom dimensions and verbose output
 *   $ ./39_gemm_permute --batch-count=96 --k=1024 --verbose=true
 *
 *   # Profile with NSight Compute
 *   $ nv-nsight-cu-cli ./39_gemm_permute --m=256 --n=192 --k=256 --verbose=true --iterations=1 --reference-check=false
 *
 * COMPILE-TIME CONFIGURATION:
 * ===========================
 * Permutation parameters (S1, S2, D1, T1, T2, T3) are compile-time constants
 * defined below. Runtime specification is not currently supported due to
 * template instantiation requirements.
 */

/////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <map>
#include <unordered_map>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/gemm/device/gemm_universal.h"

#include "cutlass/util/command_line.h"
#include "cutlass/util/distribution.h"
#include "cutlass/util/device_memory.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/device/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_norm.h"

#include "cutlass/layout/permute.h"

#include "layouts.h"
#include "permute_info.h"

///////////////////////////////////////////////////////////////////////////////////////////////////
///
/// COMPILE-TIME PERMUTATION CONFIGURATION
/// =======================================
/// These constants define the tensor reshaping and permutation patterns.
/// All values are compile-time constants and cannot be changed at runtime.
///
///////////////////////////////////////////////////////////////////////////////////////////////////

/// Batched GEMM 4D Permutation: Tensor4DPermuteBMM0213
/// ====================================================
/// Pattern: [B, M, N] → [B/D1, D1, M, N] → permute([0, 2, 1, 3]) → [B/D1, M, D1, N]
/// Use case: Batched matrix multiplication with inter-batch dimension reordering
int constexpr D1 = 12;  // Batch subdivision factor

/// Normal GEMM 5D Permutation: Tensor5DPermute20314
/// ================================================
/// Pattern: [M, N] → [M/T1, T1, T2, T3, N/T2/T3] → permute([2, 0, 3, 1, 4]) → [T2, M/T1, T3, T1, N/T2/T3]
/// Use case: Complex tensor contractions and multi-dimensional reorderings
int constexpr T1 = 16;  // Primary dimension subdivision
int constexpr T2 = 3;   // Secondary dimension subdivision
int constexpr T3 = 8;   // Tertiary dimension subdivision

/// Normal GEMM 4D Permutation: Tensor4DPermute0213
/// ===============================================
/// Pattern: [M, N] → [M/S1, S1, S2, N/S2] → permute([0, 2, 1, 3]) → [M/S1, S2, S1, N/S2]
/// Use case: Standard matrix transposition with tiling for cache efficiency
int constexpr S1 = 8;   // Row tile size
int constexpr S2 = 4;   // Column tile size

/// Memory Alignment Configuration
/// ===============================
/// Alignment requirements balance memory bandwidth with permutation constraints
/// Higher alignment improves throughput when compatible with permutation patterns
int constexpr AlignmentA = 8;  // Matrix A alignment (8 elements = 16 bytes for FP16)
int constexpr AlignmentB = 8;  // Matrix B alignment (8 elements = 16 bytes for FP16)
int constexpr AlignmentC = 8;  // Matrix C/D alignment (8 elements = 16 bytes for FP16)

/// Data Type Configuration
/// =======================
/// Optimized for modern GPU architectures with mixed-precision support
using ElementInput = cutlass::half_t;     // Input matrices: FP16 for memory efficiency
using ElementOutput = cutlass::half_t;    // Output matrix: FP16 for downstream compatibility
using ElementAccumulator = float;         // Internal accumulation: FP32 for numerical accuracy

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Error Handling Macros
/// ======================
/// Convenience macros for consistent error checking throughout the example

// CUDA Runtime API error checking
#define CHECK_CUDA_CALL(call, handler) \
do { \
  hipError_t __err = (call); \
  if (__err != hipSuccess) { \
    std::cerr << #call " failed: " << hipGetErrorString(__err) << std::endl; \
    handler; \
  } \
} while(0)

// CUTLASS API error checking
#define CHECK_CUTLASS_CALL(call, handler) \
do { \
  cutlass::Status __status = (call); \
  if (__status != cutlass::Status::kSuccess) { \
    std::cerr << #call " failed: " << cutlass::cutlassGetStatusString(__status) << std::endl; \
    handler; \
  } \
} while(0)

/////////////////////////////////////////////////////////////////////////////////////////////////

/// Command Line Configuration
/// ===========================
/// Comprehensive options structure for controlling GEMM dimensions,
/// batching parameters, and execution settings
struct Options {

  bool help;
  bool error;
  bool reference_check;

  cutlass::gemm::GemmCoord problem_each;

  int batch_count;
  int iterations;
  int cuda_streams;
  bool verbose;
  float alpha;
  float beta;

  //
  // Methods
  // 

  Options():
    help(false),
    error(false),
    reference_check(true),    // Enable numerical verification by default
    batch_count(-1),          // Will be set to default in parse() if not specified
    iterations(20),           // Performance measurement iterations
    cuda_streams(0),          // Number of CUDA streams (0 = synchronous)
    verbose(false),           // Detailed output disabled by default
    alpha(1),                 // GEMM scaling factor
    beta()                    // Bias scaling factor (zero by default)
  { }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
      return;
    }

    cmd.get_cmd_line_argument("alpha", alpha, 1.0f);
    cmd.get_cmd_line_argument("beta", beta, 0.0f);    
    cmd.get_cmd_line_argument("iterations", iterations, 20);
    cmd.get_cmd_line_argument("streams", cuda_streams, 0);
    cmd.get_cmd_line_argument("verbose", verbose, false);
    cmd.get_cmd_line_argument("reference-check", reference_check, true);

    int m, n, k;

    // Parse GEMM dimensions with reasonable defaults for demonstration
    cmd.get_cmd_line_argument("m", m, 384);           // M dimension (384 = multiple of tile sizes)
    cmd.get_cmd_line_argument("n", n, 192);           // N dimension (192 = multiple of tile sizes)
    cmd.get_cmd_line_argument("k", k, 384);           // K dimension (384 = multiple of tile sizes)
    cmd.get_cmd_line_argument("batch-count", batch_count, 96);  // 96 batches for meaningful statistics

    problem_each = cutlass::gemm::GemmCoord(m, n, k);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << 
      "39_gemm_permute\n"
      "\n"
      " This example tests and profiles the performance of normal GEMM and batched GEMM with different"
      " combinations of fused permutations of input and output tensors."
      "\n"
      " Permutations considered in this example:\n"
      "\n"
      " Normal GEMM:\n"
      " 1) Tensor4DPermute0213: matrix of shape [X, Y] is reshaped as [X/S1, S1, S2, Y/S2] and has its dimensions"
      " permuted as [0, 2, 1, 3], resulting in shape [X/S1, S2, S1, Y/S2] viewed as matrix of shape [X*S2/S1, Y*S1/S2].\n"
      " 2) Tensor5DPermute20314: matrix of shape [X, Y] is reshaped as [X/T1, T1, T2, T3, Y/T2/T3] and has its dimensions"
      " permuted as [2, 0, 3, 1, 4], resulting in shape [T2, X/T1, T3, T1, Y/T2/T3] viewed as matrix of shape [X*T2/T1, Y*T1/T2].\n"
       "\n"
      " Batched GEMM:\n"
      " 3) Tensor4DPermuteBMM0213: batched tensor of 3D shape [B, X, Y] is reshaped as 4D shape [B/D1, D1, X, Y]"
      " and has its dimensions permuted as [0, 2, 1, 3], resulting in shape [B/D1, X, D1, Y] viewed as"
      " a matrix of shape [B/D1, X, Y*D1] for batched GEMM purposes.\n"
      "\n"
      " Note: S1, S2, D1, D2, T1, T2, T3 are compile-time constants defined in gemm_permute.cu."
      " Runtime specification of these values is not supported."
      " These values along with alignment requirements place constraints on supported matrix sizes.\n"
      "\n"
      " Note: X, Y above may refer to M, N or K dimensions of GEMM problem, depending on the tensor considered (A, B or D)."
      " For the output tensor D the values correspond directly to dimensions of D, whereas for A and B the original dimensions"
      " X', Y' are inferred from the ones supplied to the GEMM, taking into account the permute operation.\n"
      "\n"
      "Options:\n"
      "\n"
      "  --help                      If specified, displays this usage statement.\n\n"
      "  --batch-count=<int>         Sets the number of batches in batched GEMM (batch number for BMM). (default: --batch-count=768)\n"
      "  --m=<int>                   Sets the M dimension for both batched GEMM and normal GEMM problems. (default: --m=128)\n"
      "  --n=<int>                   Sets the N dimension for both batched GEMM and normal GEMM problems. (default: --n=192)\n"
      "  --k=<int>                   Sets the K dimension for both batched GEMM and normal GEMM problems. (default: --k=384)\n"
      "  --alpha=<f32>               Epilogue scalar alpha (real part)\n"
      "  --beta=<f32>                Epilogue scalar beta (real part)\n\n"
      "  --iterations=<int>          Number of profiling iterations to perform.\n"
      "  --reference-check=<bool>    If true, performs reference check.\n"
      "  --verbose=<bool>            If true, prints problem sizes and batching structure.\n"
      "\n"
      "Examples:\n"
      "\n"
      "# Runs a batched GEMM with 96 batches\n"
      "$ ./examples/39_gemm_permute/39_gemm_permute --batch-count=96\n"
      "\n"
      "# Runs a batched GEMM with 96 batches (with GEMM-K dimension equal to 1024)\n"
      "$ ./examples/39_gemm_permute/39_gemm_permute --batch-count=96 --k=1024 --verbose=true\n"
      "\n"
      "# Execute batched GEMM and profile with NSight\n"
      "$ nv-nsight-cu-cli ./examples/39_gemm_permute/39_gemm_permute --m=256 --n=192 --k=256 --verbose=true --iterations=1 --reference-check=false\n"
      "\n";

    return out;
  }

  /// Performance Calculation
  /// =======================
  /// Computes effective throughput in GFLOP/s for the permuted GEMM operations
  double gflops(double runtime_s, bool batched) const {

    // Calculate total multiply-add operations
    // Each GEMM contributes M*N*K multiply-adds
    int64_t fmas = problem_each.product() * (batched ? batch_count : 1);

    // Convert to floating-point operations (2 ops per multiply-add)
    return 2.0 * double(fmas) / double(1.0e9) / runtime_s;
  }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

namespace { // Anonymous namespace for implementation details

/// Recursive Host-Based Permutation Implementation
/// ===============================================
/// Template-based recursive function for applying arbitrary tensor permutations on CPU
/// Used for reference computation and verification purposes
template<int I, typename Element, typename Layout, typename PermuteOp, typename Coord>
void permute_host_impl(
    cutlass::TensorView<Element const, Layout> const & input,   // Source tensor view
    cutlass::TensorView<Element, Layout> const & output,        // Destination tensor view
    PermuteOp && permute,                                       // Permutation operation
    Coord & coord                                               // Current coordinate being processed
) {
  static_assert(Layout::kRank == Coord::kRank, "Layout and Coordinate ranks must match");

  if constexpr (I == Coord::kRank) {
    // Base case: copy element with permuted coordinates
    output.at(permute(coord)) = input.at(coord);
  } else {
    // Recursive case: iterate through dimension I
    for (coord[I] = 0; coord[I] < input.extent(I); ++coord[I]) {
      permute_host_impl<I+1>(input, output, std::forward<PermuteOp>(permute), coord);
    }
  }
}

} // namespace (anonymous)

/// Host-Based Reference Permutation
/// =================================
/// Performs tensor permutation on CPU for verification against GPU kernel results
/// Supports arbitrary permutation patterns defined by PermuteLayout template parameter
template<typename PermuteLayout, typename Element, typename Layout>
void permute_host(
    cutlass::TensorView<Element const, Layout> const &input,    // Input tensor (device memory)
    cutlass::TensorView<Element, Layout> const &output,         // Output tensor (device memory)
    int batch_count                                             // Number of batched tensors
) {
  // Extract tensor properties and allocate host memory
  Layout layout = input.layout();
  cutlass::MatrixCoord extent = input.extent();

  std::size_t num_elems = layout.capacity(extent) * batch_count;
  std::vector<Element> h_input(num_elems);   // Host input buffer
  std::vector<Element> h_output(num_elems);  // Host output buffer

  // Copy input data from device to host
  cutlass::device_memory::copy_to_host(h_input.data(), input.data(), num_elems);

  // Configure permutation using template parameter information
  using Info = PermuteInfo<PermuteLayout>;
  using TensorLayout = typename Info::Layout;

  // Calculate original and permuted tensor shapes
  auto shape_orig = Info::original_shape(extent, batch_count);
  auto shape_perm = Info::permute(shape_orig);

  // Create tensor views for the permutation operation
  cutlass::TensorView<Element const, TensorLayout> view_input(
    h_input.data(), TensorLayout::packed(shape_orig), shape_orig);
  cutlass::TensorView<Element, TensorLayout> view_output(
    h_output.data(), TensorLayout::packed(shape_perm), shape_perm);

  // Execute the permutation using recursive template implementation
  decltype(shape_orig) coord;
  permute_host_impl<0>(view_input, view_output, Info::permute, coord);

  // Copy permuted results back to device memory
  cutlass::device_memory::copy_to_device(output.data(), h_output.data(), num_elems);
}

/// Layout Information Helper
/// =========================
/// Template specializations for converting layout types to human-readable names
/// Used for verbose output and debugging information

template<typename Layout>
struct LayoutInfo;

template<>
struct LayoutInfo<cutlass::layout::RowMajor> {
  static std::string name() { return "RowMajor"; }
};

template<>
struct LayoutInfo<cutlass::layout::ColumnMajor> {
  static std::string name() { return "ColumnMajor"; }
};

///////////////////////////////////////////////////////////////////////////////////////////////////

/// GEMM Permutation Testbed
/// =========================
/// Comprehensive test harness for validating and benchmarking GEMM operations
/// with various tensor permutation patterns applied to inputs and outputs
template <typename ElementA, typename ElementB, typename ElementC>
class Testbed {
private:

  //
  // Internal State and Configuration
  // ================================
  //

  Options & options;  // Reference to command-line configuration

  // Random initialization parameters for reproducible testing
  cutlass::Distribution::Kind init_A;  // Distribution type for matrix A
  cutlass::Distribution::Kind init_B;  // Distribution type for matrix B
  cutlass::Distribution::Kind init_C;  // Distribution type for matrix C
  uint32_t seed;                       // Random seed for deterministic results

  // GPU memory allocations for input/output matrices
  cutlass::DeviceAllocation<ElementA> block_A;  // Input matrix A
  cutlass::DeviceAllocation<ElementB> block_B;  // Input matrix B
  cutlass::DeviceAllocation<ElementC> block_C;  // Input matrix C (bias)
  cutlass::DeviceAllocation<ElementC> block_D;  // Output matrix D

public:

  //
  // Methods
  //

  /// Constructor: Testbed Initialization
  /// ===================================
  /// Configures the test environment with specified data distributions and random seed
  Testbed(
    Options &options_,
    cutlass::Distribution::Kind init_A_ = cutlass::Distribution::Uniform,  // Matrix A: uniform distribution
    cutlass::Distribution::Kind init_B_ = cutlass::Distribution::Uniform,  // Matrix B: uniform distribution
    cutlass::Distribution::Kind init_C_ = cutlass::Distribution::Uniform,  // Matrix C: uniform distribution
    uint32_t seed_ = 3090                                                  // Default random seed
  ):
    options(options_), init_A(init_A_), init_B(init_B_), init_C(init_C_), seed(seed_) { }

private:

  /// Tensor Information Display
  /// ===========================
  /// Prints detailed information about tensor dimensions and permutation patterns
  /// for debugging and verification purposes
  template<typename PermuteLayout>
  void print_tensor_info(
      std::ostream & os,           // Output stream for information display
      std::string const &tensor_name,  // Human-readable tensor identifier
      int row_dim,                     // Row dimension index in problem coordinates
      int col_dim                      // Column dimension index in problem coordinates
  ) {

    // Extract tensor dimensions and permutation metadata
    cutlass::MatrixCoord extent(options.problem_each.at(row_dim), options.problem_each.at(col_dim));
    using Info = PermuteInfo<PermuteLayout>;

    // Display basic tensor information
    os << "Tensor " << tensor_name << ": " << Info::desc() << "\n";
    os << "    Extent: [" << extent.row() << ", " << extent.column() << "]";
    if (Info::kBatched) {
      os << ", Batch count: " << options.batch_count;
    }
    os << "\n";

    // Display permutation details for non-trivial cases
    if (!cutlass::layout::is_trivial_permute<PermuteLayout>) {
      auto shape_orig = Info::original_shape(extent, options.batch_count);
      auto shape_perm = Info::permute(shape_orig);
      os << "    Original shape: [" << shape_orig << "]\n";
      os << "    Permuted shape: [" << shape_perm << "]\n";
    }
  }

  /// Tensor Shape Validation
  /// ========================
  /// Validates that tensor dimensions are compatible with permutation requirements
  /// and memory alignment constraints
  template<typename Layout, typename PermuteLayout, int Alignment>
  bool check_tensor_shape(
      std::string const &tensor_name,  // Tensor identifier for error reporting
      int row_dim,                     // Row dimension index
      int col_dim                      // Column dimension index
  ) {

    // Extract tensor dimensions and permutation requirements
    cutlass::MatrixCoord extent(options.problem_each.at(row_dim), options.problem_each.at(col_dim));
    using Info = PermuteInfo<PermuteLayout>;

    // Calculate alignment requirements based on memory layout
    // Column-major: alignment applies to rows (leading dimension)
    // Row-major: alignment applies to columns (leading dimension)
    auto rowAlign = cutlass::platform::is_same<Layout, cutlass::layout::ColumnMajor>::value ? Alignment : 1;
    auto colAlign = cutlass::platform::is_same<Layout, cutlass::layout::RowMajor>::value ? Alignment : 1;

    // Combine permutation and alignment requirements
    auto rowFactor = Info::kRowFactor * rowAlign;
    auto colFactor = Info::kColumnFactor * colAlign;

    // Validate row dimension divisibility
    bool const valid_row = extent.row() % rowFactor == 0;
    if (!valid_row) {
      std::cerr << "ERROR: Tensor " << tensor_name << " row size (" << extent.row()
                << ") must be divisible by " << rowFactor
                << " (required by " << Info::name()
                << (rowAlign > 1 ? (" + alignment " + std::to_string(rowAlign)) : "")
                << ")" << std::endl;
    }

    // Validate column dimension divisibility
    bool const valid_col = extent.column() % colFactor == 0;
    if (!valid_col) {
      std::cerr << "ERROR: Tensor " << tensor_name << " column size (" << extent.column()
                << ") must be divisible by " << colFactor
                << " (required by " << Info::name()
                << (colAlign > 1 ? (" + alignment " + std::to_string(colAlign)) : "")
                << ")" << std::endl;
    }

    // Validate batch count divisibility for batched operations
    bool const valid_bsz = options.batch_count % Info::kBatchFactor == 0;
    if (!valid_bsz) {
      std::cerr << "ERROR: Batch count (" << options.batch_count
                << ") must be divisible by " << Info::kBatchFactor
                << " (required by " << Info::name() << ")" << std::endl;
    }

    return valid_row && valid_col && valid_bsz;
  }

  /// Tensor Data Initialization
  /// ===========================
  /// Fills tensor memory with values from specified probability distribution
  /// Range selection ensures numerical stability and meaningful verification
  template <typename Element>
  void initialize_tensor_(
      Element *ptr,                           // Device memory pointer
      size_t capacity,                        // Number of elements to initialize
      cutlass::Distribution::Kind dist_kind,  // Distribution type
      uint32_t seed                           // Random seed
  ) {

    if (dist_kind == cutlass::Distribution::Uniform) {
      // Determine value range based on element precision
      Element scope_max, scope_min;
      int bits_input = cutlass::sizeof_bits<Element>::value;
      int bits_output = cutlass::sizeof_bits<ElementC>::value;

      if (bits_input == 1) {
        // Binary values
        scope_max = 2;
        scope_min = 0;
      } else if (bits_input <= 8) {
        // Low precision (INT8, etc.)
        scope_max = 2;
        scope_min = -2;
      } else if (bits_output == 16) {
        // Half precision output
        if (cutlass::sizeof_bits<ElementAccumulator>::value <= 16) {
          scope_max = 5;   // Conservative range for FP16 accumulation
          scope_min = -5;
        } else {
          scope_max = 8;   // Wider range for FP32 accumulation
          scope_min = -8;
        }
      } else {
        // Full precision
        scope_max = 8;
        scope_min = -8;
      }

      // Fill with uniform random values in calculated range
      cutlass::reference::device::BlockFillRandomUniform(
        ptr, capacity, seed, scope_max, scope_min, 0);
    } 
    else if (dist_kind == cutlass::Distribution::Gaussian) {
      // Gaussian distribution with mean=0, stddev=0.5
      cutlass::reference::device::BlockFillRandomGaussian(
        ptr, capacity, seed, Element(0), Element(0.5f));
    }
    else if (dist_kind == cutlass::Distribution::Sequential) {
      // Sequential values: 0, 1, 2, 3, ... (useful for debugging)
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(1), Element(0));
    }
    else {
      // Identity/constant fill: all elements = 1
      cutlass::reference::device::BlockFillSequential(
        ptr, capacity, Element(0), Element(1));
    }
  }

  /// Memory Allocation and Data Initialization
  /// ===========================================
  /// Allocates GPU memory and fills tensors with test data
  void initialize(int batch_count) {

    // Set random seed for reproducible results
    srand(seed);

    // Calculate total memory requirements for all batches
    int64_t total_elements_A = options.problem_each.m() * options.problem_each.k() * batch_count;
    int64_t total_elements_B = options.problem_each.n() * options.problem_each.k() * batch_count;
    int64_t total_elements_C = options.problem_each.m() * options.problem_each.n() * batch_count;
    int64_t total_elements_D = options.problem_each.m() * options.problem_each.n() * batch_count;

    // Allocate GPU memory for all matrices
    block_A.reset(total_elements_A);  // Input matrix A
    block_B.reset(total_elements_B);  // Input matrix B
    block_C.reset(total_elements_C);  // Input matrix C (bias)
    block_D.reset(total_elements_D);  // Output matrix D

    // Initialize input tensors with specified distributions
    // Different seeds ensure uncorrelated random data
    initialize_tensor_(block_A.get(), total_elements_A, init_A, seed * 2021);
    initialize_tensor_(block_B.get(), total_elements_B, init_B, seed * 2022);
    initialize_tensor_(block_C.get(), total_elements_C, init_C, seed * 2023);

    // Initialize output tensor to zero (will be overwritten)
    cutlass::reference::device::BlockFillSequential(
      block_D.get(), total_elements_D, ElementC(0), ElementC(0));
  }


  /// Numerical Verification Against Reference Implementation
  /// =======================================================
  /// Validates GPU kernel results by comparing against CPU-based reference computation
  /// with separate host-side permutation operations
  template<typename Gemm>
  bool validate(Gemm const &gemm) {

    // Determine if this is a batched operation by checking permutation layouts
    bool constexpr kBatched = PermuteInfo<typename Gemm::PermuteALayout>::kBatched
                           || PermuteInfo<typename Gemm::PermuteBLayout>::kBatched
                           || PermuteInfo<typename Gemm::PermuteDLayout>::kBatched;

    int const batch_count = kBatched ? options.batch_count : 1;

    // Extract problem dimensions and create tensor layouts
    cutlass::gemm::GemmCoord problem = options.problem_each;
    cutlass::MatrixCoord extent_A{problem.m(), problem.k()};
    cutlass::MatrixCoord extent_B{problem.k(), problem.n()};
    cutlass::MatrixCoord extent_C{problem.m(), problem.n()};

    // Extract layout types from GEMM template
    using LayoutA = typename Gemm::LayoutA;
    using LayoutB = typename Gemm::LayoutB;
    using LayoutC = typename Gemm::LayoutC;

    // Create packed layouts for efficient memory access
    LayoutA layout_A(LayoutA::packed(extent_A));
    LayoutB layout_B(LayoutB::packed(extent_B));
    LayoutC layout_C(LayoutC::packed(extent_C));

    // Calculate total memory sizes including batching
    auto size_A = layout_A.capacity(extent_A) * batch_count;
    auto size_B = layout_B.capacity(extent_B) * batch_count;
    auto size_C = layout_C.capacity(extent_C) * batch_count;
    
    // Create tensor views for original data
    cutlass::TensorView<ElementA, LayoutA> view_A(block_A.get(), layout_A, extent_A);
    cutlass::TensorView<ElementB, LayoutB> view_B(block_B.get(), layout_B, extent_B);
    cutlass::TensorView<ElementC, LayoutC> view_C(block_C.get(), layout_C, extent_C);
    cutlass::TensorView<ElementC, LayoutC> view_D(block_D.get(), layout_C, extent_C);

    // Allocate temporary storage for permuted input matrices
    cutlass::DeviceAllocation<ElementA> block_A_perm(size_A);
    cutlass::DeviceAllocation<ElementA> block_B_perm(size_B);

    // Create tensor views for permuted data
    cutlass::TensorView<ElementA, LayoutA> view_A_perm(block_A_perm.get(), layout_A, extent_A);
    cutlass::TensorView<ElementB, LayoutB> view_B_perm(block_B_perm.get(), layout_B, extent_B);

    // Apply input permutations using host-based reference implementation
    permute_host<typename Gemm::PermuteALayout>(view_A.const_view(), view_A_perm, batch_count);
    permute_host<typename Gemm::PermuteBLayout>(view_B.const_view(), view_B_perm, batch_count);

    // Allocate storage for reference GEMM output
    cutlass::DeviceAllocation<ElementC> block_D_ref(size_C);
    cutlass::TensorView<ElementC, LayoutC> view_D_ref(block_D_ref.get(), layout_C, extent_C);

    // Extract epilogue configuration from GEMM template
    using EpilogueOutputOp = typename Gemm::GemmKernel::Epilogue::OutputOp;

    // Compute reference GEMM using permuted inputs
    // This performs: D_ref = alpha * A_permuted @ B_permuted + beta * C
    cutlass::reference::device::GemmComplex<
        ElementA, LayoutA,                              // Input A configuration
        ElementB, LayoutB,                              // Input B configuration
        ElementC, LayoutC,                              // Output configuration
        typename EpilogueOutputOp::ElementCompute,      // Epilogue compute type
        typename Gemm::ElementAccumulator               // Accumulator type
    >(
      problem,                                        // GEMM dimensions
      options.alpha,                                  // Scaling factor alpha
      view_A_perm,                                    // Permuted matrix A
      Gemm::kTransformA,                              // Transform operation on A
      view_B_perm,                                    // Permuted matrix B
      Gemm::kTransformB,                              // Transform operation on B
      options.beta,                                   // Scaling factor beta
      view_C,                                         // Input matrix C
      view_D_ref,                                     // Reference output D
      ElementAccumulator(0),                          // Initial accumulator value
      batch_count,                                    // Number of batches
      options.problem_each.m() * options.problem_each.k(),  // Batch stride A
      options.problem_each.n() * options.problem_each.k(),  // Batch stride B
      options.problem_each.m() * options.problem_each.n(),  // Batch stride C
      options.problem_each.m() * options.problem_each.n()   // Batch stride D
    );

    // Apply output permutation to reference results
    cutlass::DeviceAllocation<ElementC> block_D_perm(size_C);
    cutlass::TensorView<ElementC, LayoutC> view_D_perm(block_D_perm.get(), layout_C, extent_C);
    permute_host<typename Gemm::PermuteDLayout>(view_D_ref.const_view(), view_D_perm, batch_count);

    // Compare permuted reference output against kernel output
    return cutlass::reference::device::BlockCompareEqual(view_D_perm.data(), view_D.data(), size_C);
}

public:

  /// GEMM Permutation Profiling and Validation
  /// ===========================================
  /// Complete workflow for testing a specific GEMM permutation configuration
  /// including validation, performance measurement, and detailed reporting
  template<typename Gemm>
  bool profile_GEMM_permute() {

    using LayoutA = typename Gemm::LayoutA;
    using LayoutB = typename Gemm::LayoutB;
    using LayoutC = typename Gemm::LayoutC;

    using PermuteALayout = typename Gemm::PermuteALayout;
    using PermuteBLayout = typename Gemm::PermuteBLayout;
    using PermuteDLayout = typename Gemm::PermuteDLayout;

    bool constexpr kBatched = PermuteInfo<PermuteALayout>::kBatched 
                           || PermuteInfo<PermuteBLayout>::kBatched 
                           || PermuteInfo<PermuteDLayout>::kBatched;

    // Display configuration header
    std::cout << "\n"
                 "====================================================\n"
                 << (kBatched ? "Batched" : "Normal") << " GEMM with Permutation:"
                 << "\n  Matrix A: " << LayoutInfo<LayoutA>::name() << " + " << PermuteInfo<PermuteALayout>::name()
                 << "\n  Matrix B: " << LayoutInfo<LayoutB>::name() << " + " << PermuteInfo<PermuteBLayout>::name()
                 << "\n  Matrix D: " << LayoutInfo<LayoutC>::name() << " + " << PermuteInfo<PermuteDLayout>::name()
                 << "\n"
                 "====================================================\n";

    // Display detailed tensor information if requested
    if (options.verbose) {
      print_tensor_info<PermuteALayout>(std::cout, "A", 0, 2);  // A: M x K
      print_tensor_info<PermuteBLayout>(std::cout, "B", 2, 1);  // B: K x N
      print_tensor_info<PermuteDLayout>(std::cout, "D", 0, 1);  // D: M x N
    }
    std::cout << std::endl;

    // Validate tensor shapes and alignment requirements
    bool valid = true;
    valid &= check_tensor_shape<LayoutA, PermuteALayout, Gemm::kAlignmentA>("A", 0, 2);
    valid &= check_tensor_shape<LayoutB, PermuteBLayout, Gemm::kAlignmentB>("B", 2, 1);
    valid &= check_tensor_shape<LayoutC, PermuteDLayout, Gemm::kAlignmentC>("D", 0, 1);
    if (!valid) {
      std::cout << "SKIPPED: Invalid tensor dimensions for this permutation pattern" << std::endl;
      return true;
    }

    // Determine effective batch count and initialize data
    int const batch_count = kBatched ? options.batch_count : 1;
    initialize(batch_count);

    // Configure epilogue operation (linear combination: alpha*AB + beta*C)
    using EpilogueOutputOp = typename Gemm::GemmKernel::Epilogue::OutputOp;
    typename EpilogueOutputOp::Params epilogue_op(options.alpha, options.beta);

    // Extract problem dimensions (uniform across all batches)
    auto problem = options.problem_each;
    cutlass::MatrixCoord extent_A{problem.m(), problem.k()};
    cutlass::MatrixCoord extent_B{problem.k(), problem.n()};
    cutlass::MatrixCoord extent_C{problem.m(), problem.n()};

    // Create optimized memory layouts
    LayoutA layout_A(LayoutA::packed(extent_A));
    LayoutB layout_B(LayoutB::packed(extent_B));
    LayoutC layout_C(LayoutC::packed(extent_C));

    // Configure comprehensive GEMM kernel arguments
    typename Gemm::Arguments arguments{
      kBatched ? cutlass::gemm::GemmUniversalMode::kBatched : cutlass::gemm::GemmUniversalMode::kGemm,
      problem,                                              // Problem dimensions
      batch_count,                                          // Number of batches
      epilogue_op,                                          // Linear combination parameters
      (void*)block_A.get(),                                 // Matrix A device pointer
      (void*)block_B.get(),                                 // Matrix B device pointer
      (void*)block_C.get(),                                 // Matrix C device pointer
      (void*)block_D.get(),                                 // Matrix D device pointer
      // Batch stride configuration (critical for permuted layouts)
      // Non-trivial permutations require batch_stride = 0
      cutlass::layout::is_trivial_permute<PermuteALayout> ? layout_A.capacity(extent_A) : 0,
      cutlass::layout::is_trivial_permute<PermuteBLayout> ? layout_B.capacity(extent_B) : 0,
      layout_C.capacity(extent_C),                         // Matrix C batch stride
      cutlass::layout::is_trivial_permute<PermuteDLayout> ? layout_C.capacity(extent_C) : 0,
      layout_A.stride(0),                                   // Matrix leading dimensions
      layout_B.stride(0),
      layout_C.stride(0),
      layout_C.stride(0),
    };

    //
    // Kernel Execution
    // ================
    //

    // Initialize GEMM kernel with configured arguments
    Gemm gemm_permute;
    CHECK_CUTLASS_CALL(gemm_permute.initialize(arguments, nullptr), return false);

    // Execute initial kernel run
    CHECK_CUTLASS_CALL(gemm_permute.run(), return false);

    // Synchronize to ensure completion before verification
    CHECK_CUDA_CALL(hipDeviceSynchronize(), return false);

    //
    // Numerical Verification
    // ======================
    //
    if (options.reference_check) {
      if (validate(gemm_permute)) {
        std::cout << "\n✓ PASSED: Numerical verification successful\n" << std::endl;
      } else {
        std::cerr << "\n✗ FAILED: Numerical verification failed\n" << std::endl;
        std::cerr << "Kernel output does not match reference implementation.\n" << std::endl;
        return false;
      }
    }

    //
    // Performance Measurement
    // =======================
    //

    // Warm-up run to stabilize GPU clocks
    CHECK_CUTLASS_CALL(gemm_permute.run(), return false);

    // Create timing events
    hipEvent_t events[2];
    for (auto & event : events) {
      CHECK_CUDA_CALL(hipEventCreate(&event), return false);
    }

    // Start timing measurement
    CHECK_CUDA_CALL(hipEventRecord(events[0]), return false);

    // Execute performance measurement loop
    for (int iter = 0; iter < options.iterations; ++iter) {
      gemm_permute();
    }

    // End timing measurement
    CHECK_CUDA_CALL(hipEventRecord(events[1]), return false);

    // Wait for all operations to complete
    CHECK_CUDA_CALL(hipEventSynchronize(events[1]), return false);

    // Calculate performance metrics
    float runtime_total_ms = 0;
    CHECK_CUDA_CALL(hipEventElapsedTime(&runtime_total_ms, events[0], events[1]), return false);

    double runtime_avg_ms = double(runtime_total_ms) / double(options.iterations);
    double gflops = options.gflops(runtime_avg_ms / 1000.0, kBatched);

    // Calculate effective bandwidth accounting for permutation overhead
    double total_bytes = double(sizeof(ElementInput)) *
                        (options.problem_each.product() * 2) * // A + B matrices
                        (kBatched ? options.batch_count : 1) +
                        double(sizeof(ElementOutput)) *
                        (options.problem_each.m() * options.problem_each.n()) * // D matrix
                        (kBatched ? options.batch_count : 1);
    double bandwidth_gbps = total_bytes / (runtime_avg_ms / 1000.0) / 1e9;

    // Cleanup timing resources
    for (auto event : events) {
      CHECK_CUDA_CALL(hipEventDestroy(event), return false);
    }

    // Display performance results
    std::cout << "Performance Results:" << std::endl;
    std::cout << "  Average Runtime: " << runtime_avg_ms << " ms" << std::endl;
    std::cout << "  Throughput: " << gflops << " GFLOP/s" << std::endl;
    std::cout << "  Memory Bandwidth: " << bandwidth_gbps << " GB/s" << std::endl;

    return true;
  }
};

/// Shorthand alist for GEMM instantiations
template<typename LayoutA, typename PermuteALayout,
         typename LayoutB, typename PermuteBLayout,
         typename LayoutC, typename PermuteDLayout>
using GemmPermute = cutlass::gemm::device::GemmUniversal<
  ElementInput, LayoutA,
  ElementInput, LayoutB,
  ElementOutput, LayoutC,
  ElementAccumulator,
  cutlass::arch::OpClassTensorOp,
  cutlass::arch::Sm80,
  cutlass::gemm::GemmShape<128, 128, 32>,
  cutlass::gemm::GemmShape<64, 64, 32>,
  cutlass::gemm::GemmShape<16, 8, 16>,
  cutlass::epilogue::thread::LinearCombination<
    ElementOutput, 
    AlignmentC, //128 / cutlass::sizeof_bits<ElementOutput>::value,
    ElementAccumulator, 
    ElementAccumulator
  >,
  cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<8>,
  4,              /*kStages*/
  AlignmentA,     /*AlignmentA*/
  AlignmentB,     /*AlignmentB*/
  cutlass::arch::OpMultiplyAdd,
  cutlass::ComplexTransform::kNone,
  cutlass::ComplexTransform::kNone,
  false,  /*GatherA*/
  false,  /*GatherB*/
  false,  /*ScatterD*/
  PermuteDLayout,  /*PermuteDLayout*/
  typename cutlass::layout::InversePermute<PermuteALayout>::type,  /*PermuteALayout*/
  typename cutlass::layout::InversePermute<PermuteBLayout>::type   /*PermuteBLayout*/
>;

///////////////////////////////////////////////////////////////////////////////////////////////////

/// Main Entry Point
/// ================
/// Orchestrates the complete GEMM permutation demonstration including
/// hardware validation, configuration setup, and execution of all test cases
int main(int argc, char const **args) {

  //
  // Hardware and Software Requirements Validation
  // =============================================
  //

  hipDeviceProp_t props;
  CHECK_CUDA_CALL(hipGetDeviceProperties(&props, 0), return EXIT_FAILURE);

  // Verify Ampere architecture and CUDA 11+ for Tensor Core support
  if (__CUDACC_VER_MAJOR__ < 11 || props.major < 8) {
    std::cout << "CUTLASS GEMM+Permutation example requires:\n"
                 "  - NVIDIA Ampere architecture (compute capability 8.0+)\n"
                 "  - CUDA Toolkit 11.0 or later\n"
                 "Current configuration is not supported.\n";
    return EXIT_SUCCESS;
  }

  //
  // Command Line Processing
  // =======================
  //

  Options options;
  options.parse(argc, args);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return EXIT_SUCCESS;
  }

  if (options.error) {
    std::cerr << "ERROR: Invalid command line arguments." << std::endl;
    return EXIT_FAILURE;
  }

  //
  // Define GEMM types to test
  //

  //
  // TTT (Row-major) GEMMs
  //

  using TTTGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteA = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using TTTGemmNormalPermuteB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using TTTGemmNormalPermuteD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using TTTGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  //
  // NNN (Col-major) GEMMs
  //

  using NNNGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteA = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using NNNGemmNormalPermuteB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using NNNGemmNormalPermuteD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using NNNGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  //
  // NNT (Col-major inputs, row-major output) GEMMs
  //

  using NNTGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteA = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using NNTGemmNormalPermuteB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using NNTGemmNormalPermuteD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  using NNTGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute
  >;

  using NNTGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermute0213ColumnMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor5DPermute20314RowMajor<T1, T2, T3>
  >;

  //
  // TTN (Row-major inputs, col-major output) GEMMs
  //

  using TTNGemmNormalPermuteNone = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteA = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteAD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using TTNGemmNormalPermuteB = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteBD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using TTNGemmNormalPermuteD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::RowMajor,    cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  using TTNGemmNormalPermuteAB = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using TTNGemmNormalPermuteABD = GemmPermute<
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::RowMajor,    cutlass::layout::Tensor4DPermute0213RowMajor<S1, S2>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor5DPermute02413ColumnMajor<T1, T2, T3>
  >;

  //
  // TTT (Row-major) BMMs
  //

  using TTTGemmBatchedPermuteA = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmBatchedPermuteAD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute
  >;

  using TTTGemmBatchedPermuteBD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteAB = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::NoPermute,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  using TTTGemmBatchedPermuteABD = GemmPermute<
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>,
    cutlass::layout::RowMajor, cutlass::layout::Tensor4DPermuteBMM0213RowMajor<D1>
  >;

  //
  // NNN (Col-major) BMMs
  //

  using NNNGemmBatchedPermuteA = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmBatchedPermuteAD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  using NNNGemmBatchedPermuteB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmBatchedPermuteBD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  using NNNGemmBatchedPermuteD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  using NNNGemmBatchedPermuteAB = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::NoPermute
  >;

  using NNNGemmBatchedPermuteABD = GemmPermute<
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>,
    cutlass::layout::ColumnMajor, cutlass::layout::Tensor4DPermuteBMM0321ColumnMajor<D1>
  >;

  //
  // Test Execution: Comprehensive Permutation Pattern Evaluation
  // ============================================================
  //

  std::cout << "CUTLASS GEMM Permutation Example" << std::endl;
  std::cout << "================================" << std::endl;
  std::cout << "Testing various tensor permutation patterns with GEMM operations." << std::endl;
  std::cout << "Device: " << props.name << " (Compute Capability " << props.major << "." << props.minor << ")" << std::endl;
  std::cout << std::endl;

  Testbed<ElementInput, ElementInput, ElementOutput> testbed(options);

  bool result = true;

  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<TTTGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<NNNGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<NNTGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteNone>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteA>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteAD>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteB>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteBD>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteD>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteAB>();
  result &= testbed.profile_GEMM_permute<TTNGemmNormalPermuteABD>();

  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteA>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteAD>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteB>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteBD>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteD>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteAB>();
  result &= testbed.profile_GEMM_permute<TTTGemmBatchedPermuteABD>();

  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteA>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteAD>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteB>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteBD>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteD>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteAB>();
  result &= testbed.profile_GEMM_permute<NNNGemmBatchedPermuteABD>();

  //
  // Final Results Summary
  // ====================
  //
  std::cout << "\n"
               "====================================================\n"
               "GEMM Permutation Example: " << (result ? "✓ ALL TESTS PASSED" : "✗ SOME TESTS FAILED") << "\n";
  if (result) {
    std::cout << "All permutation patterns executed successfully.\n"
                 "Performance results demonstrate efficient tensor\n"
                 "permutation fusion with GEMM operations.\n";
  } else {
    std::cout << "One or more permutation patterns failed validation.\n"
                 "Check tensor dimensions and alignment requirements.\n";
  }
  std::cout << "====================================================" << std::endl;

  return result ? EXIT_SUCCESS : EXIT_FAILURE;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
