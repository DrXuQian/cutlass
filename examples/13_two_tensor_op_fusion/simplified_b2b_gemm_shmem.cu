#include "hip/hip_runtime.h"
/*
 * Simplified B2B GEMM with Shared Memory Residency
 *
 * 这个文件演示了CUTLASS风格的B2B GEMM融合，使用共享内存存储中间结果。
 *
 * 关键特性：
 * 1. Shmem驻留（Shared Memory Residency）：中间结果C保存在共享内存中
 * 2. Device/Kernel分离架构：遵循CUTLASS的设计模式
 * 3. 协作式加载：线程块内的线程协作加载数据到共享内存
 * 4. Union共享内存：通过union节省共享内存使用
 *
 * 与RF版本的对比：
 * - RF版本：中间结果在寄存器，适合小Tile
 * - Shmem版本：中间结果在共享内存，支持更大的Tile
 *
 * 内存层次对比：
 * - 寄存器：<1 cycle，每线程255个32位寄存器
 * - 共享内存：~30 cycles，每SM 48-164KB
 * - 全局内存：~500 cycles，8-24GB
 *
 * SM80 FP16 only - 针对Ampere架构优化
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/epilogue/thread/linear_combination_relu.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"

///////////////////////////////////////////////////////////////////////////////
// Simplified B2B GEMM Kernel - Shared Memory Residency Version
//
// 核心概念：共享内存驻留（Shared Memory Residency）
// 中间结果C保存在共享内存中，不写入全局内存，避免了：
// 1. 一次全局内存写入（~500 cycles）
// 2. 一次全局内存读取（~500 cycles）
// 共享内存访问只需要~30 cycles，相比全局内存有巨大性能提升
//
// 共享内存的优势：
// - 比全局内存快约16倍
 // - 支持bank-conflict-free访问模式
// - 线程块内所有线程可共享数据
// - 支持原子操作和同步
//
// 共享内存的限制：
// - 容量有限（SM80: 最大164KB/SM）
// - 只在线程块内可见
// - 需要显式同步（__syncthreads）
///////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace gemm {
namespace kernel {

/**
 * @brief 简化的B2B GEMM kernel类（共享内存驻留版本）
 *
 * @tparam ThreadblockShape0_ 第一个GEMM的线程块Tile形状 [M, N, K]
 * @tparam ThreadblockShape1_ 第二个GEMM的线程块Tile形状 [M, N, K]
 * @tparam WarpShape0_ 第一个GEMM的Warp级Tile形状
 * @tparam WarpShape1_ 第二个GEMM的Warp级Tile形状
 * @tparam InstructionShape_ Tensor Core指令形状（如mma.sync）
 * @tparam EpilogueOutputOp0_ 第一个GEMM的epilogue操作（如ReLU）
 * @tparam EpilogueOutputOp1_ 第二个GEMM的epilogue操作
 */
template <
    typename ThreadblockShape0_,
    typename ThreadblockShape1_,
    typename WarpShape0_,
    typename WarpShape1_,
    typename InstructionShape_,
    typename EpilogueOutputOp0_,
    typename EpilogueOutputOp1_
>
class SimplifiedB2bGemmShmem {
public:
    using ThreadblockShape0 = ThreadblockShape0_;
    using ThreadblockShape1 = ThreadblockShape1_;
    using WarpShape0 = WarpShape0_;
    using WarpShape1 = WarpShape1_;
    using InstructionShape = InstructionShape_;
    using EpilogueOutputOp0 = EpilogueOutputOp0_;
    using EpilogueOutputOp1 = EpilogueOutputOp1_;

    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;

    /**
     * @brief Kernel参数结构体
     *
     * 包含所有kernel执行所需的参数：
     * - 问题尺寸：两个GEMM的维度
     * - 张量引用：指向设备内存的指针和stride信息
     * - Epilogue参数：alpha/beta缩放因子等
     */
    struct Params {
        cutlass::gemm::GemmCoord problem_size_0;  // 第一个GEMM: [M,N,K]
        cutlass::gemm::GemmCoord problem_size_1;  // 第二个GEMM: [M,P,N]
        cutlass::TensorRef<ElementA const, LayoutA> ref_A0;  // A矩阵引用
        cutlass::TensorRef<ElementB const, LayoutB> ref_B0;  // B0矩阵引用
        cutlass::TensorRef<ElementB const, LayoutB> ref_B1;  // B1矩阵引用
        cutlass::TensorRef<ElementC, LayoutC> ref_D1;        // 输出D矩阵引用
        typename EpilogueOutputOp0::Params epilogue0;  // 第一个epilogue参数
        typename EpilogueOutputOp1::Params epilogue1;  // 第二个epilogue参数
    };

    /**
     * @brief 共享内存结构体
     *
     * 使用union节省共享内存：
     * - 第一个GEMM时：存储A和B的Tile
     * - 第一个GEMM后：存储中间结果C和B1的Tile
     *
     * 这是关键优化：通过union复用内存空间，
     * 因为A/B Tiles和C/B1 Tiles不会同时使用。
     *
     * 内存布局优化：
     * - 避免bank conflict
     * - 支持向量化访问
     * - 内存对齐
     */
    union SharedStorage {
        struct {
            // 第一个GEMM期间：存储A和B的Tiles
            ElementA tile_A[ThreadblockShape0::kM][ThreadblockShape0::kK];  // A的Tile
            ElementB tile_B[ThreadblockShape0::kK][ThreadblockShape0::kN];  // B0的Tile
        } gemm1;

        struct {
            // 第二个GEMM期间：存储中间结果C和B1的Tile
            ElementC tile_C[ThreadblockShape0::kM][ThreadblockShape0::kN];   // 中间结果C（关键！）
            ElementB tile_B1[ThreadblockShape1::kK][ThreadblockShape1::kN];  // B1的Tile
        } intermediate;
    };

    /**
     * @brief Kernel主函数，执行B2B GEMM融合操作
     *
     * @param params 包含所有kernel参数的结构体
     * @param shared_storage 共享内存空间
     *
     * 执行流程：
     * 1. 第一个GEMM：C = A * B0
     *    - 协作加载A和B0的Tiles到共享内存
     *    - 执行矩阵乘法
     *    - 应用epilogue（如ReLU）
     * 2. 将C存储到共享内存（Shmem驻留）
     * 3. 第二个GEMM：D = C * B1
     *    - 从共享内存读取C
     *    - 协作加载B1的Tiles
     *    - 执行矩阵乘法
     * 4. 应用epilogue并写入全局内存
     */
    CUTLASS_DEVICE
    void operator()(Params const &params, SharedStorage &shared_storage) {
        // 线程和块标识
        // GPU执行模型：Grid -> Block -> Warp（32线程） -> Thread
        int thread_idx = threadIdx.x;      // 线程在block内的索引
        int warp_idx = thread_idx / 32;    // Warp索引（每个Warp 32个线程）
        int lane_idx = thread_idx % 32;    // 线程在Warp内的索引
        int block_idx_x = blockIdx.x;      // Block在Grid x维度的索引
        int block_idx_y = blockIdx.y;      // Block在Grid y维度的索引

        // 计算线程块级别的矩阵偏移
        // 每个线程块处理输出矩阵的一个Tile
        int block_m = block_idx_x * ThreadblockShape0::kM;  // M维度偏移
        int block_n = block_idx_y * ThreadblockShape0::kN;  // N维度偏移

        // ===== 第一个GEMM: C = A * B0 =====

        // 每线程累加器
        // 简化版本：每个线程计算2x2=4个输出元素
        // 实际CUTLASS会根据Warp形状和指令形状计算
        ElementAccumulator accumulator[4];  // 存储在寄存器中
        for (int i = 0; i < 4; ++i) {
            accumulator[i] = 0.0f;  // 初始化为0
        }

        // 第一个GEMM的主循环：沿K维度分块
        for (int k_tile = 0; k_tile < params.problem_size_0.k(); k_tile += ThreadblockShape0::kK) {

            // 协作加载A的Tile到共享内存
            // 所有线程协同工作，将全局内存数据加载到共享内存
            // 这是CUTLASS的核心优化：通过共享内存减少全局内存访问
            __syncthreads();  // 同步确保之前的共享内存操作完成

            // 线程协作模式：每个线程加载多个元素
            for (int i = thread_idx; i < ThreadblockShape0::kM * ThreadblockShape0::kK;
                 i += blockDim.x) {  // 步长为线程块大小
                // 计算在Tile内的位置
                int row = i / ThreadblockShape0::kK;  // Tile内的行
                int col = i % ThreadblockShape0::kK;  // Tile内的列

                // 计算全局矩阵中的位置
                int global_row = block_m + row;  // 全局行索引
                int global_col = k_tile + col;   // 全局列索引

                // 边界检查并加载数据
                if (global_row < params.problem_size_0.m() && global_col < params.problem_size_0.k()) {
                    // 从全局内存加载到共享内存
                    shared_storage.gemm1.tile_A[row][col] =
                        params.ref_A0.at({global_row, global_col});
                } else {
                    // 越界位置填充0（padding）
                    shared_storage.gemm1.tile_A[row][col] = ElementA(0);
                }
            }

            // 协作加载B0的Tile到共享内存
            // 与加载A类似，所有线程协同工作
            for (int i = thread_idx; i < ThreadblockShape0::kK * ThreadblockShape0::kN;
                 i += blockDim.x) {
                int row = i / ThreadblockShape0::kN;
                int col = i % ThreadblockShape0::kN;
                int global_row = k_tile + row;
                int global_col = block_n + col;

                if (global_row < params.problem_size_0.k() && global_col < params.problem_size_0.n()) {
                    shared_storage.gemm1.tile_B[row][col] =
                        params.ref_B0.at({global_row, global_col});
                } else {
                    shared_storage.gemm1.tile_B[row][col] = ElementB(0);
                }
            }

            __syncthreads();  // 确保所有数据加载完成后再计算

            // 执行矩阵乘法计算
            // 线程到输出的映射：每个线程计算2x2的输出块
            // 这是简化的映射，实际CUTLASS使用更复杂的映射策略
            int thread_row = (thread_idx / 8) * 4;  // 该线程负责的起始行
            int thread_col = (thread_idx % 8) * 4;  // 该线程负责的起始列

            if (thread_row < ThreadblockShape0::kM && thread_col < ThreadblockShape0::kN) {
                // 内层K循环：执行点积运算
                for (int k = 0; k < ThreadblockShape0::kK; ++k) {
                    // 从共享内存读取A和B的元素
                    // 共享内存访问比全局内存快约16倍
                    float a_val = float(shared_storage.gemm1.tile_A[thread_row][k]);
                    float b_val = float(shared_storage.gemm1.tile_B[k][thread_col]);

                    // 累加到寄存器（最快的存储）
                    accumulator[0] += a_val * b_val;  // [0,0]位置

                    // 计算2x2块的其他元素
                    // 这提高了指令级并行性（ILP）
                    if (thread_row + 1 < ThreadblockShape0::kM) {
                        float a_val_1 = float(shared_storage.gemm1.tile_A[thread_row + 1][k]);
                        accumulator[1] += a_val_1 * b_val;  // [1,0]位置
                    }
                    if (thread_col + 1 < ThreadblockShape0::kN) {
                        float b_val_1 = float(shared_storage.gemm1.tile_B[k][thread_col + 1]);
                        accumulator[2] += a_val * b_val_1;
                    }
                    if (thread_row + 1 < ThreadblockShape0::kM && thread_col + 1 < ThreadblockShape0::kN) {
                        float a_val_1 = float(shared_storage.gemm1.tile_A[thread_row + 1][k]);
                        float b_val_1 = float(shared_storage.gemm1.tile_B[k][thread_col + 1]);
                        accumulator[3] += a_val_1 * b_val_1;
                    }
                }
            }
        }

        // ========== 存储中间结果C到共享内存 ==========
        // 这是Shmem驻留的核心：C保持在共享内存中，不写入全局内存
        __syncthreads();  // 确保所有线程完成第一个GEMM

        // 应用第一个GEMM的epilogue操作（如ReLU）
        int thread_row = (thread_idx / 8) * 4;
        int thread_col = (thread_idx % 8) * 4;

        if (thread_row < ThreadblockShape0::kM && thread_col < ThreadblockShape0::kN) {
            // 应用epilogue（如ReLU: max(0, x)）并存储到共享内存
            // 关键：结果存储在共享内存，而不是全局内存！
            float result = params.epilogue0(accumulator[0]);
            shared_storage.intermediate.tile_C[thread_row][thread_col] = ElementC(result);

            if (thread_row + 1 < ThreadblockShape0::kM) {
                result = params.epilogue0(accumulator[1]);
                shared_storage.intermediate.tile_C[thread_row + 1][thread_col] = ElementC(result);
            }
            if (thread_col + 1 < ThreadblockShape0::kN) {
                result = params.epilogue0(accumulator[2]);
                shared_storage.intermediate.tile_C[thread_row][thread_col + 1] = ElementC(result);
            }
            if (thread_row + 1 < ThreadblockShape0::kM && thread_col + 1 < ThreadblockShape0::kN) {
                result = params.epilogue0(accumulator[3]);
                shared_storage.intermediate.tile_C[thread_row + 1][thread_col + 1] = ElementC(result);
            }
        }

        __syncthreads();  // 确保C完全写入共享内存

        // ===== 第二个GEMM: D = C * B1 =====
        // 关键优化：C现在在共享内存中，无需从全局内存读取！
        // 这避免了~500 cycles的全局内存访问延迟

        // 计算P维度的块偏移（第二个GEMM输出的列维度）
        int block_p = block_idx_y * ThreadblockShape1::kN;

        // 重置累加器用于第二个GEMM
        for (int i = 0; i < 4; ++i) {
            accumulator[i] = 0.0f;
        }

        // 第二个GEMM的主循环：沿N维度分块
        // N是第一个GEMM的输出列，第二个GEMM的K维度
        for (int n_tile = 0; n_tile < params.problem_size_0.n(); n_tile += ThreadblockShape1::kK) {

            // 协作加载B1的Tile到共享内存
            __syncthreads();
            for (int i = thread_idx; i < ThreadblockShape1::kK * ThreadblockShape1::kN;
                 i += blockDim.x) {
                int row = i / ThreadblockShape1::kN;
                int col = i % ThreadblockShape1::kN;
                int global_row = n_tile + row;
                int global_col = block_p + col;

                if (global_row < params.problem_size_0.n() && global_col < params.problem_size_1.n()) {
                    shared_storage.intermediate.tile_B1[row][col] =
                        params.ref_B1.at({global_row, global_col});
                } else {
                    shared_storage.intermediate.tile_B1[row][col] = ElementB(0);
                }
            }

            __syncthreads();  // 确保B1加载完成

            // 使用共享内存中的C进行计算
            // 这是关键：C从共享内存读取，而不是全局内存
            thread_row = (thread_idx / 8) * 4;
            int thread_p = (thread_idx % 8) * 4;  // P维度的位置

            if (thread_row < ThreadblockShape0::kM && thread_p < ThreadblockShape1::kN) {
                // 第二个GEMM的矩阵乘法
                // 维度匹配：C是[M x N]，B1是[N x P]，输出D是[M x P]
                for (int n = 0; n < min(ThreadblockShape1::kK, (int)ThreadblockShape0::kN); ++n) {
                    if (n_tile + n < params.problem_size_0.n()) {
                        // 关键：从共享内存读取C（而不是全局内存）
                        // 这是Shmem驻留的核心优势
                        float c_val = float(shared_storage.intermediate.tile_C[thread_row][n]);

                        // 从共享内存读取B1
                        if (thread_p < ThreadblockShape1::kN && n < ThreadblockShape1::kK) {
                            float b1_val = float(shared_storage.intermediate.tile_B1[n][thread_p]);

                            // 累加：D[m,p] += C[m,n] * B1[n,p]
                            accumulator[0] += c_val * b1_val;
                        }
                    }
                }
            }
        }

        // ========== 存储最终结果到全局内存 ==========
        // 这是整个B2B GEMM中唯一的全局内存写操作

        // 应用第二个epilogue并计算全局位置
        thread_row = block_m + (thread_idx / 8) * 4;  // 全局M位置
        int thread_p = block_p + (thread_idx % 8) * 4;  // 全局P位置

        // 边界检查后写入全局内存
        if (thread_row < params.problem_size_1.m() && thread_p < params.problem_size_1.n()) {
            // 应用epilogue（线性组合等）
            float result = params.epilogue1(accumulator[0]);

            // 写入全局内存（唯一的全局内存写操作）
            params.ref_D1.at({thread_row, thread_p}) = ElementC(result);
        }
    }
};

} // namespace kernel
} // namespace gemm
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////
// Simplified Device-level B2B GEMM
//
// Device层是CUTLASS的API层，负责：
// 1. 参数打包和验证
// 2. Kernel启动配置计算
// 3. 共享内存大小计算
// 4. 错误处理
//
// 设计模式：Device类封装Kernel类，提供高层接口
///////////////////////////////////////////////////////////////////////////////

/**
 * @brief Device级别的B2B GEMM类（共享内存版本）
 *
 * 提供用户友好的接口，隐藏kernel细节。
 * 管理kernel参数和启动配置。
 *
 * @tparam ThreadblockShape0 第一个GEMM的线程块形状
 * @tparam ThreadblockShape1 第二个GEMM的线程块形状
 * @tparam WarpShape0 第一个GEMM的Warp形状
 * @tparam WarpShape1 第二个GEMM的Warp形状
 * @tparam InstructionShape Tensor Core指令形状
 * @tparam EpilogueOutputOp0 第一个GEMM的epilogue操作
 * @tparam EpilogueOutputOp1 第二个GEMM的epilogue操作
 */
template <
    typename ThreadblockShape0,
    typename ThreadblockShape1,
    typename WarpShape0,
    typename WarpShape1,
    typename InstructionShape,
    typename EpilogueOutputOp0,
    typename EpilogueOutputOp1
>
class SimplifiedB2bGemmDevice {
public:
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;

    using B2bGemmKernel = typename cutlass::gemm::kernel::SimplifiedB2bGemmShmem<
        ThreadblockShape0,
        ThreadblockShape1,
        WarpShape0,
        WarpShape1,
        InstructionShape,
        EpilogueOutputOp0,
        EpilogueOutputOp1
    >;

    /**
     * @brief 用户参数结构体
     *
     * 这是用户接口，包含所有B2B GEMM需要的参数。
     * Device类将其转换为Kernel::Params格式。
     */
    struct Arguments {
        cutlass::gemm::GemmCoord problem_size_0;  // 第一个GEMM尺寸 [M,N,K]
        cutlass::gemm::GemmCoord problem_size_1;  // 第二个GEMM尺寸 [M,P,N]
        cutlass::TensorRef<ElementA const, LayoutA> ref_A0;   // 输入A
        cutlass::TensorRef<ElementB const, LayoutB> ref_B0;   // 输入B0
        cutlass::TensorRef<ElementB const, LayoutB> ref_B1;   // 输入B1
        cutlass::TensorRef<ElementC, LayoutC> ref_D1;         // 输出D
        typename EpilogueOutputOp0::Params epilogue0;  // epilogue参数1
        typename EpilogueOutputOp1::Params epilogue1;  // epilogue参数2

        Arguments(
            cutlass::gemm::GemmCoord problem_size_0_,
            cutlass::gemm::GemmCoord problem_size_1_,
            cutlass::TensorRef<ElementA const, LayoutA> ref_A0_,
            cutlass::TensorRef<ElementB const, LayoutB> ref_B0_,
            cutlass::TensorRef<ElementB const, LayoutB> ref_B1_,
            cutlass::TensorRef<ElementC, LayoutC> ref_D1_,
            float alpha0 = 1.0f,
            float beta0 = 0.0f,
            float alpha1 = 1.0f,
            float beta1 = 0.0f
        ):
            problem_size_0(problem_size_0_),
            problem_size_1(problem_size_1_),
            ref_A0(ref_A0_),
            ref_B0(ref_B0_),
            ref_B1(ref_B1_),
            ref_D1(ref_D1_),
            epilogue0({alpha0, beta0}),
            epilogue1({alpha1, beta1})
        {}
    };

private:
    typename B2bGemmKernel::Params params_;

public:
    /**
     * @brief 初始化函数
     *
     * 将用户参数转换为kernel参数。
     * 实际CUTLASS中还会进行：
     * - 参数验证（尺寸、对齐等）
     * - 优化配置选择
     * - 内存布局转换
     *
     * @param args 用户提供的参数
     * @return 状态码（成功/失败）
     */
    cutlass::Status initialize(Arguments const &args) {
        // 构造kernel参数
        params_ = typename B2bGemmKernel::Params{
            args.problem_size_0,
            args.problem_size_1,
            args.ref_A0,
            args.ref_B0,
            args.ref_B1,
            args.ref_D1,
            args.epilogue0,
            args.epilogue1
        };
        return cutlass::Status::kSuccess;
    }

    /**
     * @brief 执行B2B GEMM
     *
     * 计算启动配置并执行kernel。
     *
     * @param stream CUDA流（可选）
     * @return 执行状态
     */
    cutlass::Status run(hipStream_t stream = nullptr) {
        // 计算Grid维度（线程块数量）
        // Grid覆盖整个输出矩阵，每个Block处理一个Tile
        dim3 grid(
            (params_.problem_size_0.m() + ThreadblockShape0::kM - 1) / ThreadblockShape0::kM,  // M方向块数
            (params_.problem_size_1.n() + ThreadblockShape1::kN - 1) / ThreadblockShape1::kN   // P方向块数
        );

        // Block维度（线程数）
        // 128线程 = 4个Warp，这是常见配置
        dim3 block(128);  // 4 warps * 32 threads/warp

        // 计算共享内存大小
        // 共享内存用于：
        // 1. 第一个GEMM：存储A和B的Tiles
        // 2. 中间阶段：存储C的结果（关键！）
        // 3. 第二个GEMM：存储B1的Tiles
        // Union结构使得这些存储可以复用空间
        int smem_size = sizeof(typename B2bGemmKernel::SharedStorage);

        // 共享内存配置
        // 实际实现中需要：
        // 1. hipFuncSetAttribute设置最大共享内存
        // 2. hipFuncSetCacheConfig配置L1/共享内存比例
        // 3. 检查共享内存是否足够

        // Kernel启动（简化版本）
        // 实际CUTLASS使用复杂的启动机制：
        // - cutlass::Kernel类封装
        // - 动态共享内存配置
        // - Occupancy优化

        // Note: Direct kernel launch commented out due to template complexity
        // The kernel would be launched here in production code
        // For demonstration, showing the structure only

        std::cout << "Note: Kernel launch simplified for demonstration\n";
        std::cout << "Grid: (" << grid.x << ", " << grid.y << "), Block: " << block.x << "\n";
        std::cout << "Shared memory: " << smem_size << " bytes\n";
        std::cout << "Key optimization: Intermediate C stays in shared memory ("
                  << sizeof(ElementC) * ThreadblockShape0::kM * ThreadblockShape0::kN
                  << " bytes for C)\n";
        std::cout << "This shows CUTLASS B2B GEMM structure with Shmem residency\n";

        return cutlass::Status::kSuccess;
    }
};

///////////////////////////////////////////////////////////////////////////////
// Test harness - 测试框架
//
// 演示如何使用简化的B2B GEMM（共享内存版本）：
// 1. 定义配置（Tile大小、数据类型等）
// 2. 分配和初始化数据
// 3. 创建和执行B2B GEMM
// 4. 验证结果
//
// 注意：共享内存版本使用较小的Tile以适应共享内存限制
///////////////////////////////////////////////////////////////////////////////

/**
 * @brief 主函数 - B2B GEMM测试入口（共享内存版本）
 *
 * 演示完整的使用流程：
 * 1. 配置问题尺寸（注意：比RF版本小，因为共享内存有限）
 * 2. 定义GEMM配置（Tile大小、epilogue等）
 * 3. 分配内存
 * 4. 初始化数据
 * 5. 执行GPU计算
 * 6. 计算CPU参考结果
 * 7. 验证正确性
 */
int main() {
    std::cout << "\n=== Simplified B2B GEMM with Shared Memory Residency (CUTLASS-style) ===\n";

    // 定义问题尺寸
    // 注意：相比RF版本，尺寸更小，因为共享内存容量有限
    // SM80共享内存：最大164KB/SM，但通常配置为48KB
    int M = 64;  // 矩阵A的行数，也是最终输出的行数
    int N = 64;  // 中间矩阵C的列数
    int K = 64;  // 矩阵A的列数，B0的行数
    int P = 32;  // 最终输出D的列数

    cutlass::gemm::GemmCoord problem_size_0(M, N, K);
    cutlass::gemm::GemmCoord problem_size_1(M, P, N);

    std::cout << "First GEMM:  [" << M << "," << K << "] x [" << K << "," << N << "] = [" << M << "," << N << "]\n";
    std::cout << "Second GEMM: [" << M << "," << N << "] x [" << N << "," << P << "] = [" << M << "," << P << "]\n\n";

    // Define types
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;

    // 定义Tile尺寸配置
    // 关键：Tile尺寸必须适应共享内存限制

    // 线程块Tile：每个线程块处理的输出大小
    // 32x32x16比RF版本的64x64x32小，以适应共享内存
    using ThreadblockShape0 = cutlass::gemm::GemmShape<32, 32, 16>;  // [M=32, N=32, K=16]
    using ThreadblockShape1 = cutlass::gemm::GemmShape<32, 32, 16>;  // [M=32, N=32, K=16]

    // Warp Tile：每个Warp处理的输出大小
    using WarpShape0 = cutlass::gemm::GemmShape<16, 16, 16>;  // [M=16, N=16, K=16]
    using WarpShape1 = cutlass::gemm::GemmShape<16, 16, 16>;  // [M=16, N=16, K=16]

    // 指令形状：适配较小的Tensor Core操作
    using InstructionShape = cutlass::gemm::GemmShape<8, 8, 4>;  // [M=8, N=8, K=4]

    // 定义Epilogue操作
    // 与RF版本相同的epilogue配置

    // 第一个GEMM的epilogue：线性组合 + ReLU激活
    using EpilogueOutputOp0 = cutlass::epilogue::thread::LinearCombinationRelu<
        ElementC, 1, ElementAccumulator, float
    >;

    // 第二个GEMM的epilogue：仅线性组合
    using EpilogueOutputOp1 = cutlass::epilogue::thread::LinearCombination<
        ElementC, 1, ElementAccumulator, float
    >;

    // Allocate tensors
    cutlass::HostTensor<ElementA, LayoutA> tensor_A0(problem_size_0.mk());
    cutlass::HostTensor<ElementB, LayoutB> tensor_B0(problem_size_0.kn());
    cutlass::HostTensor<ElementB, LayoutB> tensor_B1(problem_size_1.kn());
    cutlass::HostTensor<ElementC, LayoutC> tensor_D1(problem_size_1.mn());
    cutlass::HostTensor<ElementC, LayoutC> tensor_D1_ref(problem_size_1.mn());

    // 初始化张量数据
    // 使用[-0.5, 0.5]范围的随机数，避免FP16溢出
    // 较小的值范围有助于数值稳定性
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_A0.host_view(), 1, ElementA(0.5), ElementA(-0.5), 0);   // seed=0
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_B0.host_view(), 1, ElementB(0.5), ElementB(-0.5), 1);   // seed=1
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_B1.host_view(), 1, ElementB(0.5), ElementB(-0.5), 2);   // seed=2

    // 输出初始化为0
    cutlass::reference::host::TensorFill(
        tensor_D1.host_view(), ElementC(0));

    // Copy to device
    tensor_A0.sync_device();
    tensor_B0.sync_device();
    tensor_B1.sync_device();
    tensor_D1.sync_device();

    // Create B2B GEMM instance
    SimplifiedB2bGemmDevice<
        ThreadblockShape0, ThreadblockShape1,
        WarpShape0, WarpShape1,
        InstructionShape,
        EpilogueOutputOp0, EpilogueOutputOp1
    > b2b_gemm_op;

    // Setup arguments
    typename decltype(b2b_gemm_op)::Arguments args(
        problem_size_0,
        problem_size_1,
        tensor_A0.device_ref(),
        tensor_B0.device_ref(),
        tensor_B1.device_ref(),
        tensor_D1.device_ref(),
        1.0f, 0.0f,  // alpha0, beta0
        1.0f, 0.0f   // alpha1, beta1
    );

    // Initialize
    cutlass::Status status = b2b_gemm_op.initialize(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to initialize\n";
        return -1;
    }

    // 执行B2B GEMM kernel
    std::cout << "Running fused B2B GEMM with shared memory residency...\n";

    // 显示共享内存使用情况
    // 这是关键信息，显示中间结果C确实存储在共享内存中
    std::cout << "Total shared memory size: "
              << sizeof(typename decltype(b2b_gemm_op)::B2bGemmKernel::SharedStorage) << " bytes\n";
    std::cout << "Memory for intermediate C: "
              << sizeof(ElementC) * 32 * 32 << " bytes (in shared memory!)\n";

    status = b2b_gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Kernel failed\n";
        return -1;
    }

    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "\n";
        return -1;
    }

    // Copy result back
    tensor_D1.sync_host();

    // Compute reference on CPU
    std::cout << "Computing reference on CPU...\n";
    cutlass::HostTensor<ElementC, LayoutC> tensor_C0_ref(problem_size_0.mn());

    // Reference GEMM 1
    cutlass::reference::host::Gemm<
        ElementA, LayoutA,
        ElementB, LayoutB,
        ElementC, LayoutC,
        ElementAccumulator, ElementAccumulator
    > reference_gemm;

    reference_gemm(
        problem_size_0,
        ElementAccumulator(1),
        tensor_A0.host_view(),
        tensor_B0.host_view(),
        ElementAccumulator(0),
        tensor_C0_ref.host_view()
    );

    // Apply ReLU to reference
    for (int i = 0; i < problem_size_0.m() * problem_size_0.n(); ++i) {
        tensor_C0_ref.host_data()[i] = ElementC(fmaxf(0.0f, float(tensor_C0_ref.host_data()[i])));
    }

    // Reference GEMM 2
    reference_gemm(
        problem_size_1,
        ElementAccumulator(1),
        tensor_C0_ref.host_view(),
        tensor_B1.host_view(),
        ElementAccumulator(0),
        tensor_D1_ref.host_view()
    );

    // Compare results
    bool passed = cutlass::reference::host::TensorEquals(
        tensor_D1.host_view(),
        tensor_D1_ref.host_view()
    );

    if (passed) {
        std::cout << "\n*** PASSED ***\n";
    } else {
        std::cout << "\n*** FAILED ***\n";

        // 调试输出：打印前几个元素用于比较
        std::cout << "\nFirst 4x4 elements of output:\n";
        std::cout << "GPU result (with Shmem residency):\n";
        for (int i = 0; i < std::min(4, M); ++i) {
            for (int j = 0; j < std::min(4, P); ++j) {
                std::cout << float(tensor_D1.at({i, j})) << " ";
            }
            std::cout << "\n";
        }
        std::cout << "\nCPU reference:\n";
        for (int i = 0; i < std::min(4, M); ++i) {
            for (int j = 0; j < std::min(4, P); ++j) {
                std::cout << float(tensor_D1_ref.at({i, j})) << " ";
            }
            std::cout << "\n";
        }
    }

    return passed ? 0 : -1;
}