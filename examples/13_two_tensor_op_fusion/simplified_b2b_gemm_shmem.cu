#include "hip/hip_runtime.h"
/*
 * Simplified B2B GEMM with Shared Memory Residency
 * This is a simplified version for understanding the core concepts
 * Only supports SM80 FP16
 */

#include <iostream>
#include <hip/hip_fp16.h>
#include <mma.h>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gemm.h"

using namespace nvcuda;

// Shared memory structure for intermediate results
template<int TILE_M, int TILE_N>
struct SharedMemory {
    half intermediate[TILE_M][TILE_N];  // Store intermediate C matrix
    half tileA[TILE_M][TILE_N];         // Tile for matrix A
    half tileB[TILE_N][TILE_N];         // Tile for matrix B
};

// Simple B2B GEMM kernel with Shared Memory residency
template<int TILE_M, int TILE_N, int TILE_K>
__global__ void b2b_gemm_shmem_kernel(
    half const* __restrict__ A,    // M x K
    half const* __restrict__ B0,   // K x N
    half const* __restrict__ B1,   // N x P
    half* __restrict__ D,          // M x P
    int M, int N, int K, int P
) {
    // Allocate shared memory
    __shared__ SharedMemory<TILE_M, TILE_N> smem;

    // Thread and block indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Global row and column for this thread
    int row = bx * TILE_M + ty;
    int col = by * TILE_N + tx;

    // ========== First GEMM: C = A * B0 ==========
    float accumulator = 0.0f;

    // Loop over tiles in K dimension
    for (int tileIdx = 0; tileIdx < (K + TILE_K - 1) / TILE_K; ++tileIdx) {
        // Collaborative loading of A tile into shared memory
        int aRow = bx * TILE_M + ty;
        int aCol = tileIdx * TILE_K + tx;
        if (aRow < M && aCol < K && ty < TILE_M && tx < TILE_K) {
            smem.tileA[ty][tx] = A[aRow * K + aCol];
        } else {
            smem.tileA[ty][tx] = __float2half(0.0f);
        }

        // Collaborative loading of B0 tile into shared memory
        int bRow = tileIdx * TILE_K + ty;
        int bCol = by * TILE_N + tx;
        if (bRow < K && bCol < N && ty < TILE_K && tx < TILE_N) {
            // Note: B0 is in column major, so we transpose during load
            smem.tileB[ty][tx] = B0[bRow + bCol * K];
        } else {
            smem.tileB[ty][tx] = __float2half(0.0f);
        }

        __syncthreads();

        // Compute partial dot product
        if (ty < TILE_M && tx < TILE_N) {
            for (int k = 0; k < TILE_K && k < (K - tileIdx * TILE_K); ++k) {
                accumulator += __half2float(smem.tileA[ty][k]) *
                              __half2float(smem.tileB[k][tx]);
            }
        }

        __syncthreads();
    }

    // ========== Store intermediate result C in shared memory ==========
    if (row < M && col < N && ty < TILE_M && tx < TILE_N) {
        smem.intermediate[ty][tx] = __float2half(accumulator);
    }
    __syncthreads();

    // ========== Second GEMM: D = C * B1 ==========
    // Now smem.intermediate contains the result of first GEMM
    // Use it for second GEMM

    // Reset accumulator for second GEMM
    accumulator = 0.0f;

    // Different column index for second GEMM output
    int colP = by * TILE_N + tx;  // Column in P dimension

    if (colP < P) {
        // Loop over tiles in N dimension
        for (int tileIdx = 0; tileIdx < (N + TILE_K - 1) / TILE_K; ++tileIdx) {
            // Load C tile from shared memory (already there for first tile)
            // For subsequent tiles, we would need to load from global memory
            // This is simplified - showing only the first tile

            if (tileIdx == 0) {
                // C tile is already in smem.intermediate
                // Just need to load corresponding B1 tile
            } else {
                // In full implementation, load next C tile
                // For simplification, we skip this
                break;
            }

            // Load B1 tile into shared memory
            int b1Row = tileIdx * TILE_K + ty;
            int b1Col = colP;
            if (b1Row < N && b1Col < P && ty < TILE_K && tx < 1) {
                smem.tileB[ty][0] = B1[b1Row + b1Col * N];
            } else if (ty < TILE_K) {
                smem.tileB[ty][0] = __float2half(0.0f);
            }

            __syncthreads();

            // Compute partial dot product for second GEMM
            if (ty < TILE_M && tx < 1 && row < M && colP < P) {
                for (int n = 0; n < TILE_N && n < N; ++n) {
                    accumulator += __half2float(smem.intermediate[ty][n]) *
                                  __half2float(smem.tileB[n][0]);
                }
            }

            __syncthreads();
        }
    }

    // Store final result D
    if (row < M && colP < P && ty < TILE_M && tx < 1) {
        D[row * P + colP] = __float2half(accumulator);
    }
}

// Full implementation using CUTLASS-style shared memory B2B GEMM
template<int BLOCK_M, int BLOCK_N, int BLOCK_K>
__global__ void b2b_gemm_shmem_full_kernel(
    half const* __restrict__ A,
    half const* __restrict__ B0,
    half const* __restrict__ B1,
    half* __restrict__ D,
    int M, int N, int K, int P
) {
    // Shared memory for tiles and intermediate results
    extern __shared__ half shared_mem[];

    half* shmem_A = shared_mem;
    half* shmem_B = shmem_A + BLOCK_M * BLOCK_K;
    half* shmem_C = shmem_B + BLOCK_K * BLOCK_N;

    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int block_row = blockIdx.x;
    int block_col = blockIdx.y;

    // ========== First GEMM: C = A * B0 ==========

    // Initialize C tile in shared memory
    for (int i = tid; i < BLOCK_M * BLOCK_N; i += blockDim.x * blockDim.y) {
        shmem_C[i] = __float2half(0.0f);
    }
    __syncthreads();

    // Loop over K dimension
    for (int k_tile = 0; k_tile < (K + BLOCK_K - 1) / BLOCK_K; ++k_tile) {
        // Load A tile collaboratively
        for (int i = tid; i < BLOCK_M * BLOCK_K; i += blockDim.x * blockDim.y) {
            int local_row = i / BLOCK_K;
            int local_col = i % BLOCK_K;
            int global_row = block_row * BLOCK_M + local_row;
            int global_col = k_tile * BLOCK_K + local_col;

            if (global_row < M && global_col < K) {
                shmem_A[i] = A[global_row * K + global_col];
            } else {
                shmem_A[i] = __float2half(0.0f);
            }
        }

        // Load B0 tile collaboratively
        for (int i = tid; i < BLOCK_K * BLOCK_N; i += blockDim.x * blockDim.y) {
            int local_row = i / BLOCK_N;
            int local_col = i % BLOCK_N;
            int global_row = k_tile * BLOCK_K + local_row;
            int global_col = block_col * BLOCK_N + local_col;

            if (global_row < K && global_col < N) {
                shmem_B[i] = B0[global_row + global_col * K];
            } else {
                shmem_B[i] = __float2half(0.0f);
            }
        }

        __syncthreads();

        // Compute tile matrix multiply
        for (int i = tid; i < BLOCK_M * BLOCK_N; i += blockDim.x * blockDim.y) {
            int tile_row = i / BLOCK_N;
            int tile_col = i % BLOCK_N;

            float sum = __half2float(shmem_C[i]);
            for (int k = 0; k < BLOCK_K; ++k) {
                sum += __half2float(shmem_A[tile_row * BLOCK_K + k]) *
                       __half2float(shmem_B[k * BLOCK_N + tile_col]);
            }
            shmem_C[i] = __float2half(sum);
        }

        __syncthreads();
    }

    // ========== Shared Memory Residency ==========
    // shmem_C now contains intermediate result C
    // Keep it in shared memory for second GEMM

    // ========== Second GEMM: D = C * B1 ==========

    // Reuse shmem_A for output D
    half* shmem_D = shmem_A;

    // Initialize D tile
    for (int i = tid; i < BLOCK_M * BLOCK_N; i += blockDim.x * blockDim.y) {
        shmem_D[i] = __float2half(0.0f);
    }
    __syncthreads();

    // For second GEMM, block_col now indexes into P dimension
    if (block_col * BLOCK_N < P) {
        // Loop over N dimension
        for (int n_tile = 0; n_tile < (N + BLOCK_K - 1) / BLOCK_K; ++n_tile) {
            // Load B1 tile
            for (int i = tid; i < BLOCK_K * BLOCK_N; i += blockDim.x * blockDim.y) {
                int local_row = i / BLOCK_N;
                int local_col = i % BLOCK_N;
                int global_row = n_tile * BLOCK_K + local_row;
                int global_col = block_col * BLOCK_N + local_col;

                if (global_row < N && global_col < P) {
                    shmem_B[i] = B1[global_row + global_col * N];
                } else {
                    shmem_B[i] = __float2half(0.0f);
                }
            }

            __syncthreads();

            // Matrix multiply for second GEMM
            // Note: C is BLOCK_M x N, we need to use appropriate tiles
            for (int i = tid; i < BLOCK_M * BLOCK_N; i += blockDim.x * blockDim.y) {
                int tile_row = i / BLOCK_N;
                int tile_col = i % BLOCK_N;

                if (block_col * BLOCK_N + tile_col < P) {
                    float sum = __half2float(shmem_D[i]);

                    // Use only the first BLOCK_K columns of C for this simplified version
                    int k_limit = min(BLOCK_K, N - n_tile * BLOCK_K);
                    for (int k = 0; k < k_limit; ++k) {
                        if (n_tile == 0) {  // Simplified: only use first tile of C
                            sum += __half2float(shmem_C[tile_row * BLOCK_N + k]) *
                                   __half2float(shmem_B[k * BLOCK_N + tile_col]);
                        }
                    }
                    shmem_D[i] = __float2half(sum);
                }
            }

            __syncthreads();
        }

        // Store D tile to global memory
        for (int i = tid; i < BLOCK_M * BLOCK_N; i += blockDim.x * blockDim.y) {
            int tile_row = i / BLOCK_N;
            int tile_col = i % BLOCK_N;
            int global_row = block_row * BLOCK_M + tile_row;
            int global_col = block_col * BLOCK_N + tile_col;

            if (global_row < M && global_col < P) {
                D[global_row * P + global_col] = shmem_D[i];
            }
        }
    }
}

// Simplified launcher for B2B GEMM with Shared Memory residency
class SimplifiedB2bGemmShmem {
public:
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementD = cutlass::half_t;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;
    using LayoutD = cutlass::layout::RowMajor;

    bool run(int M, int N, int K, int P) {
        std::cout << "\n=== Simplified B2B GEMM with Shared Memory Residency ===\n";
        std::cout << "Problem: [" << M << "," << K << "] * [" << K << "," << N << "] = ["
                  << M << "," << N << "]\n";
        std::cout << "        [" << M << "," << N << "] * [" << N << "," << P << "] = ["
                  << M << "," << P << "]\n\n";

        // Allocate host tensors
        cutlass::HostTensor<ElementA, LayoutA> tensor_A({M, K});
        cutlass::HostTensor<ElementB, LayoutB> tensor_B0({K, N});
        cutlass::HostTensor<ElementB, LayoutB> tensor_B1({N, P});
        cutlass::HostTensor<ElementD, LayoutD> tensor_D({M, P});
        cutlass::HostTensor<ElementD, LayoutD> tensor_D_ref({M, P});

        // Initialize tensors
        cutlass::reference::host::TensorFillRandomUniform(
            tensor_A.host_view(), 1, ElementA(1), ElementA(-1), 0);
        cutlass::reference::host::TensorFillRandomUniform(
            tensor_B0.host_view(), 1, ElementB(1), ElementB(-1), 1);
        cutlass::reference::host::TensorFillRandomUniform(
            tensor_B1.host_view(), 1, ElementB(1), ElementB(-1), 2);
        cutlass::reference::host::TensorFill(
            tensor_D.host_view(), ElementD(0));

        // Copy to device
        tensor_A.sync_device();
        tensor_B0.sync_device();
        tensor_B1.sync_device();
        tensor_D.sync_device();

        // Launch configuration
        const int BLOCK_M = 32;
        const int BLOCK_N = 32;
        const int BLOCK_K = 16;

        dim3 gridDim(
            (M + BLOCK_M - 1) / BLOCK_M,
            (std::max(N, P) + BLOCK_N - 1) / BLOCK_N
        );
        dim3 blockDim(16, 16);  // 256 threads per block

        // Calculate shared memory size
        size_t shmem_size = sizeof(half) * (
            BLOCK_M * BLOCK_K +     // A tile
            BLOCK_K * BLOCK_N +     // B tile
            BLOCK_M * BLOCK_N       // C/D tile
        );

        std::cout << "Launching kernel with:\n";
        std::cout << "  Grid: (" << gridDim.x << ", " << gridDim.y << ")\n";
        std::cout << "  Block: (" << blockDim.x << ", " << blockDim.y << ")\n";
        std::cout << "  Shared memory: " << shmem_size << " bytes\n";

        // Set shared memory config
        hipFuncSetAttribute(reinterpret_cast<const void*>(
            b2b_gemm_shmem_full_kernel<BLOCK_M), BLOCK_N, BLOCK_K>,
            hipFuncAttributeMaxDynamicSharedMemorySize,
            shmem_size
        );

        // Launch kernel
        b2b_gemm_shmem_full_kernel<BLOCK_M, BLOCK_N, BLOCK_K>
            <<<gridDim, blockDim, shmem_size>>>(
            (half const*)tensor_A.device_data(),
            (half const*)tensor_B0.device_data(),
            (half const*)tensor_B1.device_data(),
            (half*)tensor_D.device_data(),
            M, N, K, P
        );

        hipError_t error = hipDeviceSynchronize();
        if (error != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(error) << "\n";
            return false;
        }

        // Copy result back
        tensor_D.sync_host();

        // Compute reference on CPU
        std::cout << "Computing reference on CPU...\n";

        cutlass::HostTensor<ElementC, LayoutC> tensor_C_ref({M, N});

        // First GEMM
        cutlass::reference::host::Gemm<
            ElementA, LayoutA,
            ElementB, LayoutB,
            ElementC, LayoutC,
            float, float
        > gemm_op;

        gemm_op(
            {M, N, K},
            float(1),
            tensor_A.host_view(),
            tensor_B0.host_view(),
            float(0),
            tensor_C_ref.host_view()
        );

        // Second GEMM
        gemm_op(
            {M, P, N},
            float(1),
            tensor_C_ref.host_view(),
            tensor_B1.host_view(),
            float(0),
            tensor_D_ref.host_view()
        );

        // Compare results
        bool passed = true;
        float max_error = 0.0f;
        for (int i = 0; i < M * P; ++i) {
            float diff = std::abs(float(tensor_D.host_data()[i]) -
                                 float(tensor_D_ref.host_data()[i]));
            max_error = std::max(max_error, diff);
            if (diff > 0.1f) {
                passed = false;
            }
        }

        std::cout << "Max error: " << max_error << "\n";

        if (passed) {
            std::cout << "*** PASSED ***\n";
        } else {
            std::cout << "*** FAILED ***\n";

            // Print comparison
            std::cout << "\nFirst 4x4 elements:\n";
            std::cout << "GPU result:\n";
            for (int i = 0; i < std::min(4, M); ++i) {
                for (int j = 0; j < std::min(4, P); ++j) {
                    std::cout << float(tensor_D.at({i, j})) << " ";
                }
                std::cout << "\n";
            }
            std::cout << "\nCPU reference:\n";
            for (int i = 0; i < std::min(4, M); ++i) {
                for (int j = 0; j < std::min(4, P); ++j) {
                    std::cout << float(tensor_D_ref.at({i, j})) << " ";
                }
                std::cout << "\n";
            }
        }

        return passed;
    }
};

int main() {
    // Check GPU
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 0);
    std::cout << "Running on: " << props.name << " (SM" << props.major << props.minor << ")\n";

    if (props.major < 8) {
        std::cerr << "This example requires SM80 or newer for FP16 support\n";
        return -1;
    }

    SimplifiedB2bGemmShmem b2b_gemm;

    // Test with small problem sizes
    bool passed = b2b_gemm.run(64, 32, 48, 32);

    return passed ? 0 : -1;
}