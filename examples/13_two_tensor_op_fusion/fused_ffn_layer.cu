#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * This example demonstrates fusing a complete FFN (Feed-Forward Network) layer commonly used in
 * transformer models like LLaMA/GPT. The fusion includes:
 *
 * 本示例演示了在LLaMA/GPT等transformer模型中常用的完整FFN（前馈网络）层的融合。
 * 融合包括以下步骤：
 *
 * Input: [9600, 1024]  // 输入: [序列长度*批量大小, 隐藏维度]
 * 1. GEMM1: [9600, 1024] x [1024, 2730] -> [9600, 2730] (gate projection)  // 门控投影
 * 2. GEMM2: [9600, 1024] x [1024, 2730] -> [9600, 2730] (up projection)    // 上投影
 * 3. SiLU activation on GEMM1 output  // 对GEMM1输出应用SiLU激活函数
 * 4. Element-wise multiplication: SiLU(GEMM1) * GEMM2  // 逐元素乘法：SiLU(GEMM1) * GEMM2
 * 5. LayerNorm on the multiplication result  // 对乘法结果进行层归一化
 * 6. GEMM3: [9600, 2730] x [2730, 1024] -> [9600, 1024] (down projection)  // 下投影
 *
 * This mimics the MLP layer in modern transformer architectures.
 * 这模拟了现代transformer架构中的MLP层。
 **************************************************************************************************/

#include <iostream>       // C++标准输入输出流
#include <vector>         // C++标准向量容器
#include <cmath>          // C数学库（用于exp, sqrt等函数）
#include <hip/hip_runtime.h> // CUDA运行时API
#include <hipblas.h>    // cuBLAS库（虽然这里未使用，但可以用于对比）

#include "cutlass/cutlass.h"                               // CUTLASS核心头文件
#include "cutlass/gemm/device/gemm.h"                      // CUTLASS设备端GEMM操作
#include "cutlass/util/host_tensor.h"                      // CUTLASS主机端张量容器
#include "cutlass/util/tensor_view_io.h"                   // 张量视图输入/输出工具
#include "cutlass/util/reference/host/tensor_fill.h"       // 张量填充工具（用于初始化）
#include "cutlass/util/reference/host/tensor_copy.h"       // 张量复制工具
#include "cutlass/util/reference/host/tensor_compare.h"    // 张量比较工具（用于验证）
#include "cutlass/util/reference/host/gemm.h"              // 参考实现的GEMM（用于验证）

////////////////////////////////////////////////////////////////////////////////

// Problem sizes for FFN layer  // FFN层的问题规模
// Typical LLaMA-style dimensions  // 典型的LLaMA风格维度
constexpr int kSeqLength = 9600;   // Sequence length * batch size  // 序列长度 * 批量大小
constexpr int kHiddenDim = 1024;   // Model hidden dimension  // 模型隐藏层维度
constexpr int kFFNDim = 2730;      // FFN intermediate dimension (typically 8/3 * hidden_dim)  // FFN中间层维度（通常是隐藏维度的8/3）

////////////////////////////////////////////////////////////////////////////////

// Simple SiLU activation kernel  // 简单的SiLU激活函数内核
__global__ void silu_multiply_kernel(
    cutlass::half_t const* gate,    // 门控投影的输出（GEMM1的结果）
    cutlass::half_t const* up,      // 上投影的输出（GEMM2的结果）
    cutlass::half_t* output,         // 输出张量（存储SiLU(gate) * up的结果）
    int size                         // 总元素数量
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // 计算全局线程索引
    if (idx < size) {  // 边界检查
        float gate_val = float(gate[idx]);  // 将half转换为float进行计算
        float up_val = float(up[idx]);      // 将half转换为float进行计算

        // SiLU(x) = x * sigmoid(x)  // SiLU激活函数的定义：x * sigmoid(x)
        float sigmoid = 1.0f / (1.0f + expf(-gate_val));  // 计算sigmoid函数
        float silu = gate_val * sigmoid;                  // 计算SiLU激活值

        output[idx] = cutlass::half_t(silu * up_val);     // 将结果转换回half并存储
    }
}

// Simple LayerNorm kernel  // 简单的层归一化内核
__global__ void layernorm_kernel(
    cutlass::half_t const* input,    // 输入张量
    cutlass::half_t* output,          // 输出张量（归一化后的结果）
    int seq_length,                   // 序列长度（行数）
    int hidden_dim,                   // 隐藏维度（列数）
    float eps = 1e-5f                 // 数值稳定性的小量（避免除零）
) {
    int row = blockIdx.x;  // 每个block处理一行
    if (row < seq_length) {  // 边界检查
        // Compute mean  // 计算均值
        float mean = 0.0f;
        for (int i = 0; i < hidden_dim; ++i) {  // 遍历该行的所有元素
            mean += float(input[row * hidden_dim + i]);  // 累加元素值
        }
        mean /= hidden_dim;  // 计算平均值

        // Compute variance  // 计算方差
        float variance = 0.0f;
        for (int i = 0; i < hidden_dim; ++i) {  // 再次遍历该行
            float diff = float(input[row * hidden_dim + i]) - mean;  // 计算与均值的差
            variance += diff * diff;  // 累加平方差
        }
        variance /= hidden_dim;  // 计算方差

        // Normalize  // 归一化
        float stddev = sqrtf(variance + eps);  // 计算标准差（加eps避免除零）
        for (int i = 0; i < hidden_dim; ++i) {  // 第三次遍历该行
            output[row * hidden_dim + i] = cutlass::half_t(
                (float(input[row * hidden_dim + i]) - mean) / stddev  // 标准化：(x - mean) / stddev
            );
        }
    }
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {  // 主函数入口

    std::cout << "=== Fused FFN Layer Example ===\n";  // 融合FFN层示例标题
    std::cout << "This example demonstrates fusing a complete transformer FFN layer:\n";  // 说明这是一个完整的transformer FFN层融合示例
    std::cout << "Input[" << kSeqLength << "," << kHiddenDim << "] -> ";  // 输入维度
    std::cout << "GEMM -> [" << kSeqLength << "," << kFFNDim << "] -> ";  // 第一阶段GEMM后的维度
    std::cout << "SiLU*Up -> LayerNorm -> GEMM -> [" << kSeqLength << "," << kHiddenDim << "]\n\n";  // 完整的处理流程

    // Check GPU  // 检查GPU设备
    hipDeviceProp_t props;  // CUDA设备属性结构体
    hipError_t error = hipGetDeviceProperties(&props, 0);  // 获取设备0的属性
    if (error != hipSuccess) {  // 如果获取失败
        std::cerr << "hipGetDeviceProperties() failed: " << hipGetErrorString(error) << "\n";  // 输出错误信息
        return -1;  // 返回错误代码
    }

    std::cout << "Running on GPU: " << props.name << " (SM" << props.major << props.minor << ")\n";  // 输出GPU名称和计算能力

    // Define data types  // 定义数据类型
    using ElementInput = cutlass::half_t;      // 输入元素类型：半精度浮点数（FP16）
    using ElementOutput = cutlass::half_t;     // 输出元素类型：半精度浮点数（FP16）
    using ElementAccumulator = float;          // 累加器元素类型：单精度浮点数（FP32）

    using LayoutInput = cutlass::layout::RowMajor;     // 输入布局：行主序（每行连续存储）
    using LayoutWeight = cutlass::layout::ColumnMajor;  // 权重布局：列主序（每列连续存储）
    using LayoutOutput = cutlass::layout::RowMajor;    // 输出布局：行主序（每行连续存储）

    // Allocate host tensors  // 分配主机端张量
    cutlass::HostTensor<ElementInput, LayoutInput> tensor_input({kSeqLength, kHiddenDim});          // 输入张量 [9600, 1024]
    cutlass::HostTensor<ElementInput, LayoutWeight> tensor_gate_weight({kHiddenDim, kFFNDim});      // 门控权重 [1024, 2730]
    cutlass::HostTensor<ElementInput, LayoutWeight> tensor_up_weight({kHiddenDim, kFFNDim});        // 上投影权重 [1024, 2730]
    cutlass::HostTensor<ElementInput, LayoutWeight> tensor_down_weight({kFFNDim, kHiddenDim});      // 下投影权重 [2730, 1024]
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_output({kSeqLength, kHiddenDim});       // 最终输出 [9600, 1024]

    // Intermediate tensors  // 中间张量
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_gate_out({kSeqLength, kFFNDim});    // 门控投影输出 [9600, 2730]
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_up_out({kSeqLength, kFFNDim});      // 上投影输出 [9600, 2730]
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_activated({kSeqLength, kFFNDim});   // SiLU激活后的输出 [9600, 2730]
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_normed({kSeqLength, kFFNDim});      // 层归一化后的输出 [9600, 2730]

    // Initialize input tensors with random data  // 用随机数据初始化输入张量
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_input.host_view(),  // 要填充的张量视图
        1,                          // 随机数生成的种子
        ElementInput(2),            // 随机数范围的最大值
        ElementInput(-2),           // 随机数范围的最小值
        0                           // 偏移（通常为0）
    );

    cutlass::reference::host::TensorFillRandomUniform(
        tensor_gate_weight.host_view(),
        1,
        ElementInput(0.5),
        ElementInput(-0.5),
        1
    );

    cutlass::reference::host::TensorFillRandomUniform(
        tensor_up_weight.host_view(),
        1,
        ElementInput(0.5),
        ElementInput(-0.5),
        2
    );

    cutlass::reference::host::TensorFillRandomUniform(
        tensor_down_weight.host_view(),
        1,
        ElementInput(0.5),
        ElementInput(-0.5),
        3
    );

    // Copy to device  // 复制到设备端（GPU）
    tensor_input.sync_device();         // 同步输入张量到GPU
    tensor_gate_weight.sync_device();   // 同步门控权重到GPU
    tensor_up_weight.sync_device();     // 同步上投影权重到GPU
    tensor_down_weight.sync_device();   // 同步下投影权重到GPU

    std::cout << "Executing FFN operations...\n";

    // Define GEMM operation for FP16  // 定义FP16的GEMM操作
    using Gemm = cutlass::gemm::device::Gemm<
        ElementInput, LayoutInput,           // A matrix  // A矩阵：输入类型和布局
        ElementInput, LayoutWeight,           // B matrix  // B矩阵：权重类型和布局
        ElementOutput, LayoutOutput,          // C matrix  // C矩阵：输出类型和布局
        ElementAccumulator                    // Accumulator  // 累加器类型（FP32用于更高精度）
    >;

    // GEMM1: Gate projection  // GEMM1：门控投影
    Gemm gemm_gate;  // 创建门控GEMM操作对象
    typename Gemm::Arguments args_gate(  // 配置GEMM参数
        {kSeqLength, kFFNDim, kHiddenDim},   // Problem size  // 问题规模：M=9600, N=2730, K=1024
        tensor_input.device_ref(),            // A  // A矩阵：输入张量 [9600, 1024]
        tensor_gate_weight.device_ref(),      // B  // B矩阵：门控权重 [1024, 2730]
        tensor_gate_out.device_ref(),         // C (unused)  // C矩阵（未使用，因为beta=0）
        tensor_gate_out.device_ref(),         // D (output)  // D矩阵：输出 [9600, 2730]
        {ElementAccumulator(1), ElementAccumulator(0)}  // alpha, beta  // D = alpha*A*B + beta*C，这里alpha=1, beta=0
    );

    cutlass::Status status = gemm_gate(args_gate);  // 执行门控GEMM操作
    if (status != cutlass::Status::kSuccess) {  // 检查执行状态
        std::cerr << "Gate GEMM failed\n";  // 如果失败，输出错误信息
        return -1;  // 返回错误代码
    }

    // GEMM2: Up projection  // GEMM2：上投影
    Gemm gemm_up;  // 创建上投影GEMM操作对象
    typename Gemm::Arguments args_up(  // 配置GEMM参数
        {kSeqLength, kFFNDim, kHiddenDim},  // 问题规模：与门控投影相同
        tensor_input.device_ref(),           // A矩阵：同样使用输入张量 [9600, 1024]
        tensor_up_weight.device_ref(),       // B矩阵：上投影权重 [1024, 2730]
        tensor_up_out.device_ref(),          // C矩阵（未使用）
        tensor_up_out.device_ref(),          // D矩阵：输出 [9600, 2730]
        {ElementAccumulator(1), ElementAccumulator(0)}  // alpha=1, beta=0
    );

    status = gemm_up(args_up);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Up GEMM failed\n";
        return -1;
    }

    // Apply SiLU activation and multiply  // 应用SiLU激活函数并进行逐元素乘法
    dim3 block(256);  // 每个block有256个线程
    dim3 grid((kSeqLength * kFFNDim + block.x - 1) / block.x);  // 计算需要的grid大小（向上取整）
    silu_multiply_kernel<<<grid, block>>>(  // 启动CUDA内核
        tensor_gate_out.device_data(),      // 门控投影的输出（将应用SiLU）
        tensor_up_out.device_data(),        // 上投影的输出（将与SiLU结果相乘）
        tensor_activated.device_data(),     // 输出：SiLU(gate) * up
        kSeqLength * kFFNDim                // 总元素数：9600 * 2730
    );

    // Apply LayerNorm  // 应用层归一化
    layernorm_kernel<<<kSeqLength, 1>>>(  // 每行使用一个block（简单实现，非最优）
        tensor_activated.device_data(),   // 输入：激活后的张量
        tensor_normed.device_data(),      // 输出：归一化后的张量
        kSeqLength,                       // 序列长度（行数）
        kFFNDim                           // FFN维度（列数）
    );

    // GEMM3: Down projection  // GEMM3：下投影
    Gemm gemm_down;  // 创建下投影GEMM操作对象
    typename Gemm::Arguments args_down(  // 配置GEMM参数
        {kSeqLength, kHiddenDim, kFFNDim},  // 问题规模：M=9600, N=1024, K=2730
        tensor_normed.device_ref(),          // A矩阵：归一化后的张量 [9600, 2730]
        tensor_down_weight.device_ref(),     // B矩阵：下投影权重 [2730, 1024]
        tensor_output.device_ref(),          // C矩阵（未使用）
        tensor_output.device_ref(),          // D矩阵：最终输出 [9600, 1024]
        {ElementAccumulator(1), ElementAccumulator(0)}  // alpha=1, beta=0
    );

    status = gemm_down(args_down);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Down GEMM failed\n";
        return -1;
    }

    // Synchronize and copy back  // 同步并复制回主机
    hipDeviceSynchronize();  // 等待所有CUDA操作完成
    tensor_output.sync_host();  // 将输出张量从GPU复制回CPU

    std::cout << "FFN layer execution completed successfully!\n\n";

    // Performance measurement  // 性能测量
    std::cout << "=== Performance Benchmark ===\n";  // 性能基准测试标题
    std::cout << "Problem size: [" << kSeqLength << ", " << kHiddenDim << "] -> ["  // 输出问题规模
              << kSeqLength << ", " << kFFNDim << "] -> [" << kSeqLength << ", " << kHiddenDim << "]\n";  // 显示完整的数据流维度变化

    hipEvent_t start, stop;  // CUDA事件，用于精确计时
    hipEventCreate(&start);   // 创建开始事件
    hipEventCreate(&stop);    // 创建结束事件

    const int num_iterations = 100;  // 迭代次数，用于取平均值

    // Warm-up  // 预热（让GPU达到稳定状态）
    for (int i = 0; i < 10; ++i) {  // 执行10次预热迭代
        gemm_gate(args_gate);
        gemm_up(args_up);
        silu_multiply_kernel<<<grid, block>>>(
            tensor_gate_out.device_data(),
            tensor_up_out.device_data(),
            tensor_activated.device_data(),
            kSeqLength * kFFNDim
        );
        layernorm_kernel<<<kSeqLength, 1>>>(
            tensor_activated.device_data(),
            tensor_normed.device_data(),
            kSeqLength,
            kFFNDim
        );
        gemm_down(args_down);
    }
    hipDeviceSynchronize();

    // Benchmark  // 性能基准测试
    hipEventRecord(start);  // 记录开始时间
    for (int i = 0; i < num_iterations; ++i) {  // 执行100次迭代
        gemm_gate(args_gate);
        gemm_up(args_up);
        silu_multiply_kernel<<<grid, block>>>(
            tensor_gate_out.device_data(),
            tensor_up_out.device_data(),
            tensor_activated.device_data(),
            kSeqLength * kFFNDim
        );
        layernorm_kernel<<<kSeqLength, 1>>>(
            tensor_activated.device_data(),
            tensor_normed.device_data(),
            kSeqLength,
            kFFNDim
        );
        gemm_down(args_down);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_ms;  // 存储经过的时间（毫秒）
    hipEventElapsedTime(&time_ms, start, stop);  // 计算start和stop之间的时间差

    std::cout << "Average time per FFN layer: " << time_ms / num_iterations << " ms\n";  // 输出每个FFN层的平均执行时间

    // Calculate FLOPs  // 计算浮点运算次数
    double flops = 2.0 * kSeqLength * kHiddenDim * kFFNDim * 2 +  // Two GEMMs to FFN dim  // 两个GEMM到FFN维度（门控和上投影）
                   2.0 * kSeqLength * kFFNDim * kHiddenDim +       // Down projection  // 下投影GEMM
                   kSeqLength * kFFNDim * 5;                        // SiLU and multiply  // SiLU激活和乘法运算

    double tflops = (flops * num_iterations) / (time_ms * 1e9);  // 计算TFLOPS（每秒万亿次浮点运算）
    std::cout << "Performance: " << tflops << " TFLOPS\n";  // 输出性能指标

    // Memory bandwidth  // 内存带宽计算
    double bytes = sizeof(ElementInput) * (  // 计算总字节数
        kSeqLength * kHiddenDim +           // Input  // 输入数据
        kHiddenDim * kFFNDim * 2 +           // Gate and up weights  // 门控和上投影权重
        kFFNDim * kHiddenDim +               // Down weight  // 下投影权重
        kSeqLength * kFFNDim * 4 +           // Intermediate results  // 中间结果（gate_out, up_out, activated, normed）
        kSeqLength * kHiddenDim              // Output  // 输出数据
    );
    double bandwidth = (bytes * num_iterations) / (time_ms * 1e6);  // 计算带宽（GB/s）
    std::cout << "Memory bandwidth: " << bandwidth << " GB/s\n";  // 输出内存带宽

    std::cout << "\n=== Fusion Opportunities ===\n";  // 融合优化机会
    std::cout << "1. Fuse gate and up GEMMs (share input loading)\n";  // 1. 融合门控和上投影GEMM（共享输入加载）
    std::cout << "2. Fuse SiLU activation with GEMM epilogue\n";  // 2. 将SiLU激活函数融合到GEMM的epilogue阶段
    std::cout << "3. Keep intermediate results in shared memory\n";  // 3. 将中间结果保存在共享内存中
    std::cout << "4. Fuse LayerNorm with down projection prologue\n";  // 4. 将层归一化融合到下投影的prologue阶段
    std::cout << "5. Use persistent kernels to avoid global memory traffic\n";  // 5. 使用持久化内核避免全局内存访问

    hipEventDestroy(start);  // 销毁开始事件
    hipEventDestroy(stop);   // 销毁结束事件

    return 0;  // 程序正常退出
}