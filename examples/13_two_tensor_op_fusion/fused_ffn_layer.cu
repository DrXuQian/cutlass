#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * This example demonstrates fusing a complete FFN (Feed-Forward Network) layer commonly used in
 * transformer models like LLaMA/GPT. The fusion includes:
 *
 * Input: [9600, 1024]
 * 1. GEMM1: [9600, 1024] x [1024, 2730] -> [9600, 2730] (gate projection)
 * 2. GEMM2: [9600, 1024] x [1024, 2730] -> [9600, 2730] (up projection)
 * 3. SiLU activation on GEMM1 output
 * 4. Element-wise multiplication: SiLU(GEMM1) * GEMM2
 * 5. LayerNorm on the multiplication result
 * 6. GEMM3: [9600, 2730] x [2730, 1024] -> [9600, 1024] (down projection)
 *
 * This mimics the MLP layer in modern transformer architectures.
 **************************************************************************************************/

#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/gemm.h"

////////////////////////////////////////////////////////////////////////////////

// Problem sizes for FFN layer
// Typical LLaMA-style dimensions
constexpr int kSeqLength = 9600;   // Sequence length * batch size
constexpr int kHiddenDim = 1024;   // Model hidden dimension
constexpr int kFFNDim = 2730;      // FFN intermediate dimension (typically 8/3 * hidden_dim)

////////////////////////////////////////////////////////////////////////////////

// Simple SiLU activation kernel
__global__ void silu_multiply_kernel(
    cutlass::half_t const* gate,
    cutlass::half_t const* up,
    cutlass::half_t* output,
    int size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float gate_val = float(gate[idx]);
        float up_val = float(up[idx]);

        // SiLU(x) = x * sigmoid(x)
        float sigmoid = 1.0f / (1.0f + expf(-gate_val));
        float silu = gate_val * sigmoid;

        output[idx] = cutlass::half_t(silu * up_val);
    }
}

// Simple LayerNorm kernel
__global__ void layernorm_kernel(
    cutlass::half_t const* input,
    cutlass::half_t* output,
    int seq_length,
    int hidden_dim,
    float eps = 1e-5f
) {
    int row = blockIdx.x;
    if (row < seq_length) {
        // Compute mean
        float mean = 0.0f;
        for (int i = 0; i < hidden_dim; ++i) {
            mean += float(input[row * hidden_dim + i]);
        }
        mean /= hidden_dim;

        // Compute variance
        float variance = 0.0f;
        for (int i = 0; i < hidden_dim; ++i) {
            float diff = float(input[row * hidden_dim + i]) - mean;
            variance += diff * diff;
        }
        variance /= hidden_dim;

        // Normalize
        float stddev = sqrtf(variance + eps);
        for (int i = 0; i < hidden_dim; ++i) {
            output[row * hidden_dim + i] = cutlass::half_t(
                (float(input[row * hidden_dim + i]) - mean) / stddev
            );
        }
    }
}

////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **argv) {

    std::cout << "=== Fused FFN Layer Example ===\n";
    std::cout << "This example demonstrates fusing a complete transformer FFN layer:\n";
    std::cout << "Input[" << kSeqLength << "," << kHiddenDim << "] -> ";
    std::cout << "GEMM -> [" << kSeqLength << "," << kFFNDim << "] -> ";
    std::cout << "SiLU*Up -> LayerNorm -> GEMM -> [" << kSeqLength << "," << kHiddenDim << "]\n\n";

    // Check GPU
    hipDeviceProp_t props;
    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
        std::cerr << "hipGetDeviceProperties() failed: " << hipGetErrorString(error) << "\n";
        return -1;
    }

    std::cout << "Running on GPU: " << props.name << " (SM" << props.major << props.minor << ")\n";

    // Define data types
    using ElementInput = cutlass::half_t;
    using ElementOutput = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutInput = cutlass::layout::RowMajor;
    using LayoutWeight = cutlass::layout::ColumnMajor;
    using LayoutOutput = cutlass::layout::RowMajor;

    // Allocate host tensors
    cutlass::HostTensor<ElementInput, LayoutInput> tensor_input({kSeqLength, kHiddenDim});
    cutlass::HostTensor<ElementInput, LayoutWeight> tensor_gate_weight({kHiddenDim, kFFNDim});
    cutlass::HostTensor<ElementInput, LayoutWeight> tensor_up_weight({kHiddenDim, kFFNDim});
    cutlass::HostTensor<ElementInput, LayoutWeight> tensor_down_weight({kFFNDim, kHiddenDim});
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_output({kSeqLength, kHiddenDim});

    // Intermediate tensors
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_gate_out({kSeqLength, kFFNDim});
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_up_out({kSeqLength, kFFNDim});
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_activated({kSeqLength, kFFNDim});
    cutlass::HostTensor<ElementOutput, LayoutOutput> tensor_normed({kSeqLength, kFFNDim});

    // Initialize input tensors with random data
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_input.host_view(),
        1,
        ElementInput(2),
        ElementInput(-2),
        0
    );

    cutlass::reference::host::TensorFillRandomUniform(
        tensor_gate_weight.host_view(),
        1,
        ElementInput(0.5),
        ElementInput(-0.5),
        1
    );

    cutlass::reference::host::TensorFillRandomUniform(
        tensor_up_weight.host_view(),
        1,
        ElementInput(0.5),
        ElementInput(-0.5),
        2
    );

    cutlass::reference::host::TensorFillRandomUniform(
        tensor_down_weight.host_view(),
        1,
        ElementInput(0.5),
        ElementInput(-0.5),
        3
    );

    // Copy to device
    tensor_input.sync_device();
    tensor_gate_weight.sync_device();
    tensor_up_weight.sync_device();
    tensor_down_weight.sync_device();

    std::cout << "Executing FFN operations...\n";

    // Define GEMM operation for FP16
    using Gemm = cutlass::gemm::device::Gemm<
        ElementInput, LayoutInput,           // A matrix
        ElementInput, LayoutWeight,           // B matrix
        ElementOutput, LayoutOutput,          // C matrix
        ElementAccumulator                    // Accumulator
    >;

    // GEMM1: Gate projection
    Gemm gemm_gate;
    typename Gemm::Arguments args_gate(
        {kSeqLength, kFFNDim, kHiddenDim},   // Problem size
        tensor_input.device_ref(),            // A
        tensor_gate_weight.device_ref(),      // B
        tensor_gate_out.device_ref(),         // C (unused)
        tensor_gate_out.device_ref(),         // D (output)
        {ElementAccumulator(1), ElementAccumulator(0)}  // alpha, beta
    );

    cutlass::Status status = gemm_gate(args_gate);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Gate GEMM failed\n";
        return -1;
    }

    // GEMM2: Up projection
    Gemm gemm_up;
    typename Gemm::Arguments args_up(
        {kSeqLength, kFFNDim, kHiddenDim},
        tensor_input.device_ref(),
        tensor_up_weight.device_ref(),
        tensor_up_out.device_ref(),
        tensor_up_out.device_ref(),
        {ElementAccumulator(1), ElementAccumulator(0)}
    );

    status = gemm_up(args_up);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Up GEMM failed\n";
        return -1;
    }

    // Apply SiLU activation and multiply
    dim3 block(256);
    dim3 grid((kSeqLength * kFFNDim + block.x - 1) / block.x);
    silu_multiply_kernel<<<grid, block>>>(
        tensor_gate_out.device_data(),
        tensor_up_out.device_data(),
        tensor_activated.device_data(),
        kSeqLength * kFFNDim
    );

    // Apply LayerNorm
    layernorm_kernel<<<kSeqLength, 1>>>(
        tensor_activated.device_data(),
        tensor_normed.device_data(),
        kSeqLength,
        kFFNDim
    );

    // GEMM3: Down projection
    Gemm gemm_down;
    typename Gemm::Arguments args_down(
        {kSeqLength, kHiddenDim, kFFNDim},
        tensor_normed.device_ref(),
        tensor_down_weight.device_ref(),
        tensor_output.device_ref(),
        tensor_output.device_ref(),
        {ElementAccumulator(1), ElementAccumulator(0)}
    );

    status = gemm_down(args_down);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Down GEMM failed\n";
        return -1;
    }

    // Synchronize and copy back
    hipDeviceSynchronize();
    tensor_output.sync_host();

    std::cout << "FFN layer execution completed successfully!\n\n";

    // Performance measurement
    std::cout << "=== Performance Benchmark ===\n";
    std::cout << "Problem size: [" << kSeqLength << ", " << kHiddenDim << "] -> ["
              << kSeqLength << ", " << kFFNDim << "] -> [" << kSeqLength << ", " << kHiddenDim << "]\n";

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    const int num_iterations = 100;

    // Warm-up
    for (int i = 0; i < 10; ++i) {
        gemm_gate(args_gate);
        gemm_up(args_up);
        silu_multiply_kernel<<<grid, block>>>(
            tensor_gate_out.device_data(),
            tensor_up_out.device_data(),
            tensor_activated.device_data(),
            kSeqLength * kFFNDim
        );
        layernorm_kernel<<<kSeqLength, 1>>>(
            tensor_activated.device_data(),
            tensor_normed.device_data(),
            kSeqLength,
            kFFNDim
        );
        gemm_down(args_down);
    }
    hipDeviceSynchronize();

    // Benchmark
    hipEventRecord(start);
    for (int i = 0; i < num_iterations; ++i) {
        gemm_gate(args_gate);
        gemm_up(args_up);
        silu_multiply_kernel<<<grid, block>>>(
            tensor_gate_out.device_data(),
            tensor_up_out.device_data(),
            tensor_activated.device_data(),
            kSeqLength * kFFNDim
        );
        layernorm_kernel<<<kSeqLength, 1>>>(
            tensor_activated.device_data(),
            tensor_normed.device_data(),
            kSeqLength,
            kFFNDim
        );
        gemm_down(args_down);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float time_ms;
    hipEventElapsedTime(&time_ms, start, stop);

    std::cout << "Average time per FFN layer: " << time_ms / num_iterations << " ms\n";

    // Calculate FLOPs
    double flops = 2.0 * kSeqLength * kHiddenDim * kFFNDim * 2 +  // Two GEMMs to FFN dim
                   2.0 * kSeqLength * kFFNDim * kHiddenDim +       // Down projection
                   kSeqLength * kFFNDim * 5;                        // SiLU and multiply

    double tflops = (flops * num_iterations) / (time_ms * 1e9);
    std::cout << "Performance: " << tflops << " TFLOPS\n";

    // Memory bandwidth
    double bytes = sizeof(ElementInput) * (
        kSeqLength * kHiddenDim +           // Input
        kHiddenDim * kFFNDim * 2 +           // Gate and up weights
        kFFNDim * kHiddenDim +               // Down weight
        kSeqLength * kFFNDim * 4 +           // Intermediate results
        kSeqLength * kHiddenDim              // Output
    );
    double bandwidth = (bytes * num_iterations) / (time_ms * 1e6);
    std::cout << "Memory bandwidth: " << bandwidth << " GB/s\n";

    std::cout << "\n=== Fusion Opportunities ===\n";
    std::cout << "1. Fuse gate and up GEMMs (share input loading)\n";
    std::cout << "2. Fuse SiLU activation with GEMM epilogue\n";
    std::cout << "3. Keep intermediate results in shared memory\n";
    std::cout << "4. Fuse LayerNorm with down projection prologue\n";
    std::cout << "5. Use persistent kernels to avoid global memory traffic\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}