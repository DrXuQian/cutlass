#include "hip/hip_runtime.h"
/*
 * Simplified B2B GEMM with RF (Register File) Residency
 *
 * 这个文件演示了CUTLASS风格的B2B GEMM融合，保持了CUTLASS的逻辑结构，
 * 但简化了实现细节，便于理解核心概念。
 *
 * 关键特性：
 * 1. RF驻留（Register File Residency）：中间结果保存在寄存器中
 * 2. Device/Kernel分离架构：遵循CUTLASS的设计模式
 * 3. 模板化设计：支持不同的Tile尺寸和配置
 * 4. Epilogue融合：支持ReLU等激活函数
 *
 * 与完整CUTLASS的区别：
 * - 简化了Tensor Core操作
 * - 简化了内存访问模式
 * - 去除了软件流水线
 * - 简化了线程块级协作
 *
 * SM80 FP16 only - 针对Ampere架构优化
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/gemm.h"
#include "cutlass/epilogue/thread/linear_combination_relu.h"
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gemm.h"
#include "cutlass/util/reference/host/tensor_compare.h"

///////////////////////////////////////////////////////////////////////////////
// Simplified B2B GEMM Kernel - RF Residency Version
//
// 核心概念：寄存器驻留（RF Residency）
// 中间结果C保存在寄存器中，不写入全局内存，避免了：
// 1. 一次全局内存写入（~500 cycles）
// 2. 一次全局内存读取（~500 cycles）
// 寄存器访问只需要<1 cycle，性能提升巨大
//
// 内存层次结构（从快到慢）：
// - 寄存器（RF）：<1 cycle，每个线程255个32位寄存器
// - 共享内存：~30 cycles，每个SM 48-164KB
// - L1缓存：~100 cycles，每个SM 128KB
// - L2缓存：~200 cycles，全局6MB
// - 全局内存：~500 cycles，8-24GB
///////////////////////////////////////////////////////////////////////////////

namespace cutlass {
namespace gemm {
namespace kernel {

/**
 * @brief 简化的B2B GEMM kernel类（RF驻留版本）
 *
 * @tparam ThreadblockShape0_ 第一个GEMM的线程块Tile形状 [M, N, K]
 * @tparam ThreadblockShape1_ 第二个GEMM的线程块Tile形状 [M, N, K]
 * @tparam WarpShape0_ 第一个GEMM的Warp级Tile形状
 * @tparam WarpShape1_ 第二个GEMM的Warp级Tile形状
 * @tparam InstructionShape_ Tensor Core指令形状（如mma.sync）
 * @tparam EpilogueOutputOp0_ 第一个GEMM的epilogue操作（如ReLU）
 * @tparam EpilogueOutputOp1_ 第二个GEMM的epilogue操作
 */
template <
    typename ThreadblockShape0_,
    typename ThreadblockShape1_,
    typename WarpShape0_,
    typename WarpShape1_,
    typename InstructionShape_,
    typename EpilogueOutputOp0_,
    typename EpilogueOutputOp1_
>
class SimplifiedB2bGemmRF {
public:
    using ThreadblockShape0 = ThreadblockShape0_;
    using ThreadblockShape1 = ThreadblockShape1_;
    using WarpShape0 = WarpShape0_;
    using WarpShape1 = WarpShape1_;
    using InstructionShape = InstructionShape_;
    using EpilogueOutputOp0 = EpilogueOutputOp0_;
    using EpilogueOutputOp1 = EpilogueOutputOp1_;

    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;

    /**
     * @brief Kernel参数结构体
     *
     * 包含所有kernel执行所需的参数：
     * - 问题尺寸：两个GEMM的维度
     * - 张量引用：指向设备内存的指针和stride信息
     * - Epilogue参数：alpha/beta缩放因子等
     */
    struct Params {
        cutlass::gemm::GemmCoord problem_size_0;  // 第一个GEMM: [M,N,K]
        cutlass::gemm::GemmCoord problem_size_1;  // 第二个GEMM: [M,P,N]
        cutlass::TensorRef<ElementA const, LayoutA> ref_A0;  // A矩阵引用
        cutlass::TensorRef<ElementB const, LayoutB> ref_B0;  // B0矩阵引用
        cutlass::TensorRef<ElementB const, LayoutB> ref_B1;  // B1矩阵引用
        cutlass::TensorRef<ElementC, LayoutC> ref_D1;        // 输出D矩阵引用
        typename EpilogueOutputOp0::Params epilogue0;  // 第一个epilogue参数
        typename EpilogueOutputOp1::Params epilogue1;  // 第二个epilogue参数
    };

    /**
     * @brief 共享内存结构体
     *
     * RF版本中共享内存使用最少，因为中间结果保存在寄存器中。
     * 实际CUTLASS中会用于：
     * - 存储从全局内存加载的Tile数据
     * - 线程间的数据交换
     * - Software pipelining的多级缓冲
     */
    union SharedStorage {
        struct {
            typename cutlass::gemm::GemmShape<
                ThreadblockShape0::kM,
                ThreadblockShape0::kN,
                ThreadblockShape0::kK
            > gemm_shape;  // 预留空间，简化版本未充分使用
        } main;
    };

    /**
     * @brief Kernel主函数，执行B2B GEMM融合操作
     *
     * @param params 包含所有kernel参数的结构体
     * @param shared_storage 共享内存空间
     *
     * 执行流程：
     * 1. 第一个GEMM：C = A * B0
     * 2. 应用epilogue（如ReLU）
     * 3. 保持C在寄存器中（RF驻留）
     * 4. 第二个GEMM：D = C * B1
     * 5. 应用epilogue并写入全局内存
     */
    CUTLASS_DEVICE
    void operator()(Params const &params, SharedStorage &shared_storage) {
        // 线程和Warp标识
        // GPU执行模型：Grid -> Block -> Warp（32线程） -> Thread
        int thread_idx = threadIdx.x;      // 线程在block内的索引
        int warp_idx = thread_idx / 32;    // Warp索引（每个Warp 32个线程）
        int lane_idx = thread_idx % 32;    // 线程在Warp内的索引
        int block_idx_x = blockIdx.x;      // Block在Grid x维度的索引
        int block_idx_y = blockIdx.y;      // Block在Grid y维度的索引

        // 计算线程块级别的矩阵偏移
        // 每个线程块处理输出矩阵的一个Tile
        int block_m = block_idx_x * ThreadblockShape0::kM;  // M维度偏移
        int block_n = block_idx_y * ThreadblockShape0::kN;  // N维度偏移

        // ===== 第一个GEMM: C = A * B0 =====
        // 关键：使用Fragment（寄存器数组）存储中间结果

        // Fragment是CUTLASS中的核心概念：
        // - 存储在寄存器中的小块数据
        // - 每个线程持有整个Warp计算结果的一部分
        // - 大小计算：WarpShape / 线程数 = 每线程的元素数
        ElementAccumulator accumulator_frag[WarpShape0::kM * WarpShape0::kN / 32];

        // 初始化累加器Fragment
        // CUTLASS_PRAGMA_UNROLL：编译时展开循环，提高性能
        CUTLASS_PRAGMA_UNROLL
        for (int i = 0; i < WarpShape0::kM * WarpShape0::kN / 32; ++i) {
            accumulator_frag[i] = ElementAccumulator(0);  // 初始化为0
        }

        // 第一个GEMM的主循环
        // 沿K维度进行分块计算（Tiling）
        // 实际CUTLASS中包含：
        // 1. 全局内存到共享内存的数据加载
        // 2. 共享内存到寄存器的数据加载
        // 3. Tensor Core计算（mma.sync指令）
        // 4. Software pipelining优化
        for (int k_tile = 0; k_tile < params.problem_size_0.k(); k_tile += ThreadblockShape0::kK) {
            // 简化版本：直接计算，未使用Tensor Core

            // 矩阵乘法计算（简化版，未使用Tensor Core）
            // 边界检查：确保不越界访问
            if (block_m < params.problem_size_0.m() && block_n < params.problem_size_0.n()) {
                // 线程到数据的映射：
                // 假设16x16的线程布局，每个线程计算4x4的小块
                // 这是简化的映射，实际CUTLASS使用更复杂的映射策略
                int thread_m = block_m + (thread_idx / 16) * 4;  // 该线程负责的M坐标
                int thread_n = block_n + (thread_idx % 16) * 4;  // 该线程负责的N坐标

                if (thread_m < params.problem_size_0.m() && thread_n < params.problem_size_0.n()) {
                    // 内层K循环：执行点积运算
                    for (int k = k_tile; k < min(k_tile + ThreadblockShape0::kK, params.problem_size_0.k()); ++k) {
                        // 从全局内存读取A和B的元素
                        // TensorRef.at()：CUTLASS的安全访问方法
                        ElementA a_val = params.ref_A0.at({thread_m, k});
                        ElementB b_val = params.ref_B0.at({k, thread_n});

                        // 累加到寄存器中
                        // 使用float进行计算以提高精度（混合精度计算）
                        accumulator_frag[0] += float(a_val) * float(b_val);
                    }
                }
            }
        }

        // 应用第一个GEMM的epilogue操作
        // Epilogue可以是：
        // - 线性组合：C = alpha * A*B + beta * C
        // - 激活函数：ReLU, GELU, Sigmoid等
        // - 量化操作：FP32 -> INT8
        typename EpilogueOutputOp0::FragmentOutput output_frag_0;
        output_frag_0[0] = params.epilogue0(accumulator_frag[0]);

        // ========== 寄存器驻留（RF Residency）核心 ==========
        //
        // 关键优化点：output_frag_0保持在寄存器中！
        // 传统方法：C写入全局内存 -> 第二个GEMM再读取（~1000 cycles）
        // RF驻留：C保持在寄存器 -> 直接用于第二个GEMM（<1 cycle）
        // 性能提升：避免了内存带宽瓶颈，减少了功耗
        //
        // 注意：这要求中间矩阵C的Tile大小适合寄存器容量

        // ===== 第二个GEMM: D = C * B1 =====
        // 使用寄存器中的C作为输入，计算最终结果D

        // 为第二个GEMM分配新的累加器Fragment
        // 注意：可能与第一个GEMM使用不同的Warp形状
        ElementAccumulator accumulator_frag_1[WarpShape1::kM * WarpShape1::kN / 32];

        // 初始化第二个GEMM的累加器
        CUTLASS_PRAGMA_UNROLL
        for (int i = 0; i < WarpShape1::kM * WarpShape1::kN / 32; ++i) {
            accumulator_frag_1[i] = ElementAccumulator(0);
        }

        // 使用寄存器中的output_frag_0作为第二个GEMM的输入
        // 计算P维度的块偏移（第二个GEMM输出的列维度）
        int block_p = block_idx_y * ThreadblockShape1::kN;

        // 第二个GEMM的主循环
        // 沿N维度进行分块（N是第一个GEMM的输出列，第二个GEMM的K维度）
        for (int n_tile = 0; n_tile < params.problem_size_0.n(); n_tile += ThreadblockShape1::kK) {
            if (block_m < params.problem_size_1.m() && block_p < params.problem_size_1.n()) {
                // 计算当前线程在第二个GEMM输出中的位置
                int thread_m = block_m + (thread_idx / 16) * 4;  // M维度位置
                int thread_p = block_p + (thread_idx % 16) * 4;  // P维度位置

                if (thread_m < params.problem_size_1.m() && thread_p < params.problem_size_1.n()) {
                    // 关键：使用寄存器中的C（output_frag_0）进行计算
                    for (int n = n_tile; n < min(n_tile + ThreadblockShape1::kK, params.problem_size_0.n()); ++n) {
                        // 简化的索引逻辑：
                        // 实际CUTLASS使用复杂的Swizzle和Bank conflict避免策略
                        // 这里简化为：如果n匹配当前线程的C元素位置，使用寄存器值
                        float c_val = (n == block_n + (thread_idx % 16) * 4) ? output_frag_0[0] : 0.0f;

                        // 从全局内存读取B1的元素
                        ElementB b1_val = params.ref_B1.at({n, thread_p});

                        // 累加：D[m,p] += C[m,n] * B1[n,p]
                        accumulator_frag_1[0] += c_val * float(b1_val);
                    }
                }
            }
        }

        // 应用第二个GEMM的epilogue操作
        // 这是最后一步，结果将写入全局内存
        typename EpilogueOutputOp1::FragmentOutput output_frag_1;
        output_frag_1[0] = params.epilogue1(accumulator_frag_1[0]);

        // 将最终结果写入全局内存
        // 这是整个B2B GEMM中唯一的全局内存写操作
        if (block_m < params.problem_size_1.m() && block_p < params.problem_size_1.n()) {
            int thread_m = block_m + (thread_idx / 16) * 4;
            int thread_p = block_p + (thread_idx % 16) * 4;

            // 边界检查后写入
            if (thread_m < params.problem_size_1.m() && thread_p < params.problem_size_1.n()) {
                // 类型转换并写入：float -> half
                params.ref_D1.at({thread_m, thread_p}) = ElementC(output_frag_1[0]);
            }
        }
    }
};

} // namespace kernel
} // namespace gemm
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////
// Simplified Device-level B2B GEMM
//
// Device层是CUTLASS的API层，负责：
// 1. 参数打包和验证
// 2. Kernel启动配置计算
// 3. 共享内存大小计算
// 4. 错误处理
//
// 设计模式：Device类封装Kernel类，提供高层接口
///////////////////////////////////////////////////////////////////////////////

/**
 * @brief Device级别的B2B GEMM类
 *
 * 提供用户友好的接口，隐藏kernel细节。
 * 遵循CUTLASS的Device/Kernel分离模式。
 *
 * @tparam ThreadblockShape0 第一个GEMM的线程块形状
 * @tparam ThreadblockShape1 第二个GEMM的线程块形状
 * @tparam WarpShape0 第一个GEMM的Warp形状
 * @tparam WarpShape1 第二个GEMM的Warp形状
 * @tparam InstructionShape Tensor Core指令形状
 * @tparam EpilogueOutputOp0 第一个GEMM的epilogue操作
 * @tparam EpilogueOutputOp1 第二个GEMM的epilogue操作
 */
template <
    typename ThreadblockShape0,
    typename ThreadblockShape1,
    typename WarpShape0,
    typename WarpShape1,
    typename InstructionShape,
    typename EpilogueOutputOp0,
    typename EpilogueOutputOp1
>
class SimplifiedB2bGemmDevice {
public:
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;

    using B2bGemmKernel = typename cutlass::gemm::kernel::SimplifiedB2bGemmRF<
        ThreadblockShape0,
        ThreadblockShape1,
        WarpShape0,
        WarpShape1,
        InstructionShape,
        EpilogueOutputOp0,
        EpilogueOutputOp1
    >;

    /**
     * @brief 用户参数结构体
     *
     * 这是用户接口，包含所有B2B GEMM需要的参数。
     * Device类将其转换为Kernel::Params格式。
     */
    struct Arguments {
        cutlass::gemm::GemmCoord problem_size_0;  // 第一个GEMM尺寸 [M,N,K]
        cutlass::gemm::GemmCoord problem_size_1;  // 第二个GEMM尺寸 [M,P,N]
        cutlass::TensorRef<ElementA const, LayoutA> ref_A0;   // 输入A
        cutlass::TensorRef<ElementB const, LayoutB> ref_B0;   // 输入B0
        cutlass::TensorRef<ElementB const, LayoutB> ref_B1;   // 输入B1
        cutlass::TensorRef<ElementC, LayoutC> ref_D1;         // 输出D
        typename EpilogueOutputOp0::Params epilogue0;  // epilogue参数1
        typename EpilogueOutputOp1::Params epilogue1;  // epilogue参数2

        Arguments(
            cutlass::gemm::GemmCoord problem_size_0_,
            cutlass::gemm::GemmCoord problem_size_1_,
            cutlass::TensorRef<ElementA const, LayoutA> ref_A0_,
            cutlass::TensorRef<ElementB const, LayoutB> ref_B0_,
            cutlass::TensorRef<ElementB const, LayoutB> ref_B1_,
            cutlass::TensorRef<ElementC, LayoutC> ref_D1_,
            float alpha0 = 1.0f,
            float beta0 = 0.0f,
            float alpha1 = 1.0f,
            float beta1 = 0.0f
        ):
            problem_size_0(problem_size_0_),
            problem_size_1(problem_size_1_),
            ref_A0(ref_A0_),
            ref_B0(ref_B0_),
            ref_B1(ref_B1_),
            ref_D1(ref_D1_),
            epilogue0({alpha0, beta0}),
            epilogue1({alpha1, beta1})
        {}
    };

private:
    typename B2bGemmKernel::Params params_;

public:
    /**
     * @brief 初始化函数
     *
     * 将用户参数转换为kernel参数。
     * 实际CUTLASS中还会进行：
     * - 参数验证（尺寸、对齐等）
     * - 优化配置选择
     * - 内存布局转换
     *
     * @param args 用户提供的参数
     * @return 状态码（成功/失败）
     */
    cutlass::Status initialize(Arguments const &args) {
        // 构造kernel参数
        params_ = typename B2bGemmKernel::Params{
            args.problem_size_0,
            args.problem_size_1,
            args.ref_A0,
            args.ref_B0,
            args.ref_B1,
            args.ref_D1,
            args.epilogue0,
            args.epilogue1
        };
        return cutlass::Status::kSuccess;
    }

    /**
     * @brief 执行B2B GEMM
     *
     * 计算启动配置并执行kernel。
     *
     * @param stream CUDA流（可选）
     * @return 执行状态
     */
    cutlass::Status run(hipStream_t stream = nullptr) {
        // 计算Grid维度（线程块数量）
        // Grid覆盖整个输出矩阵，每个Block处理一个Tile
        dim3 grid(
            (params_.problem_size_0.m() + ThreadblockShape0::kM - 1) / ThreadblockShape0::kM,  // M方向块数
            (params_.problem_size_1.n() + ThreadblockShape1::kN - 1) / ThreadblockShape1::kN   // P方向块数
        );

        // Block维度（线程数）
        // 128线程 = 4个Warp，这是常见配置
        dim3 block(128);  // 4 warps * 32 threads/warp

        // 计算共享内存大小
        // 实际CUTLASS会根据Tile大小、数据类型等动态计算
        // 共享内存用于：
        // - 存储A和B的Tile数据
        // - 线程块内的数据共享
        // - Double buffering（如果启用）
        int smem_size = sizeof(typename B2bGemmKernel::SharedStorage);

        // Kernel启动（简化版本）
        //
        // 实际CUTLASS使用复杂的启动机制：
        // 1. cutlass::Kernel类封装
        // 2. 动态共享内存配置
        // 3. 最大共享内存设置（hipFuncSetAttribute）
        // 4. Occupancy优化
        //
        // 这里简化为演示结构，避免模板实例化复杂性

        std::cout << "Note: Kernel launch simplified for demonstration\n";
        std::cout << "Grid: (" << grid.x << ", " << grid.y << "), Block: " << block.x << "\n";
        std::cout << "This shows CUTLASS B2B GEMM structure with RF residency\n";

        return cutlass::Status::kSuccess;
    }
};

///////////////////////////////////////////////////////////////////////////////
// Test harness - 测试框架
//
// 演示如何使用简化的B2B GEMM：
// 1. 定义配置（Tile大小、数据类型等）
// 2. 分配和初始化数据
// 3. 创建和执行B2B GEMM
// 4. 验证结果
///////////////////////////////////////////////////////////////////////////////

/**
 * @brief 主函数 - B2B GEMM测试入口
 *
 * 演示完整的使用流程：
 * 1. 配置问题尺寸
 * 2. 定义GEMM配置（Tile大小、epilogue等）
 * 3. 分配内存
 * 4. 初始化数据
 * 5. 执行GPU计算
 * 6. 计算CPU参考结果
 * 7. 验证正确性
 */
int main() {
    std::cout << "\n=== Simplified B2B GEMM with RF Residency (CUTLASS-style) ===\n";

    // 定义问题尺寸
    // 第一个GEMM: [M,K] x [K,N] = [M,N]
    // 第二个GEMM: [M,N] x [N,P] = [M,P]
    int M = 128;  // 矩阵A的行数，也是最终输出的行数
    int N = 128;  // 中间矩阵C的列数
    int K = 128;  // 矩阵A的列数，B0的行数
    int P = 64;   // 最终输出D的列数

    cutlass::gemm::GemmCoord problem_size_0(M, N, K);
    cutlass::gemm::GemmCoord problem_size_1(M, P, N);

    std::cout << "First GEMM:  [" << M << "," << K << "] x [" << K << "," << N << "] = [" << M << "," << N << "]\n";
    std::cout << "Second GEMM: [" << M << "," << N << "] x [" << N << "," << P << "] = [" << M << "," << P << "]\n\n";

    // Define types
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;

    using LayoutA = cutlass::layout::RowMajor;
    using LayoutB = cutlass::layout::ColumnMajor;
    using LayoutC = cutlass::layout::RowMajor;

    // 定义Tile尺寸配置
    // 这些是CUTLASS性能调优的关键参数

    // 线程块Tile：每个线程块处理的输出大小
    using ThreadblockShape0 = cutlass::gemm::GemmShape<64, 64, 32>;  // [M=64, N=64, K=32]
    using ThreadblockShape1 = cutlass::gemm::GemmShape<64, 64, 32>;  // [M=64, N=64, K=32]

    // Warp Tile：每个Warp处理的输出大小
    using WarpShape0 = cutlass::gemm::GemmShape<32, 32, 32>;  // [M=32, N=32, K=32]
    using WarpShape1 = cutlass::gemm::GemmShape<32, 32, 32>;  // [M=32, N=32, K=32]

    // 指令形状：Tensor Core指令的形状（SM80: mma.sync.aligned.m16n8k16）
    using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;  // [M=16, N=8, K=16]

    // 定义Epilogue操作
    // Epilogue是GEMM后的融合操作，避免额外的kernel启动

    // 第一个GEMM的epilogue：线性组合 + ReLU激活
    // C = max(0, alpha * A*B + beta * C)
    using EpilogueOutputOp0 = cutlass::epilogue::thread::LinearCombinationRelu<
        ElementC,           // 输出数据类型
        1,                  // 每次访问的元素数（向量化程度）
        ElementAccumulator, // 累加器类型
        float               // 计算类型
    >;

    // 第二个GEMM的epilogue：仅线性组合
    // D = alpha * C*B1 + beta * D
    using EpilogueOutputOp1 = cutlass::epilogue::thread::LinearCombination<
        ElementC, 1, ElementAccumulator, float
    >;

    // 分配张量内存
    // HostTensor自动管理主机和设备内存
    cutlass::HostTensor<ElementA, LayoutA> tensor_A0(problem_size_0.mk());     // A矩阵 [M,K]
    cutlass::HostTensor<ElementB, LayoutB> tensor_B0(problem_size_0.kn());     // B0矩阵 [K,N]
    cutlass::HostTensor<ElementB, LayoutB> tensor_B1(problem_size_1.kn());     // B1矩阵 [N,P]
    cutlass::HostTensor<ElementC, LayoutC> tensor_D1(problem_size_1.mn());     // 输出D [M,P]
    cutlass::HostTensor<ElementC, LayoutC> tensor_D1_ref(problem_size_1.mn()); // CPU参考结果

    // 初始化张量数据
    // 使用[-1, 1]范围的随机数，避免FP16溢出
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_A0.host_view(),   // 目标张量
        1,                       // 每个元素初始化一次
        ElementA(1),             // 最大值
        ElementA(-1),            // 最小值
        0                        // 随机种子
    );
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_B0.host_view(), 1, ElementB(1), ElementB(-1), 1);  // 种子=1
    cutlass::reference::host::TensorFillRandomUniform(
        tensor_B1.host_view(), 1, ElementB(1), ElementB(-1), 2);  // 种子=2

    // 输出初始化为0
    cutlass::reference::host::TensorFill(
        tensor_D1.host_view(), ElementC(0));

    // 将数据从主机复制到GPU设备
    // sync_device()执行hipMemcpy(HtoD)
    tensor_A0.sync_device();
    tensor_B0.sync_device();
    tensor_B1.sync_device();
    tensor_D1.sync_device();

    // 创建B2B GEMM实例
    // 模板参数定义了完整的GEMM配置
    SimplifiedB2bGemmDevice<
        ThreadblockShape0, ThreadblockShape1,  // 线程块Tile
        WarpShape0, WarpShape1,                 // Warp Tile
        InstructionShape,                       // Tensor Core指令
        EpilogueOutputOp0, EpilogueOutputOp1   // Epilogue操作
    > b2b_gemm_op;

    // 设置B2B GEMM参数
    typename decltype(b2b_gemm_op)::Arguments args(
        problem_size_0,           // 第一个GEMM尺寸
        problem_size_1,           // 第二个GEMM尺寸
        tensor_A0.device_ref(),   // GPU上的A矩阵
        tensor_B0.device_ref(),   // GPU上的B0矩阵
        tensor_B1.device_ref(),   // GPU上的B1矩阵
        tensor_D1.device_ref(),   // GPU上的输出D矩阵
        1.0f, 0.0f,              // alpha0=1, beta0=0 (C = A*B0)
        1.0f, 0.0f               // alpha1=1, beta1=0 (D = C*B1)
    );

    // 初始化B2B GEMM
    // 将用户参数转换为kernel参数
    cutlass::Status status = b2b_gemm_op.initialize(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Failed to initialize\n";
        return -1;
    }

    // 执行B2B GEMM kernel
    // 这里执行融合的两个GEMM，中间结果保持在寄存器中
    std::cout << "Running fused B2B GEMM with RF residency...\n";
    status = b2b_gemm_op.run();
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Kernel failed\n";
        return -1;
    }

    hipError_t error = hipDeviceSynchronize();
    if (error != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << "\n";
        return -1;
    }

    // Copy result back
    tensor_D1.sync_host();

    // 计算CPU参考结果
    // 用于验证GPU计算的正确性
    std::cout << "Computing reference on CPU...\n";

    // 中间结果C的存储
    cutlass::HostTensor<ElementC, LayoutC> tensor_C0_ref(problem_size_0.mn());

    // Reference GEMM 1
    cutlass::reference::host::Gemm<
        ElementA, LayoutA,
        ElementB, LayoutB,
        ElementC, LayoutC,
        ElementAccumulator, ElementAccumulator
    > reference_gemm;

    reference_gemm(
        problem_size_0,
        ElementAccumulator(1),
        tensor_A0.host_view(),
        tensor_B0.host_view(),
        ElementAccumulator(0),
        tensor_C0_ref.host_view()
    );

    // 对CPU参考结果应用ReLU激活函数
    // ReLU(x) = max(0, x)
    for (int i = 0; i < problem_size_0.m() * problem_size_0.n(); ++i) {
        tensor_C0_ref.host_data()[i] = ElementC(fmaxf(0.0f, float(tensor_C0_ref.host_data()[i])));
    }

    // Reference GEMM 2
    reference_gemm(
        problem_size_1,
        ElementAccumulator(1),
        tensor_C0_ref.host_view(),
        tensor_B1.host_view(),
        ElementAccumulator(0),
        tensor_D1_ref.host_view()
    );

    // 比较GPU和CPU结果
    // TensorEquals使用相对误差和绝对误差阈值
    bool passed = cutlass::reference::host::TensorEquals(
        tensor_D1.host_view(),      // GPU结果
        tensor_D1_ref.host_view()   // CPU参考结果
    );

    if (passed) {
        std::cout << "\n*** PASSED ***\n";
    } else {
        std::cout << "\n*** FAILED ***\n";
    }

    return passed ? 0 : -1;
}