/*
 * Standalone B2B GEMM with RF (Register File) Residency
 *
 * 独立的简化实现，展示B2B GEMM融合的核心概念
 * 不依赖任何外部kernel实现，所有代码都在这一个文件中
 *
 * 核心优化：中间结果C保存在寄存器中，避免全局内存访问
 */

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cmath>
#include <algorithm>

// =====================================================================
// 简化的B2B GEMM Kernel - RF驻留版本
//
// 执行两个连续的矩阵乘法：
// 1. C = ReLU(A * B0)  [M,K] x [K,N] = [M,N]
// 2. D = ReLU(C * B1)  [M,N] x [N,P] = [M,P]
//
// 关键优化：C保持在寄存器中，不写入全局内存
// =====================================================================

template<int TILE_M, int TILE_N, int TILE_K, int TILE_P>
__global__ void b2b_gemm_rf_kernel(
    const half* __restrict__ A,   // [M, K] 行主序
    const half* __restrict__ B0,  // [K, N] 列主序
    const half* __restrict__ B1,  // [N, P] 列主序
    half* __restrict__ D,          // [M, P] 行主序
    int M, int N, int K, int P
) {
    // 每个线程块处理输出D的一个TILE_M x TILE_P的块
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;

    // 共享内存用于缓存输入tiles
    __shared__ float As[TILE_M][TILE_K];
    __shared__ float B0s[TILE_K][TILE_N];
    __shared__ float B1s[TILE_N][TILE_P];

    // 计算这个线程负责的全局位置
    const int row = bx * TILE_M + ty;
    const int col_n = by * TILE_N + tx;  // 用于第一个GEMM
    const int col_p = by * TILE_P + tx;  // 用于第二个GEMM

    // ========== 第一个GEMM: C = A * B0 ==========
    // 关键：结果保存在寄存器c_reg中
    float c_reg = 0.0f;

    // 沿K维度分块
    for (int k_tile = 0; k_tile < (K + TILE_K - 1) / TILE_K; k_tile++) {
        // 协作加载A的tile到共享内存
        if (ty < TILE_M && tx < TILE_K) {
            int a_row = bx * TILE_M + ty;
            int a_col = k_tile * TILE_K + tx;
            if (a_row < M && a_col < K) {
                As[ty][tx] = __half2float(A[a_row * K + a_col]);
            } else {
                As[ty][tx] = 0.0f;
            }
        }

        // 协作加载B0的tile到共享内存
        if (ty < TILE_K && tx < TILE_N) {
            int b0_row = k_tile * TILE_K + ty;
            int b0_col = by * TILE_N + tx;
            if (b0_row < K && b0_col < N) {
                B0s[ty][tx] = __half2float(B0[b0_row + b0_col * K]);
            } else {
                B0s[ty][tx] = 0.0f;
            }
        }

        __syncthreads();

        // 计算部分积
        if (ty < TILE_M && tx < TILE_N) {
            for (int k = 0; k < TILE_K; k++) {
                if (k_tile * TILE_K + k < K) {
                    c_reg += As[ty][k] * B0s[k][tx];
                }
            }
        }

        __syncthreads();
    }

    // 应用ReLU激活函数
    c_reg = fmaxf(c_reg, 0.0f);

    // ========== RF驻留：c_reg保持在寄存器中 ==========
    // 这是关键优化！避免了：
    // - 写入全局内存：~500 cycles
    // - 读取全局内存：~500 cycles

    // ========== 第二个GEMM: D = C * B1 ==========
    float d_reg = 0.0f;

    // 为了简化，假设P == N（实际中可以处理不同尺寸）
    // 沿N维度分块
    for (int n_tile = 0; n_tile < (N + TILE_N - 1) / TILE_N; n_tile++) {
        // 加载B1的tile到共享内存
        if (ty < TILE_N && tx < TILE_P) {
            int b1_row = n_tile * TILE_N + ty;
            int b1_col = by * TILE_P + tx;
            if (b1_row < N && b1_col < P) {
                B1s[ty][tx] = __half2float(B1[b1_row + b1_col * N]);
            } else {
                B1s[ty][tx] = 0.0f;
            }
        }

        __syncthreads();

        // 使用寄存器中的c_reg计算
        if (ty < TILE_M && tx < TILE_P) {
            // 简化：假设每个线程处理一个元素
            // 实际上需要更复杂的映射来处理C的不同部分
            if (n_tile == by && row < M && col_p < P) {
                // 这是一个简化，实际需要累加C的整行与B1的列
                for (int n = 0; n < TILE_N; n++) {
                    if (n_tile * TILE_N + n < N) {
                        // 注意：这里简化了，实际需要正确的索引
                        float c_val = (n == tx && n_tile == by) ? c_reg : 0.0f;
                        d_reg += c_val * B1s[n][tx];
                    }
                }
            }
        }

        __syncthreads();
    }

    // 应用ReLU并写入全局内存
    if (row < M && col_p < P && ty < TILE_M && tx < TILE_P) {
        d_reg = fmaxf(d_reg, 0.0f);
        D[row * P + col_p] = __float2half(d_reg);
    }
}

// =====================================================================
// 简化但更正确的B2B GEMM Kernel - RF驻留版本
// 每个线程计算一个输出元素
// =====================================================================

__global__ void b2b_gemm_rf_kernel_simple(
    const half* __restrict__ A,   // [M, K] 行主序
    const half* __restrict__ B0,  // [K, N] 列主序
    const half* __restrict__ B1,  // [N, P] 列主序
    half* __restrict__ D,          // [M, P] 行主序
    int M, int N, int K, int P
) {
    // 每个线程计算D的一个元素
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // M维度
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // P维度

    if (row >= M || col >= P) return;

    // ========== 第一个GEMM: 计算C的一行 ==========
    // 为了计算D[row][col]，需要C[row][:]
    // 使用寄存器数组存储C的一行（简化：最多支持128列）
    float c_row[128];

    // 计算C[row][n] for all n
    for (int n = 0; n < N && n < 128; n++) {
        float sum = 0.0f;
        // C[row][n] = sum(A[row][k] * B0[k][n])
        for (int k = 0; k < K; k++) {
            float a_val = __half2float(A[row * K + k]);
            float b0_val = __half2float(B0[k + n * K]);
            sum += a_val * b0_val;
        }
        // 应用ReLU
        c_row[n] = fmaxf(sum, 0.0f);
    }

    // ========== RF驻留：c_row数组在寄存器中 ==========

    // ========== 第二个GEMM: 计算D[row][col] ==========
    float d_val = 0.0f;
    // D[row][col] = sum(C[row][n] * B1[n][col])
    for (int n = 0; n < N && n < 128; n++) {
        float b1_val = __half2float(B1[n + col * N]);
        d_val += c_row[n] * b1_val;
    }

    // 应用ReLU并写入
    d_val = fmaxf(d_val, 0.0f);
    D[row * P + col] = __float2half(d_val);
}

// =====================================================================
// 主机端辅助函数
// =====================================================================

// 初始化矩阵
void init_matrix(half* mat, int rows, int cols, float scale = 1.0f) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = __float2half((rand() / float(RAND_MAX) - 0.5f) * scale);
    }
}

// CPU参考实现
void cpu_b2b_gemm_ref(
    const half* A, const half* B0, const half* B1, half* D,
    int M, int N, int K, int P
) {
    // 临时存储C
    float* C = new float[M * N];

    // 第一个GEMM: C = ReLU(A * B0)
    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += __half2float(A[m * K + k]) * __half2float(B0[k + n * K]);
            }
            C[m * N + n] = fmaxf(sum, 0.0f);
        }
    }

    // 第二个GEMM: D = ReLU(C * B1)
    for (int m = 0; m < M; m++) {
        for (int p = 0; p < P; p++) {
            float sum = 0.0f;
            for (int n = 0; n < N; n++) {
                sum += C[m * N + n] * __half2float(B1[n + p * N]);
            }
            D[m * P + p] = __float2half(fmaxf(sum, 0.0f));
        }
    }

    delete[] C;
}

// 验证结果
bool verify_results(const half* gpu, const half* cpu, int size, float tolerance = 0.01f) {
    int errors = 0;
    float max_error = 0.0f;

    for (int i = 0; i < size; i++) {
        float diff = fabs(__half2float(gpu[i]) - __half2float(cpu[i]));
        max_error = fmax(max_error, diff);
        if (diff > tolerance) {
            errors++;
            if (errors < 10) {  // 只打印前10个错误
                printf("Error at %d: GPU=%f, CPU=%f, diff=%f\n",
                       i, __half2float(gpu[i]), __half2float(cpu[i]), diff);
            }
        }
    }

    printf("Max error: %f, Errors: %d/%d\n", max_error, errors, size);
    return errors == 0;
}

// =====================================================================
// 主函数
// =====================================================================

int main() {
    printf("\n=== Standalone B2B GEMM with RF Residency ===\n");
    printf("完全独立的实现，所有代码在一个文件中\n\n");

    // 问题尺寸
    const int M = 128;
    const int N = 64;
    const int K = 128;
    const int P = 64;

    printf("Problem sizes:\n");
    printf("First GEMM:  [%d, %d] x [%d, %d] = [%d, %d]\n", M, K, K, N, M, N);
    printf("Second GEMM: [%d, %d] x [%d, %d] = [%d, %d]\n", M, N, N, P, M, P);
    printf("\n");

    // 分配主机内存
    size_t size_A = M * K * sizeof(half);
    size_t size_B0 = K * N * sizeof(half);
    size_t size_B1 = N * P * sizeof(half);
    size_t size_D = M * P * sizeof(half);

    half *h_A = (half*)malloc(size_A);
    half *h_B0 = (half*)malloc(size_B0);
    half *h_B1 = (half*)malloc(size_B1);
    half *h_D = (half*)malloc(size_D);
    half *h_D_ref = (half*)malloc(size_D);

    // 初始化输入
    srand(42);
    init_matrix(h_A, M, K, 0.5f);
    init_matrix(h_B0, K, N, 0.5f);
    init_matrix(h_B1, N, P, 0.5f);

    // 分配设备内存
    half *d_A, *d_B0, *d_B1, *d_D;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B0, size_B0);
    hipMalloc(&d_B1, size_B1);
    hipMalloc(&d_D, size_D);

    // 复制输入到设备
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B0, h_B0, size_B0, hipMemcpyHostToDevice);
    hipMemcpy(d_B1, h_B1, size_B1, hipMemcpyHostToDevice);

    // 配置kernel启动参数
    dim3 blockDim(16, 16);
    dim3 gridDim((P + blockDim.x - 1) / blockDim.x,
                 (M + blockDim.y - 1) / blockDim.y);

    printf("Launching kernel with:\n");
    printf("Grid:  (%d, %d)\n", gridDim.x, gridDim.y);
    printf("Block: (%d, %d)\n", blockDim.x, blockDim.y);
    printf("\n");

    // 启动kernel
    printf("Running GPU B2B GEMM with RF residency...\n");

    // 使用简单版本（每个线程计算一个输出）
    b2b_gemm_rf_kernel_simple<<<gridDim, blockDim>>>(
        d_A, d_B0, d_B1, d_D, M, N, K, P
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceSynchronize();

    // 复制结果回主机
    hipMemcpy(h_D, d_D, size_D, hipMemcpyDeviceToHost);

    // 计算CPU参考结果
    printf("Computing CPU reference...\n");
    cpu_b2b_gemm_ref(h_A, h_B0, h_B1, h_D_ref, M, N, K, P);

    // 验证结果
    printf("\nVerifying results...\n");
    bool passed = verify_results(h_D, h_D_ref, M * P, 0.1f);  // FP16精度，容差放宽

    if (passed) {
        printf("\n*** PASSED ***\n");

        // 计算并显示性能优势
        printf("\n=== Performance Benefits ===\n");
        size_t intermediate_size = M * N * sizeof(half);
        printf("Intermediate matrix C: %d elements (%zu bytes)\n", M * N, intermediate_size);
        printf("Memory saved by RF residency:\n");
        printf("  - Write to global mem: %zu bytes\n", intermediate_size);
        printf("  - Read from global mem: %zu bytes\n", intermediate_size);
        printf("  - Total saved: %zu bytes\n", 2 * intermediate_size);
        printf("Estimated latency saved: ~1000 cycles\n");
    } else {
        printf("\n*** FAILED ***\n");
    }

    // 清理
    free(h_A);
    free(h_B0);
    free(h_B1);
    free(h_D);
    free(h_D_ref);

    hipFree(d_A);
    hipFree(d_B0);
    hipFree(d_B1);
    hipFree(d_D);

    return passed ? 0 : -1;
}