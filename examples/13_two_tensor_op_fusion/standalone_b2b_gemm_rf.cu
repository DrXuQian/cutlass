/*
 * Standalone B2B GEMM with RF (Register File) Residency
 *
 * 独立的简化实现，展示B2B GEMM融合的核心概念
 * 不依赖任何外部kernel实现，所有代码都在这一个文件中
 *
 * 核心优化：中间结果C保存在寄存器中，避免全局内存访问
 */

#include <iostream>          // 标准输入输出流
#include <hip/hip_runtime.h>    // CUDA运行时API
#include <hip/hip_fp16.h>       // CUDA半精度浮点数支持
#include <cmath>             // 数学函数库
#include <algorithm>         // 算法库

// =====================================================================
// 简化的B2B GEMM Kernel - RF驻留版本
//
// 执行两个连续的矩阵乘法：
// 1. C = ReLU(A * B0)  [M,K] x [K,N] = [M,N]
// 2. D = ReLU(C * B1)  [M,N] x [N,P] = [M,P]
//
// 关键优化：C保持在寄存器中，不写入全局内存
// =====================================================================

template<int TILE_M, int TILE_N, int TILE_K, int TILE_P>
__global__ void b2b_gemm_rf_kernel(
    const half* __restrict__ A,   // [M, K] 行主序 - 输入矩阵A
    const half* __restrict__ B0,  // [K, N] 列主序 - 第一个GEMM的输入矩阵B0
    const half* __restrict__ B1,  // [N, P] 列主序 - 第二个GEMM的输入矩阵B1
    half* __restrict__ D,          // [M, P] 行主序 - 输出矩阵D
    int M, int N, int K, int P     // 矩阵维度参数
) {
    // 每个线程块处理输出D的一个TILE_M x TILE_P的块
    const int bx = blockIdx.x;      // 线程块在x方向的索引
    const int by = blockIdx.y;      // 线程块在y方向的索引
    const int tx = threadIdx.x;     // 线程在块内x方向的索引
    const int ty = threadIdx.y;     // 线程在块内y方向的索引

    // 共享内存用于缓存输入tiles - 提高内存访问效率
    __shared__ float As[TILE_M][TILE_K];    // 缓存A矩阵的tile
    __shared__ float B0s[TILE_K][TILE_N];   // 缓存B0矩阵的tile
    __shared__ float B1s[TILE_N][TILE_P];   // 缓存B1矩阵的tile

    // 计算这个线程负责的全局位置
    const int row = bx * TILE_M + ty;       // 当前线程处理的行索引
    const int col_n = by * TILE_N + tx;     // 第一个GEMM中的列索引（N维度）
    const int col_p = by * TILE_P + tx;     // 第二个GEMM中的列索引（P维度）

    // ========== 第一个GEMM: C = A * B0 ==========
    // 关键：结果保存在寄存器c_reg中，避免写入全局内存
    float c_reg = 0.0f;  // 寄存器变量，存储中间结果C的一个元素

    // 沿K维度分块 - 将大矩阵分成小块进行计算
    for (int k_tile = 0; k_tile < (K + TILE_K - 1) / TILE_K; k_tile++) {  // 遍历K维度的所有tile
        // 协作加载A的tile到共享内存 - 所有线程共同参与数据加载
        if (ty < TILE_M && tx < TILE_K) {                   // 检查线程是否在有效范围内
            int a_row = bx * TILE_M + ty;                   // 计算A矩阵中的行索引
            int a_col = k_tile * TILE_K + tx;               // 计算A矩阵中的列索引
            if (a_row < M && a_col < K) {                   // 边界检查，防止越界访问
                As[ty][tx] = __half2float(A[a_row * K + a_col]);  // 从全局内存加载并转换为float
            } else {
                As[ty][tx] = 0.0f;                          // 越界位置填充0
            }
        }

        // 协作加载B0的tile到共享内存 - B0是列主序存储
        if (ty < TILE_K && tx < TILE_N) {                   // 检查线程是否在有效范围内
            int b0_row = k_tile * TILE_K + ty;              // 计算B0矩阵中的行索引
            int b0_col = by * TILE_N + tx;                  // 计算B0矩阵中的列索引
            if (b0_row < K && b0_col < N) {                 // 边界检查
                B0s[ty][tx] = __half2float(B0[b0_row + b0_col * K]);  // 列主序访问：row + col * rows
            } else {
                B0s[ty][tx] = 0.0f;                         // 越界位置填充0
            }
        }

        __syncthreads();  // 同步屏障 - 确保所有线程完成数据加载后再继续

        // 计算部分积 - 执行矩阵乘法的核心计算
        if (ty < TILE_M && tx < TILE_N) {                   // 确保线程在有效计算范围内
            for (int k = 0; k < TILE_K; k++) {              // 遍历K维度进行点积计算
                if (k_tile * TILE_K + k < K) {              // 边界检查，避免越界计算
                    c_reg += As[ty][k] * B0s[k][tx];        // 累加点积结果到寄存器
                }
            }
        }

        __syncthreads();  // 同步屏障 - 确保所有线程完成数据加载后再继续
    }

    // 应用ReLU激活函数 - max(x, 0)
    c_reg = fmaxf(c_reg, 0.0f);  // 将负值置为0，正值保持不变

    // ========== RF驻留：c_reg保持在寄存器中 ==========
    // 这是关键优化！避免了：
    // - 写入全局内存：~500 cycles（延迟周期）
    // - 读取全局内存：~500 cycles（延迟周期）
    // 寄存器访问只需要1个周期，性能提升巨大！

    // ========== 第二个GEMM: D = C * B1 ==========
    float d_reg = 0.0f;  // 寄存器变量，存储最终结果D的一个元素

    // 为了简化，假设P == N（实际中可以处理不同尺寸）
    // 沿N维度分块
    for (int n_tile = 0; n_tile < (N + TILE_N - 1) / TILE_N; n_tile++) {
        // 加载B1的tile到共享内存
        if (ty < TILE_N && tx < TILE_P) {
            int b1_row = n_tile * TILE_N + ty;
            int b1_col = by * TILE_P + tx;
            if (b1_row < N && b1_col < P) {
                B1s[ty][tx] = __half2float(B1[b1_row + b1_col * N]);
            } else {
                B1s[ty][tx] = 0.0f;
            }
        }

        __syncthreads();  // 同步屏障 - 确保所有线程完成数据加载后再继续

        // 使用寄存器中的c_reg计算
        if (ty < TILE_M && tx < TILE_P) {
            // 简化：假设每个线程处理一个元素
            // 实际上需要更复杂的映射来处理C的不同部分
            if (n_tile == by && row < M && col_p < P) {
                // 这是一个简化，实际需要累加C的整行与B1的列
                for (int n = 0; n < TILE_N; n++) {
                    if (n_tile * TILE_N + n < N) {
                        // 注意：这里简化了，实际需要正确的索引
                        float c_val = (n == tx && n_tile == by) ? c_reg : 0.0f;
                        d_reg += c_val * B1s[n][tx];
                    }
                }
            }
        }

        __syncthreads();  // 同步屏障 - 确保所有线程完成数据加载后再继续
    }

    // 应用ReLU并写入全局内存
    if (row < M && col_p < P && ty < TILE_M && tx < TILE_P) {
        d_reg = fmaxf(d_reg, 0.0f);
        D[row * P + col_p] = __float2half(d_reg);
    }
}

// =====================================================================
// 简化但更正确的B2B GEMM Kernel - RF驻留版本
// 每个线程计算一个输出元素
// =====================================================================

__global__ void b2b_gemm_rf_kernel_simple(
    const half* __restrict__ A,   // [M, K] 行主序 - 输入矩阵A
    const half* __restrict__ B0,  // [K, N] 列主序 - 第一个GEMM的输入矩阵B0
    const half* __restrict__ B1,  // [N, P] 列主序 - 第二个GEMM的输入矩阵B1
    half* __restrict__ D,          // [M, P] 行主序 - 输出矩阵D
    int M, int N, int K, int P     // 矩阵维度参数
) {
    // 每个线程计算D的一个元素 - 简单但有效的并行策略
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // 计算当前线程负责的行（M维度）
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // 计算当前线程负责的列（P维度）

    if (row >= M || col >= P) return;  // 边界检查 - 超出矩阵范围的线程直接返回

    // ========== 第一个GEMM: 计算C的一行 ==========
    // 为了计算D[row][col]，需要C[row][:] - C的整行
    // 使用寄存器数组存储C的一行（简化实现：最多支持128列）
    float c_row[128];  // 寄存器数组 - 存储中间结果C的一整行

    // 计算C[row][n] for all n - 计算C矩阵的一整行
    for (int n = 0; n < N && n < 128; n++) {          // 遍历N维度（限制128防止寄存器溢出）
        float sum = 0.0f;                             // 累加器初始化
        // C[row][n] = sum(A[row][k] * B0[k][n]) - 矩阵乘法的定义
        for (int k = 0; k < K; k++) {                 // 遍历K维度进行点积
            float a_val = __half2float(A[row * K + k]);     // 读取A[row][k]并转换为float
            float b0_val = __half2float(B0[k + n * K]);     // 读取B0[k][n]（列主序）
            sum += a_val * b0_val;                          // 累加乘积
        }
        // 应用ReLU激活函数
        c_row[n] = fmaxf(sum, 0.0f);                 // 存储激活后的结果到寄存器数组
    }

    // ========== RF驻留：c_row数组在寄存器中 ==========
    // 关键优化点：整行C都保存在寄存器中，无需全局内存读写

    // ========== 第二个GEMM: 计算D[row][col] ==========
    float d_val = 0.0f;                               // 累加器，存储D的一个元素
    // D[row][col] = sum(C[row][n] * B1[n][col]) - 第二个矩阵乘法
    for (int n = 0; n < N && n < 128; n++) {          // 遍历N维度
        float b1_val = __half2float(B1[n + col * N]); // 读取B1[n][col]（列主序）
        d_val += c_row[n] * b1_val;                   // 使用寄存器中的C值进行计算
    }

    // 应用ReLU并写入全局内存
    d_val = fmaxf(d_val, 0.0f);                      // ReLU激活
    D[row * P + col] = __float2half(d_val);          // 转换为half并写入结果
}

// =====================================================================
// 主机端辅助函数
// =====================================================================

// 初始化矩阵 - 用随机值填充矩阵
void init_matrix(half* mat, int rows, int cols, float scale = 1.0f) {
    for (int i = 0; i < rows * cols; i++) {                      // 遍历所有元素
        mat[i] = __float2half((rand() / float(RAND_MAX) - 0.5f) * scale);  // 生成[-0.5*scale, 0.5*scale]范围的随机数
    }
}

// CPU参考实现 - 用于验证GPU计算的正确性
void cpu_b2b_gemm_ref(
    const half* A, const half* B0, const half* B1, half* D,
    int M, int N, int K, int P
) {
    // 临时存储C - CPU版本需要显式分配中间结果内存
    float* C = new float[M * N];

    // 第一个GEMM: C = ReLU(A * B0) - 标准三重循环实现
    for (int m = 0; m < M; m++) {                        // 遍历输出行
        for (int n = 0; n < N; n++) {                    // 遍历输出列
            float sum = 0.0f;                            // 累加器
            for (int k = 0; k < K; k++) {                // K维度点积
                sum += __half2float(A[m * K + k]) * __half2float(B0[k + n * K]);  // A行主序，B0列主序
            }
            C[m * N + n] = fmaxf(sum, 0.0f);            // 应用ReLU并存储
        }
    }

    // 第二个GEMM: D = ReLU(C * B1) - 使用第一个GEMM的结果
    for (int m = 0; m < M; m++) {                        // 遍历输出行
        for (int p = 0; p < P; p++) {                    // 遍历输出列（P维度）
            float sum = 0.0f;                            // 累加器
            for (int n = 0; n < N; n++) {                // N维度点积
                sum += C[m * N + n] * __half2float(B1[n + p * N]);  // C行主序，B1列主序
            }
            D[m * P + p] = __float2half(fmaxf(sum, 0.0f));     // ReLU激活并转换为half
        }
    }

    delete[] C;  // 释放临时内存
}

// 验证结果 - 比较GPU和CPU计算结果
bool verify_results(const half* gpu, const half* cpu, int size, float tolerance = 0.01f) {
    int errors = 0;           // 错误计数器
    float max_error = 0.0f;   // 记录最大误差

    for (int i = 0; i < size; i++) {                              // 遍历所有元素
        float diff = fabs(__half2float(gpu[i]) - __half2float(cpu[i]));  // 计算绝对误差
        max_error = fmax(max_error, diff);                        // 更新最大误差
        if (diff > tolerance) {                                   // 检查是否超过容差
            errors++;                                              // 增加错误计数
            if (errors < 10) {  // 只打印前10个错误，避免输出过多
                printf("Error at %d: GPU=%f, CPU=%f, diff=%f\n",
                       i, __half2float(gpu[i]), __half2float(cpu[i]), diff);
            }
        }
    }

    printf("Max error: %f, Errors: %d/%d\n", max_error, errors, size);
    return errors == 0;
}

// =====================================================================
// 主函数
// =====================================================================

int main() {
    printf("\n=== Standalone B2B GEMM with RF Residency ===\n");
    printf("完全独立的实现，所有代码在一个文件中\n\n");

    // 问题尺寸
    const int M = 128;
    const int N = 64;
    const int K = 128;
    const int P = 64;

    printf("Problem sizes:\n");
    printf("First GEMM:  [%d, %d] x [%d, %d] = [%d, %d]\n", M, K, K, N, M, N);
    printf("Second GEMM: [%d, %d] x [%d, %d] = [%d, %d]\n", M, N, N, P, M, P);
    printf("\n");

    // 分配主机内存
    size_t size_A = M * K * sizeof(half);
    size_t size_B0 = K * N * sizeof(half);
    size_t size_B1 = N * P * sizeof(half);
    size_t size_D = M * P * sizeof(half);

    half *h_A = (half*)malloc(size_A);
    half *h_B0 = (half*)malloc(size_B0);
    half *h_B1 = (half*)malloc(size_B1);
    half *h_D = (half*)malloc(size_D);
    half *h_D_ref = (half*)malloc(size_D);

    // 初始化输入
    srand(42);
    init_matrix(h_A, M, K, 0.5f);
    init_matrix(h_B0, K, N, 0.5f);
    init_matrix(h_B1, N, P, 0.5f);

    // 分配设备内存
    half *d_A, *d_B0, *d_B1, *d_D;
    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B0, size_B0);
    hipMalloc(&d_B1, size_B1);
    hipMalloc(&d_D, size_D);

    // 复制输入到设备
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B0, h_B0, size_B0, hipMemcpyHostToDevice);
    hipMemcpy(d_B1, h_B1, size_B1, hipMemcpyHostToDevice);

    // 配置kernel启动参数
    dim3 blockDim(16, 16);                                   // 每个线程块16x16个线程
    dim3 gridDim((P + blockDim.x - 1) / blockDim.x,        // 计算需要的块数（P维度）
                 (M + blockDim.y - 1) / blockDim.y);        // 计算需要的块数（M维度）

    printf("Launching kernel with:\n");
    printf("Grid:  (%d, %d)\n", gridDim.x, gridDim.y);
    printf("Block: (%d, %d)\n", blockDim.x, blockDim.y);
    printf("\n");

    // 启动kernel
    printf("Running GPU B2B GEMM with RF residency...\n");

    // 使用简单版本（每个线程计算一个输出）
    b2b_gemm_rf_kernel_simple<<<gridDim, blockDim>>>(     // <<<grid, block>>>语法启动kernel
        d_A, d_B0, d_B1, d_D, M, N, K, P                  // 传递设备内存指针和维度参数
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipDeviceSynchronize();

    // 复制结果回主机
    hipMemcpy(h_D, d_D, size_D, hipMemcpyDeviceToHost);

    // 计算CPU参考结果
    printf("Computing CPU reference...\n");
    cpu_b2b_gemm_ref(h_A, h_B0, h_B1, h_D_ref, M, N, K, P);

    // 验证结果
    printf("\nVerifying results...\n");
    bool passed = verify_results(h_D, h_D_ref, M * P, 0.1f);  // FP16精度，容差放宽

    if (passed) {
        printf("\n*** PASSED ***\n");

        // 计算并显示性能优势
        printf("\n=== Performance Benefits ===\n");
        size_t intermediate_size = M * N * sizeof(half);
        printf("Intermediate matrix C: %d elements (%zu bytes)\n", M * N, intermediate_size);
        printf("Memory saved by RF residency:\n");
        printf("  - Write to global mem: %zu bytes\n", intermediate_size);
        printf("  - Read from global mem: %zu bytes\n", intermediate_size);
        printf("  - Total saved: %zu bytes\n", 2 * intermediate_size);
        printf("Estimated latency saved: ~1000 cycles\n");
    } else {
        printf("\n*** FAILED ***\n");
    }

    // 清理
    free(h_A);
    free(h_B0);
    free(h_B1);
    free(h_D);
    free(h_D_ref);

    hipFree(d_A);
    hipFree(d_B0);
    hipFree(d_B1);
    hipFree(d_D);

    return passed ? 0 : -1;
}