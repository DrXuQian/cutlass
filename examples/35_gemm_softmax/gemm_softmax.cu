#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
 * CUTLASS 示例 35: GEMM + Softmax 融合
 * ===================================
 *
 * 此示例展示了一个融合的 GEMM+Softmax 内核，将矩阵乘法与 softmax 激活
 * 在单个 GPU 内核启动中结合。此融合为现代深度学习工作负载中常用的
 * transformer 和注意力机制提供了显著的性能优势。
 *
 * 融合概述:
 * =========
 * 内核执行: D = softmax(alpha * A @ B + beta * C)
 * 其中 softmax 沿输出矩阵的 N 维度按行应用。
 *
 * 性能优势:
 * =========
 * 1. 内存带宽减少: 消除 GEMM 输出的中间存储
 * 2. 内核启动开销: 单个内核 vs 分离的 GEMM + Softmax 启动
 * 3. 缓存效率: 通过在寄存器/共享内存中保持中间结果来改善数据局部性
 * 4. 数值稳定性: 使用带有最大值减法的数值稳定 softmax 实现
 *
 * 关键架构特性:
 * =============
 * - GEMM 计算的 Tensor Core 加速 (Ampere 架构)
 * - 融合的 epilogue，在同一个线程块中计算 GEMM 结果和 softmax
 * - 两遍 softmax 算法: 第一遍找到最大值，第二遍计算 exp 和 sum
 * - 针对 GEMM 和归约操作优化的内存访问模式
 *
 * 常见用例:
 * =========
 * 1. Transformer 注意力: Query-Key 乘法后跟 softmax
 * 2. 分类层: 最终线性层 + softmax 激活
 * 3. 序列到序列模型: 注意力分数计算
 * 4. BERT/GPT 风格模型: 多头注意力机制
 *
 * 实现细节:
 * =========
 * - 使用 CUTLASS GemmSoftmax 模板进行优化融合
 * - 支持批处理操作以处理多个序列
 * - 针对不同问题大小可配置的线程块和 warp 形状
 * - 基于问题维度自动选择最优分块大小
 *
 * 数值考虑:
 * =========
 * 实现使用数值稳定的 softmax 算法:
 * 1. 在指数运算前从每行减去最大值
 * 2. 在单独的归约遍历中计算指数和
 * 3. 通过将每个指数除以和来标准化
 *
 * 这防止了溢出并保持了数值精度，即使对于大的输入值也是如此。
 */

// 标准 C++ 库包含
#include <cmath>        // 数学函数 (exp, log 等)
#include <iostream>     // 标准输入输出流
#include <vector>       // 动态数组容器
#include <limits>       // 数值限制定义

// CUTLASS 核心库
#include "cutlass/cutlass.h"              // CUTLASS 基础定义
#include "cutlass/arch/memory.h"          // 内存架构抽象
#include "cutlass/arch/memory_sm75.h"     // SM75 架构特定内存操作
#include "cutlass/gemm/device/gemm_complex.h"  // 复数 GEMM 设备实现
#include "cutlass/numeric_types.h"        // 数值类型定义 (half_t 等)
#include "cutlass/numeric_size.h"         // 数值大小工具
#include "cutlass/util/command_line.h"    // 命令行解析工具
#include "cutlass/util/host_tensor.h"     // 主机张量容器

// CUTLASS 参考实现 - 用于验证正确性
#include "cutlass/util/reference/host/gemm_complex.h"     // 主机端复数 GEMM 参考
#include "cutlass/util/reference/device/gemm_complex.h"   // 设备端复数 GEMM 参考
#include "cutlass/util/reference/host/tensor_reduce.h"    // 张量归约操作
#include "cutlass/util/reference/host/tensor_compare.h"   // 张量比较工具
#include "cutlass/util/reference/host/tensor_norm.h"      // 张量范数计算
#include "cutlass/util/reference/host/tensor_copy.h"      // 张量拷贝操作
#include "cutlass/util/reference/device/tensor_fill.h"    // 设备端张量填充
#include "cutlass/util/reference/host/tensor_fill.h"      // 主机端张量填充
#include "cutlass/util/reference/host/error_metrics.h"    // 误差度量计算
#include "cutlass/util/tensor_view_io.h"  // 张量视图输入输出
#include "cutlass/numeric_size.h"         // bits_to_bytes 工具函数

// CUTLASS 布局和 epilogue 支持
#include "cutlass/layout/matrix.h"                        // 矩阵布局定义
#include "cutlass/epilogue/thread/linear_combination.h"   // 线性组合 epilogue
/////////////////////////////////////////////////////////////////////////////////////////////////

// 本示例的融合 GEMM+Softmax 实现
#include "gemm_with_softmax.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

// 调试跟踪宏 - 用于输出调试信息和代码执行路径
#define TRACE(x) { std::cout << "gemm_softmax.cu:" << __LINE__ << "  " << x << std::endl; }

/////////////////////////////////////////////////////////////////////////////////////////////////

// 测试结果枚举 - 跟踪验证状态
// 用于标识测试的最终结果状态
enum class Disposition {
  kPassed,      // 所有验证成功通过
  kIncorrect,   // 数值验证失败
  kNotVerified  // 验证被跳过
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// 命令行选项解析
// 配置问题维度、批大小和执行参数
struct Options {

  bool help;                                    // 显示帮助信息标志
  cutlass::gemm::GemmCoord problem_size;        // GEMM 问题大小 (M, N, K)
  int batch_count;                              // 批处理数量
  int iterations;                               // 性能测试迭代次数
  unsigned seed;                                // 随机数种子，用于可重现的结果
  float alpha;                                  // GEMM 缩放因子: alpha * A @ B
  float beta;                                   // 偏置缩放因子: beta * C
  bool verification_enabled;                    // 启用数值正确性检查
  float tolerance;                              // 验证的可接受误差容忍度

  Options():
    help(false),                        // 默认不显示帮助
    problem_size({16, 24, 64}),         // 默认: M=16, N=24, K=64 (小测试用例)
    batch_count(16),                    // 并行处理 16 个矩阵
    iterations(20),                     // 性能测量的计时迭代次数
    seed(2022),                         // 可重现结果的随机种子
    alpha(1),                           // GEMM 缩放因子: alpha * A @ B
    beta(0),                            // 偏置缩放因子: beta * C (默认禁用)
    verification_enabled(true),         // 启用数值正确性检查
    tolerance(1e-5f)                   // 验证的可接受误差容忍度
  { }

  bool valid() {

    return true;
  }

  /// 解析命令行参数
  /// 从命令行参数中提取配置选项
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    // 检查是否需要显示帮助信息
    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    // 解析 GEMM 问题维度参数
    cmd.get_cmd_line_argument("m", problem_size.m());  // 矩阵 A 的行数
    cmd.get_cmd_line_argument("n", problem_size.n());  // 矩阵 B 的列数
    cmd.get_cmd_line_argument("k", problem_size.k());  // 内积维度

    // 解析批处理配置
    cmd.get_cmd_line_argument("batch_count", batch_count);

    // 解析 GEMM 缩放参数
    cmd.get_cmd_line_argument("alpha", alpha);  // A*B 的缩放系数
    cmd.get_cmd_line_argument("beta", beta);    // 偏置矩阵 C 的缩放系数

    // 解析测试和验证参数
    cmd.get_cmd_line_argument("iterations", iterations);           // 性能测试迭代次数
    cmd.get_cmd_line_argument("verify", verification_enabled);     // 是否启用验证
    cmd.get_cmd_line_argument("seed", seed);                       // 随机数种子
    cmd.get_cmd_line_argument("tolerance", tolerance);             // 数值误差容忍度
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "35_gemm_softmax example\n\n"
      << "  This example uses the CUTLASS Library to compute GEMM + Softmax for arbitrary problem sizes.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --batch_count=<int>         Batch number\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --seed=<int>                Random number seed (1*)\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform (0 to disable profiling).\n\n"
      << "  --verify=<bool>             If true, performs reference calculation.\n\n"
      << "  --tolerance <float>         Error tolerance\n"
    ;

    out << "\n\nExamples:\n\n"
      << "$ ./examples/35_gemm_softmax/35_gemm_softmax --m=1024 --n=512 \\\n"
      << "     --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Returns true if the environment and Toolkit support this
  bool supported(bool verbose = true) const {

    // Ampere Tensor Core operations exposed with mma.sync and ldmatrix are first available
    // in CUDA 11.0.
    //
    // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
    if (!(__CUDACC_VER_MAJOR__ >= 11)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
      }
      return false;
    }

    hipDeviceProp_t props;

    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
      if (verbose) {
        std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
      }
      return false;
    }

    if (!((props.major * 10 + props.minor) >= 80)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
                  << std::endl;
      }
      return false;
    }

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Testbed {

  //
  // Data Type Configuration
  // ======================
  // These types define the precision and layout for all matrices and computations
  //


  using ElementA = cutlass::half_t;        // Input matrix A: FP16 for memory efficiency
  using ElementB = cutlass::half_t;        // Input matrix B: FP16 for memory efficiency
  using ElementC = cutlass::half_t;        // Input bias matrix C: FP16
  using ElementCompute = float;            // Internal accumulation: FP32 for numerical accuracy
  using ElementD = ElementC;               // GEMM output matrix: FP16
  using ElementSoftmax = ElementC;         // Softmax output: FP16

  // Memory Layout Configuration
  // ===========================
  using LayoutA = cutlass::layout::RowMajor;    // A matrix: rows are contiguous (standard for inputs)
  using LayoutB = cutlass::layout::ColumnMajor;  // B matrix: columns are contiguous (optimized for GEMM)

  // Hierarchical Tile Configuration for Tensor Core Optimization
  // =============================================================
  using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;  // Threadblock tile: 128x128x32
  using WarpShape        = cutlass::gemm::GemmShape<64, 64, 32>;    // Warp tile: 64x64x32 (4 warps per threadblock)
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;     // Tensor Core instruction: 16x8x16 (Ampere)

  // Architecture Configuration
  // ==========================
  using OperatorClass = cutlass::arch::OpClassTensorOp;  // Use Tensor Core units for acceleration
  using ArchTag = cutlass::arch::Sm80;                   // Target Ampere architecture (compute capability 8.0+)

  // Softmax Reduction Tile Configuration
  // ====================================
  // ApplyShape controls the granularity of softmax computation and significantly impacts performance.
  // The configuration balances parallelism with memory access efficiency.
  //
  // Guidelines:
  // - kColumn should be the next multiple of 32 >= (problem_N / alignment) for memory coalescing
  // - kRow should be max(1, 128 / kColumn) to balance thread utilization
  // - Larger kColumn values improve memory bandwidth utilization
  // - Smaller kRow values increase parallelism across batch dimension
  using ApplyShape = cutlass::MatrixShape<1, 1024>;  // Process 1024 elements per softmax reduction

  // Pipeline Configuration
  // ======================
  static int const kStages = 3;  // Number of pipeline stages for overlapping compute and memory access

  // Epilogue Configuration
  // ======================
  // Defines the final operation applied to GEMM results before softmax
  // This linear combination computes: alpha * (A @ B) + beta * C
  using EpilogueFunctorOp = cutlass::epilogue::thread::LinearCombination<
    ElementC,                                    // Output element type
    128 / cutlass::sizeof_bits<ElementC>::value, // Vector width for memory operations
    ElementCompute,                              // Accumulation type for scaling
    ElementCompute                               // Scaling factor type (alpha, beta)
  >;

  // Fused GEMM+Softmax Kernel Configuration
  // =======================================
  // This template instantiation defines the complete fused kernel with all
  // architectural and algorithmic parameters specified above
  using GemmSoftmax = cutlass::GemmSoftmax<
    ElementA, LayoutA,      // Input matrix A configuration
    ElementB, LayoutB,      // Input matrix B configuration
    ElementC,               // Output/bias matrix element type
    ElementCompute,         // Internal computation precision
    OperatorClass,          // Tensor Core operation class
    ArchTag,                // Target GPU architecture
    ThreadblockShape,       // Threadblock-level tile dimensions
    WarpShape,              // Warp-level tile dimensions
    InstructionShape,       // Instruction-level tile dimensions
    EpilogueFunctorOp,      // Linear combination epilogue
    kStages,                // Pipeline stage count
    ApplyShape              // Softmax reduction tile dimensions
  >;

  using ElementNorm = typename GemmSoftmax::ElementNorm;
  using ElementSum = typename GemmSoftmax::ElementSum;
  using LayoutC = typename GemmSoftmax::LayoutC;
  using LayoutN = typename GemmSoftmax::LayoutN;
  using LayoutS = typename GemmSoftmax::LayoutS;
  using MatrixCoord = typename LayoutC::TensorCoord;

  //
  // Memory Management and Data Storage
  // ==================================
  // Host tensors for verification and device allocations for GPU computation
  //

  Options const &options;


  // Reference computation storage (CPU-based verification)
  cutlass::HostTensor<ElementNorm, LayoutC>     reference_N;      // Reference max values per row

  // GPU memory allocations for input/output matrices
  cutlass::DeviceAllocation<ElementA> block_A;         // Input matrix A
  cutlass::DeviceAllocation<ElementB> block_B;         // Input matrix B
  cutlass::DeviceAllocation<ElementC> block_C;         // Input bias matrix C
  cutlass::DeviceAllocation<ElementD> block_D;         // GEMM output matrix D
  cutlass::DeviceAllocation<ElementD> block_Ref;       // Reference GEMM result for verification
  cutlass::DeviceAllocation<ElementSoftmax> block_Softmax; // Final softmax output

  // Intermediate storage for softmax computation
  cutlass::DeviceAllocation<ElementNorm> block_Norm;   // Per-row maximum values (for numerical stability)
  cutlass::DeviceAllocation<ElementSum> block_Sum;     // Per-row exponential sums

  // Calculate number of threadblocks needed to cover the N dimension
  // This determines the storage requirements for partial reductions
  int block_num = (options.problem_size.n() + GemmSoftmax::ThreadblockShape::kN - 1) / GemmSoftmax::ThreadblockShape::kN;

  // Problem dimensions and matrix strides
  cutlass::gemm::GemmCoord problem = options.problem_size;

  // Leading dimensions for matrix layouts (for strided memory access)
  int64_t lda = LayoutA::packed({problem.m(), problem.k()}).stride(0);  // A matrix leading dimension
  int64_t ldb = LayoutB::packed({problem.k(), problem.n()}).stride(0);  // B matrix leading dimension
  int64_t ldc = LayoutC::packed({problem.m(), problem.n()}).stride(0);  // C/D matrix leading dimension

  // Softmax auxiliary arrays use row-major layout for efficient reduction
  int64_t ldn = problem.m();  // Norm array leading dimension
  int64_t lds = ldn;          // Sum array leading dimension (same as norm)

  // Memory size calculations for allocation
  // =======================================

  // Per-batch element counts
  int64_t total_elements_A_per_batch = problem.m() * problem.k();        // A matrix size
  int64_t total_elements_B_per_batch = problem.k() * problem.n();        // B matrix size
  int64_t total_elements_C_per_batch = problem.m() * problem.n();        // C matrix size
  int64_t total_elements_D_per_batch = problem.m() * problem.n();        // D matrix size
  int64_t total_elements_partial_norm_per_batch = block_num * problem.m(); // Partial reduction storage

  // Total element counts across all batches
  int64_t total_elements_A = total_elements_A_per_batch * options.batch_count;
  int64_t total_elements_B = total_elements_B_per_batch * options.batch_count;
  int64_t total_elements_C = total_elements_C_per_batch * options.batch_count;
  int64_t total_elements_D = total_elements_D_per_batch * options.batch_count;
  int64_t total_elements_partial_norm = total_elements_partial_norm_per_batch * options.batch_count;

  //
  // Methods
  //

  Testbed(
    Options const &options_
  ):
    options(options_)
  {
    reference_N.reset({options.problem_size.m(), 1}, false);
  }

  /// Run
  Disposition run() {

    Disposition disposition = Disposition::kNotVerified;

    //
    // Initialize the workspace
    //

    initialize();

    //
    // Launch device kernel
    //
    cutlass::Status status = cutlass::Status::kSuccess;

    status = execute_device_kernel();

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Device execution failed." << std::endl;
      return disposition;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Device synchronize failed with error "
        << hipGetErrorString(result) << std::endl;
      return disposition;
    }

    //
    // Verify
    //

    if (options.verification_enabled) {

      bool passed = verify();

      if (passed) {
        disposition = Disposition::kPassed;
      }
      else {
        disposition = Disposition::kIncorrect;
      }
    }

    //
    // Profiling
    //
    if (options.iterations) {
      profile();
    }

    return disposition;
  }

  /// Random Initialization of Input Data
  /// ====================================
  /// Fills all input matrices with random values in a controlled range
  /// to ensure numerical stability and reproducible testing
  void initialize() {

    block_A.reset(total_elements_A);
    block_B.reset(total_elements_B);
    block_C.reset(total_elements_C);
    block_D.reset(total_elements_D);
    block_Softmax.reset(total_elements_D);
    block_Ref.reset(total_elements_D_per_batch);
    block_Norm.reset(total_elements_partial_norm);
    block_Sum.reset(total_elements_partial_norm);

    // Initialize input matrices with random uniform distribution [-5, 5]
    // Different seeds ensure uncorrelated data across matrices
    cutlass::reference::device::BlockFillRandomUniform(
            block_A.get(), total_elements_A, options.seed, ElementA(5), ElementA(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_B.get(), total_elements_B, options.seed + 1, ElementB(5), ElementB(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_C.get(), total_elements_C, options.seed + 2, ElementC(5), ElementC(-5), 0);

    // Initialize output buffers (will be overwritten during computation)
    cutlass::reference::device::BlockFillRandomUniform(
            block_D.get(), total_elements_D, options.seed + 3, ElementD(5), ElementD(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_Ref.get(), total_elements_D_per_batch, options.seed + 3, ElementD(5), ElementD(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_Softmax.get(), total_elements_D, options.seed + 3, ElementSoftmax(5), ElementSoftmax(-5), 0);

    cutlass::reference::host::TensorFill(
      reference_N.host_view(),
      ElementNorm()
    );

  }

  /// GPU Kernel Execution
  /// =====================
  /// Launches the fused GEMM+Softmax kernel with all configured parameters
  cutlass::Status execute_device_kernel() {

    cutlass::Status status = cutlass::Status::kSuccess;

    //
    // Configure Kernel Arguments
    // ==========================
    // Package all matrices, dimensions, and parameters for kernel launch
    //

    GemmSoftmax::Arguments args(
      options.problem_size,                    // GEMM dimensions (M, N, K)
      options.batch_count,                     // Number of matrices to process
      {block_A.get(), lda},                    // Input matrix A (pointer + leading dimension)
      {block_B.get(), ldb},                    // Input matrix B (pointer + leading dimension)
      {block_C.get(), ldc},                    // Input bias matrix C (pointer + leading dimension)
      {block_D.get(), ldc},                    // GEMM output matrix D (pointer + leading dimension)
      {
        ElementCompute(options.alpha),         // GEMM scaling factor alpha
        ElementCompute(options.beta)           // Bias scaling factor beta
      },
      {block_Norm.get(), ldn},                 // Per-row maximum storage for numerical stability
      {block_Sum.get(), lds},                  // Per-row sum storage for normalization
      {block_Softmax.get(), ldc},              // Final softmax output matrix
      total_elements_A_per_batch,              // Batch stride for matrix A
      total_elements_B_per_batch,              // Batch stride for matrix B
      total_elements_C_per_batch,              // Batch stride for matrix C
      total_elements_D_per_batch,              // Batch stride for matrix D
      total_elements_partial_norm_per_batch,   // Batch stride for norm array
      total_elements_partial_norm_per_batch,   // Batch stride for sum array
      total_elements_D_per_batch               // Batch stride for softmax output
    );

    //
    // Kernel Initialization and Execution
    // ===================================
    //

    GemmSoftmax gemm_softmax;

    // Initialize kernel with arguments and allocate any required workspace
    status = gemm_softmax.initialize(args);
    if (status != cutlass::Status::kSuccess) {
      return status;
    }

    // Execute the fused GEMM+Softmax kernel
    status = gemm_softmax();

    return status;
  }

  template<typename Element>
  bool verify_tensor(std::vector<Element> vector_Input, \
                       std::vector<Element> vector_Input_Ref) {

    auto size = int64_t((vector_Input.size() < vector_Input_Ref.size()) ? vector_Input.size() : vector_Input_Ref.size());
    float abs_tol = options.tolerance;
    float rel_tol = options.tolerance;
    
    for (int64_t i = 0; i < size; ++i) {
      float diff = (float)(vector_Input.at(i) - vector_Input_Ref.at(i));
      float abs_diff = fabs(diff);
      float abs_ref = fabs((float)vector_Input_Ref.at(i));
      float relative_diff = abs_ref > abs_tol ? abs_diff / abs_ref : 0;
      if ( (isnan(abs_diff) || isinf(abs_diff)) ||  (abs_diff > rel_tol && relative_diff > rel_tol)) {
        printf("diff = %f, {%f, %f}.\n", abs_diff, (float)(vector_Input.at(i)), (float)(vector_Input_Ref.at(i)));
        return false;
      }

    }

    return true;
  }

  /// Numerical Verification Against Reference Implementation
  /// =======================================================
  /// Computes reference results using separate GEMM and softmax operations,
  /// then compares against the fused kernel output for correctness
  bool verify() {

    LayoutA layout_A(lda);
    LayoutB layout_B(ldb);
    LayoutC layout_C(ldc);
    LayoutN Layout_N(ldn);
    LayoutS Layout_S(lds);

    MatrixCoord extent_A{problem.m(), problem.k()};
    MatrixCoord extent_B{problem.k(), problem.n()};
    MatrixCoord extent_C{problem.m(), problem.n()};

    // Verify each batch independently
    for (int batch_idx = 0; batch_idx < options.batch_count; batch_idx++) {

      cutlass::TensorView<ElementA, LayoutA> view_A(block_A.get() + total_elements_A_per_batch * batch_idx, layout_A, extent_A);
      cutlass::TensorView<ElementB, LayoutB> view_B(block_B.get() + total_elements_B_per_batch * batch_idx, layout_B, extent_B);
      cutlass::TensorView<ElementC, LayoutC> view_C(block_C.get() + total_elements_C_per_batch * batch_idx, layout_C, extent_C);
      cutlass::TensorView<ElementC, LayoutC> view_Ref_device(block_Ref.get(), layout_C, extent_C);

      cutlass::reference::device::GemmComplex<
          ElementA, LayoutA,
          ElementB, LayoutB,
          ElementC, LayoutC, 
          ElementCompute, ElementCompute
      >(
        problem,
        options.alpha, 
        view_A,
        cutlass::ComplexTransform::kNone,
        view_B,
        cutlass::ComplexTransform::kNone,
        options.beta, 
        view_C, 
        view_Ref_device, 
        ElementCompute(0)
      );

      // Copy reference results to host memory for verification
      std::vector<ElementD> matrix_D_Ref(layout_C.capacity(extent_C));
      cutlass::device_memory::copy_to_host(matrix_D_Ref.data(), block_Ref.get(), matrix_D_Ref.size());
      cutlass::TensorView<ElementD, LayoutC> view_Ref(matrix_D_Ref.data(), layout_C, extent_C);

      std::vector<ElementSoftmax> matrix_Softmax_Ref(layout_C.capacity(extent_C));
      cutlass::TensorView<ElementSoftmax, LayoutC> view_Softmax_Ref(matrix_Softmax_Ref.data(), layout_C, extent_C);

      // Copy computed results to host memory
      std::vector<ElementD> matrix_D(layout_C.capacity(extent_C));
      cutlass::device_memory::copy_to_host(matrix_D.data(), block_D.get() + total_elements_D_per_batch * batch_idx, matrix_D.size());

      std::vector<ElementD> matrix_Softmax(layout_C.capacity(extent_C));
      cutlass::device_memory::copy_to_host(matrix_Softmax.data(), block_Softmax.get() + total_elements_D_per_batch * batch_idx, matrix_Softmax.size());

      // Compute row-wise maximum for numerical stability (reference implementation)
      // This mimics the first pass of the fused kernel's softmax computation
      for (int m = 0; m < options.problem_size.m(); ++m) {
        reference_N.at({m, 0}) = view_Ref.ref().at({m, 0});
        for (int n = 1; n < options.problem_size.n(); ++n) {
          reference_N.at({m, 0}) = std::max(reference_N.at({m, 0}), ElementNorm(view_Ref.ref().at({m, n})));
        }
      }

      // Compute reference softmax using numerically stable algorithm
      // This matches the algorithmic approach used in the fused kernel
      for (int m = 0; m < options.problem_size.m(); ++m) {

        // First pass: compute sum of exponentials (subtract max for stability)
        float sum = float();
        for (int n = 0; n < options.problem_size.n(); ++n) {
          sum += std::exp( float(view_Ref.ref().at({m, n})) - float(reference_N.at({m, 0})) );
        }

        // Compute normalization factor
        float inv_sum = float(1.0f / sum);

        // Second pass: normalize exponentials to get final softmax values
        for (int n = 0; n < options.problem_size.n(); ++n) {
          view_Softmax_Ref.ref().at({m, n}) = ElementSoftmax(
            std::exp( float(view_Ref.ref().at({m, n})) - float(reference_N.at({m, 0})) ) * inv_sum
          );
        }
      }

      // Verification checks - set any of these to 'true' to override the verification checks.
      bool verified_D = false;
      bool verified_Softmax = false;

      // Verify softmax output
      if (!verified_D) {
        verified_D = verify_tensor<ElementC>(matrix_D, matrix_D_Ref);
      }

      if (!verified_Softmax) {
        verified_Softmax = verify_tensor<ElementSoftmax>(matrix_Softmax, matrix_Softmax_Ref);
      }

      if (!verified_D || !verified_Softmax) {

        std::cerr << "Verification check failed for tensor Softmax at batch " << batch_idx << "\n";

        // Summarize which checks failed
        if (!verified_D) {
          std::cerr << "Verification of D tensor failed\n";
        }

        if (!verified_Softmax) {
          std::cerr << "Verification of Softmax tensor failed\n";
        }

        return false;
      }

    }

    return true;
  }

  /// Profiles
  bool profile() {

    //
    // Profile
    //

    cutlass::Status status = cutlass::Status::kSuccess;
    hipError_t result;
    hipEvent_t events[2];
    int const kIterations = options.iterations;

    for (hipEvent_t &evt : events) {
      result = hipEventCreate(&evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[0]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (int iter = 0; iter < kIterations; ++iter) {

      status = execute_device_kernel();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Device execution failed." << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "hipDeviceSynchronize() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    float elapsed_ms = 0;
    result = hipEventElapsedTime(&elapsed_ms, events[0], events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventElapsedTime() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (hipEvent_t &evt : events) {
      result = hipEventDestroy(evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventDestroy() failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    int64_t flops = int64_t(options.problem_size.m()) * options.problem_size.n() * options.problem_size.k() * 2;
    int64_t bytes = cutlass::bits_to_bytes<int64_t>(
      (cutlass::sizeof_bits<ElementD>::value * 2 + cutlass::sizeof_bits<ElementSoftmax>::value) *
      options.problem_size.m() * options.problem_size.n());

    double gflops_per_second = double(flops) * kIterations * options.batch_count / double(elapsed_ms / 1000.0f) / double(1.0e9);
    double gbytes_per_second = double(bytes) * kIterations * options.batch_count / double(elapsed_ms / 1000.0f) / double(1 << 30);

    double elapsed_ms_per_iter = double(elapsed_ms) / kIterations;

    std::cout << "         Problem: "
              << options.problem_size.m() << "-by-" << options.problem_size.n() << "-by-" << options.problem_size.k()
              << ", batch size: " << options.batch_count
              << std::endl;

    std::cout << "         Runtime: " << elapsed_ms_per_iter << " ms\n" << std::endl;

    std::cout << "          GFLOPs: " << gflops_per_second << "  GFLOPs" << std::endl;
    std::cout << "Memory bandwidth: " << gbytes_per_second << "  GiB/s" << std::endl;

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv) {

  // Options parsing
  Options options;
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (!options.supported()) {
    return 0;
  }

  // Run
  Testbed testbed(options);

  Disposition disposition = testbed.run();

  std::cout << std::endl;

  switch (disposition) {
    case Disposition::kPassed:
      std::cout << "Passed" << std::endl;
      break;
    case Disposition::kIncorrect:
      std::cout << "Incorrect" << std::endl;
      break;
    case Disposition::kNotVerified:
      std::cout << "Not verified" << std::endl;
      break;
  }

  return (disposition == Disposition::kPassed ? 0 : -1);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
