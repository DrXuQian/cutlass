#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
 * CUTLASS Example 35: GEMM + Softmax Fusion
 *
 * This example demonstrates a fused GEMM+Softmax kernel that combines matrix multiplication
 * with softmax activation in a single GPU kernel launch. This fusion provides significant
 * performance benefits for transformer and attention mechanisms commonly used in modern
 * deep learning workloads.
 *
 * FUSION OVERVIEW:
 * ================
 * The kernel performs: D = softmax(alpha * A @ B + beta * C)
 * where softmax is applied row-wise across the N dimension of the output matrix.
 *
 * PERFORMANCE BENEFITS:
 * ====================
 * 1. Memory Bandwidth Reduction: Eliminates intermediate storage of GEMM output
 * 2. Kernel Launch Overhead: Single kernel vs. separate GEMM + Softmax launches
 * 3. Cache Efficiency: Better data locality by keeping intermediate results in registers/shared memory
 * 4. Numerical Stability: Uses numerically stable softmax implementation with max subtraction
 *
 * KEY ARCHITECTURAL FEATURES:
 * ===========================
 * - Tensor Core acceleration for GEMM computation (Ampere architecture)
 * - Fused epilogue that computes both GEMM result and softmax in the same threadblock
 * - Two-pass softmax algorithm: first pass finds max, second pass computes exp and sum
 * - Optimized memory access patterns for both GEMM and reduction operations
 *
 * COMMON USE CASES:
 * =================
 * 1. Transformer Attention: Query-Key multiplication followed by softmax
 * 2. Classification Layers: Final linear layer + softmax activation
 * 3. Sequence-to-Sequence Models: Attention score computation
 * 4. BERT/GPT-style Models: Multi-head attention mechanisms
 *
 * IMPLEMENTATION DETAILS:
 * =======================
 * - Uses CUTLASS GemmSoftmax template for optimized fusion
 * - Supports batched operations for processing multiple sequences
 * - Configurable threadblock and warp shapes for different problem sizes
 * - Automatic selection of optimal tile sizes based on problem dimensions
 *
 * NUMERICAL CONSIDERATIONS:
 * =========================
 * The implementation uses a numerically stable softmax algorithm that:
 * 1. Subtracts the maximum value from each row before exponentiation
 * 2. Computes the sum of exponentials in a separate reduction pass
 * 3. Normalizes by dividing each exponential by the sum
 *
 * This prevents overflow and maintains numerical precision even for large input values.
 */

#include <cmath>
#include <iostream>
#include <vector>
#include <limits>

#include "cutlass/cutlass.h"
#include "cutlass/arch/memory.h"
#include "cutlass/arch/memory_sm75.h"
#include "cutlass/gemm/device/gemm_complex.h"
#include "cutlass/numeric_types.h"
#include "cutlass/numeric_size.h"
#include "cutlass/util/command_line.h"
#include "cutlass/util/host_tensor.h"

#include "cutlass/util/reference/host/gemm_complex.h"
#include "cutlass/util/reference/device/gemm_complex.h"
#include "cutlass/util/reference/host/tensor_reduce.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_norm.h"
#include "cutlass/util/reference/host/tensor_copy.h"
#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/error_metrics.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/numeric_size.h" // cutlass::bits_to_bytes

#include "cutlass/layout/matrix.h"
#include "cutlass/epilogue/thread/linear_combination.h"
/////////////////////////////////////////////////////////////////////////////////////////////////

#include "gemm_with_softmax.h"

/////////////////////////////////////////////////////////////////////////////////////////////////

#define TRACE(x) { std::cout << "gemm_softmax.cu:" << __LINE__ << "  " << x << std::endl; }

/////////////////////////////////////////////////////////////////////////////////////////////////

// Test result enumeration to track verification status
enum class Disposition {
  kPassed,      // All verifications passed successfully
  kIncorrect,   // Numerical verification failed
  kNotVerified  // Verification was skipped
};

/////////////////////////////////////////////////////////////////////////////////////////////////

// Command line options parsing
// Configures problem dimensions, batch size, and execution parameters
struct Options {

  bool help;
  cutlass::gemm::GemmCoord problem_size;
  int batch_count;
  int iterations;
  unsigned seed;
  float alpha;
  float beta;
  bool verification_enabled;
  float tolerance;

  Options():
    help(false),
    problem_size({16, 24, 64}),    // Default: M=16, N=24, K=64 (small test case)
    batch_count(16),               // Process 16 matrices in parallel
    iterations(20),                // Number of timing iterations for performance measurement
    seed(2022),                    // Random seed for reproducible results
    alpha(1),                      // GEMM scaling factor: alpha * A @ B
    beta(0),                       // Bias scaling factor: beta * C (disabled by default)
    verification_enabled(true),    // Enable numerical correctness checking
    tolerance(1e-5f)              // Acceptable error tolerance for verification
  { }

  bool valid() {

    return true;
  }

  // Parses the command line
  void parse(int argc, char const **args) {
    cutlass::CommandLine cmd(argc, args);

    if (cmd.check_cmd_line_flag("help")) {
      help = true;
    }

    cmd.get_cmd_line_argument("m", problem_size.m());
    cmd.get_cmd_line_argument("n", problem_size.n());
    cmd.get_cmd_line_argument("k", problem_size.k());

    cmd.get_cmd_line_argument("batch_count", batch_count);

    cmd.get_cmd_line_argument("alpha", alpha);
    cmd.get_cmd_line_argument("beta", beta);

    cmd.get_cmd_line_argument("iterations", iterations);
    cmd.get_cmd_line_argument("verify", verification_enabled);
    cmd.get_cmd_line_argument("seed", seed);
    cmd.get_cmd_line_argument("tolerance", tolerance);
  }

  /// Prints the usage statement.
  std::ostream & print_usage(std::ostream &out) const {

    out << "35_gemm_softmax example\n\n"
      << "  This example uses the CUTLASS Library to compute GEMM + Softmax for arbitrary problem sizes.\n\n"
      << "Options:\n\n"
      << "  --help                      If specified, displays this usage statement.\n\n"
      << "  --m=<int>                   GEMM M dimension\n"
      << "  --n=<int>                   GEMM N dimension\n"
      << "  --k=<int>                   GEMM K dimension\n"
      << "  --batch_count=<int>         Batch number\n"
      << "  --alpha=<f32>               Epilogue scalar alpha\n"
      << "  --beta=<f32>                Epilogue scalar beta\n\n"
      << "  --seed=<int>                Random number seed (1*)\n\n"
      << "  --iterations=<int>          Number of profiling iterations to perform (0 to disable profiling).\n\n"
      << "  --verify=<bool>             If true, performs reference calculation.\n\n"
      << "  --tolerance <float>         Error tolerance\n"
    ;

    out << "\n\nExamples:\n\n"
      << "$ ./examples/35_gemm_softmax/35_gemm_softmax --m=1024 --n=512 \\\n"
      << "     --alpha=2 --beta=0.707 \n\n";

    return out;
  }

  /// Returns true if the environment and Toolkit support this
  bool supported(bool verbose = true) const {

    // Ampere Tensor Core operations exposed with mma.sync and ldmatrix are first available
    // in CUDA 11.0.
    //
    // CUTLASS must be compiled with CUDA 11.0 Toolkit to run these examples.
    if (!(__CUDACC_VER_MAJOR__ >= 11)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
      }
      return false;
    }

    hipDeviceProp_t props;

    hipError_t error = hipGetDeviceProperties(&props, 0);
    if (error != hipSuccess) {
      if (verbose) {
        std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
      }
      return false;
    }

    if (!((props.major * 10 + props.minor) >= 80)) {
      if (verbose) {
        std::cerr << "Ampere Tensor Core operations must be run on a machine with compute capability at least 80."
                  << std::endl;
      }
      return false;
    }

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

struct Testbed {

  //
  // Data Type Configuration
  // ======================
  // These types define the precision and layout for all matrices and computations
  //


  using ElementA = cutlass::half_t;        // Input matrix A: FP16 for memory efficiency
  using ElementB = cutlass::half_t;        // Input matrix B: FP16 for memory efficiency
  using ElementC = cutlass::half_t;        // Input bias matrix C: FP16
  using ElementCompute = float;            // Internal accumulation: FP32 for numerical accuracy
  using ElementD = ElementC;               // GEMM output matrix: FP16
  using ElementSoftmax = ElementC;         // Softmax output: FP16

  // Memory Layout Configuration
  // ===========================
  using LayoutA = cutlass::layout::RowMajor;    // A matrix: rows are contiguous (standard for inputs)
  using LayoutB = cutlass::layout::ColumnMajor;  // B matrix: columns are contiguous (optimized for GEMM)

  // Hierarchical Tile Configuration for Tensor Core Optimization
  // =============================================================
  using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 32>;  // Threadblock tile: 128x128x32
  using WarpShape        = cutlass::gemm::GemmShape<64, 64, 32>;    // Warp tile: 64x64x32 (4 warps per threadblock)
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;     // Tensor Core instruction: 16x8x16 (Ampere)

  // Architecture Configuration
  // ==========================
  using OperatorClass = cutlass::arch::OpClassTensorOp;  // Use Tensor Core units for acceleration
  using ArchTag = cutlass::arch::Sm80;                   // Target Ampere architecture (compute capability 8.0+)

  // Softmax Reduction Tile Configuration
  // ====================================
  // ApplyShape controls the granularity of softmax computation and significantly impacts performance.
  // The configuration balances parallelism with memory access efficiency.
  //
  // Guidelines:
  // - kColumn should be the next multiple of 32 >= (problem_N / alignment) for memory coalescing
  // - kRow should be max(1, 128 / kColumn) to balance thread utilization
  // - Larger kColumn values improve memory bandwidth utilization
  // - Smaller kRow values increase parallelism across batch dimension
  using ApplyShape = cutlass::MatrixShape<1, 1024>;  // Process 1024 elements per softmax reduction

  // Pipeline Configuration
  // ======================
  static int const kStages = 3;  // Number of pipeline stages for overlapping compute and memory access

  // Epilogue Configuration
  // ======================
  // Defines the final operation applied to GEMM results before softmax
  // This linear combination computes: alpha * (A @ B) + beta * C
  using EpilogueFunctorOp = cutlass::epilogue::thread::LinearCombination<
    ElementC,                                    // Output element type
    128 / cutlass::sizeof_bits<ElementC>::value, // Vector width for memory operations
    ElementCompute,                              // Accumulation type for scaling
    ElementCompute                               // Scaling factor type (alpha, beta)
  >;

  // Fused GEMM+Softmax Kernel Configuration
  // =======================================
  // This template instantiation defines the complete fused kernel with all
  // architectural and algorithmic parameters specified above
  using GemmSoftmax = cutlass::GemmSoftmax<
    ElementA, LayoutA,      // Input matrix A configuration
    ElementB, LayoutB,      // Input matrix B configuration
    ElementC,               // Output/bias matrix element type
    ElementCompute,         // Internal computation precision
    OperatorClass,          // Tensor Core operation class
    ArchTag,                // Target GPU architecture
    ThreadblockShape,       // Threadblock-level tile dimensions
    WarpShape,              // Warp-level tile dimensions
    InstructionShape,       // Instruction-level tile dimensions
    EpilogueFunctorOp,      // Linear combination epilogue
    kStages,                // Pipeline stage count
    ApplyShape              // Softmax reduction tile dimensions
  >;

  using ElementNorm = typename GemmSoftmax::ElementNorm;
  using ElementSum = typename GemmSoftmax::ElementSum;
  using LayoutC = typename GemmSoftmax::LayoutC;
  using LayoutN = typename GemmSoftmax::LayoutN;
  using LayoutS = typename GemmSoftmax::LayoutS;
  using MatrixCoord = typename LayoutC::TensorCoord;

  //
  // Memory Management and Data Storage
  // ==================================
  // Host tensors for verification and device allocations for GPU computation
  //

  Options const &options;


  // Reference computation storage (CPU-based verification)
  cutlass::HostTensor<ElementNorm, LayoutC>     reference_N;      // Reference max values per row

  // GPU memory allocations for input/output matrices
  cutlass::DeviceAllocation<ElementA> block_A;         // Input matrix A
  cutlass::DeviceAllocation<ElementB> block_B;         // Input matrix B
  cutlass::DeviceAllocation<ElementC> block_C;         // Input bias matrix C
  cutlass::DeviceAllocation<ElementD> block_D;         // GEMM output matrix D
  cutlass::DeviceAllocation<ElementD> block_Ref;       // Reference GEMM result for verification
  cutlass::DeviceAllocation<ElementSoftmax> block_Softmax; // Final softmax output

  // Intermediate storage for softmax computation
  cutlass::DeviceAllocation<ElementNorm> block_Norm;   // Per-row maximum values (for numerical stability)
  cutlass::DeviceAllocation<ElementSum> block_Sum;     // Per-row exponential sums

  // Calculate number of threadblocks needed to cover the N dimension
  // This determines the storage requirements for partial reductions
  int block_num = (options.problem_size.n() + GemmSoftmax::ThreadblockShape::kN - 1) / GemmSoftmax::ThreadblockShape::kN;

  // Problem dimensions and matrix strides
  cutlass::gemm::GemmCoord problem = options.problem_size;

  // Leading dimensions for matrix layouts (for strided memory access)
  int64_t lda = LayoutA::packed({problem.m(), problem.k()}).stride(0);  // A matrix leading dimension
  int64_t ldb = LayoutB::packed({problem.k(), problem.n()}).stride(0);  // B matrix leading dimension
  int64_t ldc = LayoutC::packed({problem.m(), problem.n()}).stride(0);  // C/D matrix leading dimension

  // Softmax auxiliary arrays use row-major layout for efficient reduction
  int64_t ldn = problem.m();  // Norm array leading dimension
  int64_t lds = ldn;          // Sum array leading dimension (same as norm)

  // Memory size calculations for allocation
  // =======================================

  // Per-batch element counts
  int64_t total_elements_A_per_batch = problem.m() * problem.k();        // A matrix size
  int64_t total_elements_B_per_batch = problem.k() * problem.n();        // B matrix size
  int64_t total_elements_C_per_batch = problem.m() * problem.n();        // C matrix size
  int64_t total_elements_D_per_batch = problem.m() * problem.n();        // D matrix size
  int64_t total_elements_partial_norm_per_batch = block_num * problem.m(); // Partial reduction storage

  // Total element counts across all batches
  int64_t total_elements_A = total_elements_A_per_batch * options.batch_count;
  int64_t total_elements_B = total_elements_B_per_batch * options.batch_count;
  int64_t total_elements_C = total_elements_C_per_batch * options.batch_count;
  int64_t total_elements_D = total_elements_D_per_batch * options.batch_count;
  int64_t total_elements_partial_norm = total_elements_partial_norm_per_batch * options.batch_count;

  //
  // Methods
  //

  Testbed(
    Options const &options_
  ):
    options(options_)
  {
    reference_N.reset({options.problem_size.m(), 1}, false);
  }

  /// Run
  Disposition run() {

    Disposition disposition = Disposition::kNotVerified;

    //
    // Initialize the workspace
    //

    initialize();

    //
    // Launch device kernel
    //
    cutlass::Status status = cutlass::Status::kSuccess;

    status = execute_device_kernel();

    if (status != cutlass::Status::kSuccess) {
      std::cerr << "Device execution failed." << std::endl;
      return disposition;
    }

    hipError_t result = hipDeviceSynchronize();
    if (result != hipSuccess) {
      std::cerr << "Device synchronize failed with error "
        << hipGetErrorString(result) << std::endl;
      return disposition;
    }

    //
    // Verify
    //

    if (options.verification_enabled) {

      bool passed = verify();

      if (passed) {
        disposition = Disposition::kPassed;
      }
      else {
        disposition = Disposition::kIncorrect;
      }
    }

    //
    // Profiling
    //
    if (options.iterations) {
      profile();
    }

    return disposition;
  }

  /// Random Initialization of Input Data
  /// ====================================
  /// Fills all input matrices with random values in a controlled range
  /// to ensure numerical stability and reproducible testing
  void initialize() {

    block_A.reset(total_elements_A);
    block_B.reset(total_elements_B);
    block_C.reset(total_elements_C);
    block_D.reset(total_elements_D);
    block_Softmax.reset(total_elements_D);
    block_Ref.reset(total_elements_D_per_batch);
    block_Norm.reset(total_elements_partial_norm);
    block_Sum.reset(total_elements_partial_norm);

    // Initialize input matrices with random uniform distribution [-5, 5]
    // Different seeds ensure uncorrelated data across matrices
    cutlass::reference::device::BlockFillRandomUniform(
            block_A.get(), total_elements_A, options.seed, ElementA(5), ElementA(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_B.get(), total_elements_B, options.seed + 1, ElementB(5), ElementB(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_C.get(), total_elements_C, options.seed + 2, ElementC(5), ElementC(-5), 0);

    // Initialize output buffers (will be overwritten during computation)
    cutlass::reference::device::BlockFillRandomUniform(
            block_D.get(), total_elements_D, options.seed + 3, ElementD(5), ElementD(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_Ref.get(), total_elements_D_per_batch, options.seed + 3, ElementD(5), ElementD(-5), 0);

    cutlass::reference::device::BlockFillRandomUniform(
            block_Softmax.get(), total_elements_D, options.seed + 3, ElementSoftmax(5), ElementSoftmax(-5), 0);

    cutlass::reference::host::TensorFill(
      reference_N.host_view(),
      ElementNorm()
    );

  }

  /// GPU Kernel Execution
  /// =====================
  /// Launches the fused GEMM+Softmax kernel with all configured parameters
  cutlass::Status execute_device_kernel() {

    cutlass::Status status = cutlass::Status::kSuccess;

    //
    // Configure Kernel Arguments
    // ==========================
    // Package all matrices, dimensions, and parameters for kernel launch
    //

    GemmSoftmax::Arguments args(
      options.problem_size,                    // GEMM dimensions (M, N, K)
      options.batch_count,                     // Number of matrices to process
      {block_A.get(), lda},                    // Input matrix A (pointer + leading dimension)
      {block_B.get(), ldb},                    // Input matrix B (pointer + leading dimension)
      {block_C.get(), ldc},                    // Input bias matrix C (pointer + leading dimension)
      {block_D.get(), ldc},                    // GEMM output matrix D (pointer + leading dimension)
      {
        ElementCompute(options.alpha),         // GEMM scaling factor alpha
        ElementCompute(options.beta)           // Bias scaling factor beta
      },
      {block_Norm.get(), ldn},                 // Per-row maximum storage for numerical stability
      {block_Sum.get(), lds},                  // Per-row sum storage for normalization
      {block_Softmax.get(), ldc},              // Final softmax output matrix
      total_elements_A_per_batch,              // Batch stride for matrix A
      total_elements_B_per_batch,              // Batch stride for matrix B
      total_elements_C_per_batch,              // Batch stride for matrix C
      total_elements_D_per_batch,              // Batch stride for matrix D
      total_elements_partial_norm_per_batch,   // Batch stride for norm array
      total_elements_partial_norm_per_batch,   // Batch stride for sum array
      total_elements_D_per_batch               // Batch stride for softmax output
    );

    //
    // Kernel Initialization and Execution
    // ===================================
    //

    GemmSoftmax gemm_softmax;

    // Initialize kernel with arguments and allocate any required workspace
    status = gemm_softmax.initialize(args);
    if (status != cutlass::Status::kSuccess) {
      return status;
    }

    // Execute the fused GEMM+Softmax kernel
    status = gemm_softmax();

    return status;
  }

  template<typename Element>
  bool verify_tensor(std::vector<Element> vector_Input, \
                       std::vector<Element> vector_Input_Ref) {

    auto size = int64_t((vector_Input.size() < vector_Input_Ref.size()) ? vector_Input.size() : vector_Input_Ref.size());
    float abs_tol = options.tolerance;
    float rel_tol = options.tolerance;
    
    for (int64_t i = 0; i < size; ++i) {
      float diff = (float)(vector_Input.at(i) - vector_Input_Ref.at(i));
      float abs_diff = fabs(diff);
      float abs_ref = fabs((float)vector_Input_Ref.at(i));
      float relative_diff = abs_ref > abs_tol ? abs_diff / abs_ref : 0;
      if ( (isnan(abs_diff) || isinf(abs_diff)) ||  (abs_diff > rel_tol && relative_diff > rel_tol)) {
        printf("diff = %f, {%f, %f}.\n", abs_diff, (float)(vector_Input.at(i)), (float)(vector_Input_Ref.at(i)));
        return false;
      }

    }

    return true;
  }

  /// Numerical Verification Against Reference Implementation
  /// =======================================================
  /// Computes reference results using separate GEMM and softmax operations,
  /// then compares against the fused kernel output for correctness
  bool verify() {

    LayoutA layout_A(lda);
    LayoutB layout_B(ldb);
    LayoutC layout_C(ldc);
    LayoutN Layout_N(ldn);
    LayoutS Layout_S(lds);

    MatrixCoord extent_A{problem.m(), problem.k()};
    MatrixCoord extent_B{problem.k(), problem.n()};
    MatrixCoord extent_C{problem.m(), problem.n()};

    // Verify each batch independently
    for (int batch_idx = 0; batch_idx < options.batch_count; batch_idx++) {

      cutlass::TensorView<ElementA, LayoutA> view_A(block_A.get() + total_elements_A_per_batch * batch_idx, layout_A, extent_A);
      cutlass::TensorView<ElementB, LayoutB> view_B(block_B.get() + total_elements_B_per_batch * batch_idx, layout_B, extent_B);
      cutlass::TensorView<ElementC, LayoutC> view_C(block_C.get() + total_elements_C_per_batch * batch_idx, layout_C, extent_C);
      cutlass::TensorView<ElementC, LayoutC> view_Ref_device(block_Ref.get(), layout_C, extent_C);

      cutlass::reference::device::GemmComplex<
          ElementA, LayoutA,
          ElementB, LayoutB,
          ElementC, LayoutC, 
          ElementCompute, ElementCompute
      >(
        problem,
        options.alpha, 
        view_A,
        cutlass::ComplexTransform::kNone,
        view_B,
        cutlass::ComplexTransform::kNone,
        options.beta, 
        view_C, 
        view_Ref_device, 
        ElementCompute(0)
      );

      // Copy reference results to host memory for verification
      std::vector<ElementD> matrix_D_Ref(layout_C.capacity(extent_C));
      cutlass::device_memory::copy_to_host(matrix_D_Ref.data(), block_Ref.get(), matrix_D_Ref.size());
      cutlass::TensorView<ElementD, LayoutC> view_Ref(matrix_D_Ref.data(), layout_C, extent_C);

      std::vector<ElementSoftmax> matrix_Softmax_Ref(layout_C.capacity(extent_C));
      cutlass::TensorView<ElementSoftmax, LayoutC> view_Softmax_Ref(matrix_Softmax_Ref.data(), layout_C, extent_C);

      // Copy computed results to host memory
      std::vector<ElementD> matrix_D(layout_C.capacity(extent_C));
      cutlass::device_memory::copy_to_host(matrix_D.data(), block_D.get() + total_elements_D_per_batch * batch_idx, matrix_D.size());

      std::vector<ElementD> matrix_Softmax(layout_C.capacity(extent_C));
      cutlass::device_memory::copy_to_host(matrix_Softmax.data(), block_Softmax.get() + total_elements_D_per_batch * batch_idx, matrix_Softmax.size());

      // Compute row-wise maximum for numerical stability (reference implementation)
      // This mimics the first pass of the fused kernel's softmax computation
      for (int m = 0; m < options.problem_size.m(); ++m) {
        reference_N.at({m, 0}) = view_Ref.ref().at({m, 0});
        for (int n = 1; n < options.problem_size.n(); ++n) {
          reference_N.at({m, 0}) = std::max(reference_N.at({m, 0}), ElementNorm(view_Ref.ref().at({m, n})));
        }
      }

      // Compute reference softmax using numerically stable algorithm
      // This matches the algorithmic approach used in the fused kernel
      for (int m = 0; m < options.problem_size.m(); ++m) {

        // First pass: compute sum of exponentials (subtract max for stability)
        float sum = float();
        for (int n = 0; n < options.problem_size.n(); ++n) {
          sum += std::exp( float(view_Ref.ref().at({m, n})) - float(reference_N.at({m, 0})) );
        }

        // Compute normalization factor
        float inv_sum = float(1.0f / sum);

        // Second pass: normalize exponentials to get final softmax values
        for (int n = 0; n < options.problem_size.n(); ++n) {
          view_Softmax_Ref.ref().at({m, n}) = ElementSoftmax(
            std::exp( float(view_Ref.ref().at({m, n})) - float(reference_N.at({m, 0})) ) * inv_sum
          );
        }
      }

      // Verification checks - set any of these to 'true' to override the verification checks.
      bool verified_D = false;
      bool verified_Softmax = false;

      // Verify softmax output
      if (!verified_D) {
        verified_D = verify_tensor<ElementC>(matrix_D, matrix_D_Ref);
      }

      if (!verified_Softmax) {
        verified_Softmax = verify_tensor<ElementSoftmax>(matrix_Softmax, matrix_Softmax_Ref);
      }

      if (!verified_D || !verified_Softmax) {

        std::cerr << "Verification check failed for tensor Softmax at batch " << batch_idx << "\n";

        // Summarize which checks failed
        if (!verified_D) {
          std::cerr << "Verification of D tensor failed\n";
        }

        if (!verified_Softmax) {
          std::cerr << "Verification of Softmax tensor failed\n";
        }

        return false;
      }

    }

    return true;
  }

  /// Profiles
  bool profile() {

    //
    // Profile
    //

    cutlass::Status status = cutlass::Status::kSuccess;
    hipError_t result;
    hipEvent_t events[2];
    int const kIterations = options.iterations;

    for (hipEvent_t &evt : events) {
      result = hipEventCreate(&evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventCreate failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[0]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (int iter = 0; iter < kIterations; ++iter) {

      status = execute_device_kernel();

      if (status != cutlass::Status::kSuccess) {
        std::cerr << "Device execution failed." << std::endl;
        return false;
      }
    }

    result = hipEventRecord(events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventRecord() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    result = hipDeviceSynchronize();

    if (result != hipSuccess) {
      std::cerr << "hipDeviceSynchronize() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    float elapsed_ms = 0;
    result = hipEventElapsedTime(&elapsed_ms, events[0], events[1]);

    if (result != hipSuccess) {
      std::cerr << "hipEventElapsedTime() failed with error " << hipGetErrorString(result) << std::endl;
      return false;
    }

    for (hipEvent_t &evt : events) {
      result = hipEventDestroy(evt);
      if (result != hipSuccess) {
        std::cerr << "hipEventDestroy() failed with error " << hipGetErrorString(result) << std::endl;
        return false;
      }
    }

    int64_t flops = int64_t(options.problem_size.m()) * options.problem_size.n() * options.problem_size.k() * 2;
    int64_t bytes = cutlass::bits_to_bytes<int64_t>(
      (cutlass::sizeof_bits<ElementD>::value * 2 + cutlass::sizeof_bits<ElementSoftmax>::value) *
      options.problem_size.m() * options.problem_size.n());

    double gflops_per_second = double(flops) * kIterations * options.batch_count / double(elapsed_ms / 1000.0f) / double(1.0e9);
    double gbytes_per_second = double(bytes) * kIterations * options.batch_count / double(elapsed_ms / 1000.0f) / double(1 << 30);

    double elapsed_ms_per_iter = double(elapsed_ms) / kIterations;

    std::cout << "         Problem: "
              << options.problem_size.m() << "-by-" << options.problem_size.n() << "-by-" << options.problem_size.k()
              << ", batch size: " << options.batch_count
              << std::endl;

    std::cout << "         Runtime: " << elapsed_ms_per_iter << " ms\n" << std::endl;

    std::cout << "          GFLOPs: " << gflops_per_second << "  GFLOPs" << std::endl;
    std::cout << "Memory bandwidth: " << gbytes_per_second << "  GiB/s" << std::endl;

    return true;
  }
};

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv) {

  // Options parsing
  Options options;
  options.parse(argc, argv);

  if (options.help) {
    options.print_usage(std::cout) << std::endl;
    return 0;
  }

  if (!options.supported()) {
    return 0;
  }

  // Run
  Testbed testbed(options);

  Disposition disposition = testbed.run();

  std::cout << std::endl;

  switch (disposition) {
    case Disposition::kPassed:
      std::cout << "Passed" << std::endl;
      break;
    case Disposition::kIncorrect:
      std::cout << "Incorrect" << std::endl;
      break;
    case Disposition::kNotVerified:
      std::cout << "Not verified" << std::endl;
      break;
  }

  return (disposition == Disposition::kPassed ? 0 : -1);
}

/////////////////////////////////////////////////////////////////////////////////////////////////
