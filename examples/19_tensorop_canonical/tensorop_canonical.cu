#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2017 - 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/

/**
 * CUTLASS Example 19: Tensor Core 规范化布局操作示例
 *
 * 本示例展示了如何使用 CUTLASS 的 Warp 级 Tensor Core API 进行矩阵乘法。
 * 这是理解 CUTLASS 底层 Tensor Core 编程模型的基础示例。
 *
 * 核心技术要点：
 * ============
 * 1. Tensor Core 操作：利用专用硬件单元加速矩阵运算
 * 2. 规范化布局（Canonical Layout）：优化的数据排列方式，减少 bank 冲突
 * 3. Warp 级编程：32 个线程协同执行矩阵运算
 * 4. Fragment 概念：每个线程持有的矩阵数据片段
 * 5. 双缓冲技术：计算与数据加载重叠
 *
 * 性能优化策略：
 * ============
 * - 使用 Tensor Core 指令（mma.sync）实现高吞吐量
 * - 数据在寄存器和共享内存间高效传输
 * - 通过 Fragment 分布减少线程间通信
 * - 利用流水线隐藏内存延迟
 *
 * 本示例需要 NVIDIA Ampere GPU 或更新架构。
 */

// 标准库头文件
#include <iostream>
#include <sstream>
#include <vector>

// CUTLASS 核心头文件
#include "cutlass/cutlass.h"
#include "cutlass/functional.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/gemm/warp/default_mma_tensor_op.h"
#include "cutlass/epilogue/warp/fragment_iterator_tensor_op.h"
#include "cutlass/epilogue/warp/tile_iterator_tensor_op.h"

// CUTLASS 工具头文件
#include "cutlass/util/host_tensor.h"
#include "cutlass/util/tensor_view_io.h"
#include "cutlass/util/reference/host/tensor_compare.h"
#include "cutlass/util/reference/host/tensor_fill.h"
#include "cutlass/util/reference/host/gemm_complex.h"

///////////////////////////////////////////////////////////////////////////////////////////////////

// 定义 Warp 级问题的矩阵维度
// M: 输出矩阵行数, N: 输出矩阵列数, K: 矩阵乘法的归约维度
int const kM = 27;
int const kN = 31;
int const kK = 17;

///////////////////////////////////////////////////////////////////////////////////////////////////

// 定义 Warp 级 GEMM 操作符
//
// 这个模板封装了矩阵乘法操作和后处理逻辑，提供了类似 GEMM 的接口，
// 可以在设备代码中实例化。该操作符协调 Warp 内所有线程共同完成矩阵乘法。

namespace cutlass {
namespace gemm {
namespace warp {

template <
  typename Shape,
  typename InstructionShape,
  typename ElementA,
  typename LayoutA,
  typename ElementB,
  typename LayoutB,
  typename ElementC,
  typename LayoutC,
  typename ElementScalar
>
class GemmTensorOp {
public:
  // 计算对齐后的 Warp 形状，确保维度是指令形状的整数倍
  using WarpShape = GemmShape<
    ((Shape::kM + InstructionShape::kM - 1) / InstructionShape::kM) * InstructionShape::kM,
    ((Shape::kN + InstructionShape::kN - 1) / InstructionShape::kN) * InstructionShape::kN,
    InstructionShape::kK
  >;

  // 定义 Warp 级矩阵乘法操作，使用 Tensor Core 指令
  using MmaWarp = typename cutlass::gemm::warp::DefaultMmaTensorOp<
    WarpShape,
    InstructionShape,
    double,                             // A 矩阵元素数据类型
    cutlass::layout::RowMajor,          // A 矩阵布局（行主序）
    double,                             // B 矩阵元素数据类型
    cutlass::layout::ColumnMajor,       // B 矩阵布局（列主序）
    double,                             // C 矩阵元素数据类型
    cutlass::layout::RowMajor           // C 矩阵布局（行主序）
  >::Type;
 
  // K 维度分组数：将 K 维度分割成多个指令大小的块 
  int const kKgroups = (Shape::kK + InstructionShape::kK - 1) / InstructionShape::kK;

  // Fragment 迭代器：用于遍历累加器的片段
  using FragmentIterator = cutlass::epilogue::warp::FragmentIteratorTensorOp<
    typename MmaWarp::Shape,
    InstructionShape,
    double,
    typename MmaWarp::Policy::Operator::FragmentC,
    cutlass::layout::RowMajor
  >;

  // 后处理 Tile 迭代器：用于遍历共享内存中的元素片段
  using AccumulatorTileIterator = cutlass::epilogue::warp::TileIteratorTensorOpCanonical<
    typename MmaWarp::Shape,
    InstructionShape,
    double,
    cutlass::layout::RowMajor
  >;

  using TensorRefA = typename MmaWarp::IteratorA::TensorRef;
  using TensorRefB = typename MmaWarp::IteratorB::TensorRef;
  using TensorRefC = typename AccumulatorTileIterator::TensorRef;

public:
  CUTLASS_HOST_DEVICE
  GemmTensorOp() { }

  CUTLASS_DEVICE
  void operator()(
    ElementScalar alpha, 
    TensorRefA ref_A, 
    TensorRefB ref_B, 
    ElementScalar beta,
    TensorRefC ref_C,
    TensorRefC ref_D,
    int lane_id) const {
  
    // 创建指向共享内存中 A 和 B 矩阵片段的迭代器
    typename MmaWarp::IteratorA iter_A(ref_A, {Shape::kM, Shape::kK}, lane_id);
    typename MmaWarp::IteratorB iter_B(ref_B, {Shape::kK, Shape::kN}, lane_id);

    // 创建并清零用于保存 C 矩阵的累加器 tile
    typename MmaWarp::FragmentC accum;
    accum.clear();
  
    // 实例化 Warp 级矩阵乘法操作符
    MmaWarp mma_op;

    // 创建用于保存每个 Warp 持有的矩阵片段的 Fragment
    // 使用双缓冲技术（[2]）实现计算与数据加载的重叠
    typename MmaWarp::FragmentA frag_A[2];
    typename MmaWarp::FragmentB frag_B[2];
      
    // 预加载第一个 K 分组的数据片段
    iter_A.load(frag_A[0]);
    iter_B.load(frag_B[0]);

    // 移动到下一个 K 分组
    ++iter_A;
    ++iter_B;

    // 主循环：遍历所有 K 分组进行矩阵乘法
    CUTLASS_PRAGMA_UNROLL
    for (int k = 0; k < kKgroups; ++k) {

      // 预加载下一个 K 分组的数据（双缓冲）
      iter_A.load(frag_A[(k + 1) % 2]);
      iter_B.load(frag_B[(k + 1) % 2]);

      ++iter_A;
      ++iter_B;

      // 执行当前 K 分组的矩阵乘法，累加到 accum
      mma_op(accum, frag_A[k % 2], frag_B[k % 2], accum);
    }
  
    // 创建后处理阶段的迭代器
    FragmentIterator accum_frag_it(accum);
    AccumulatorTileIterator source_tile_it(ref_C, {Shape::kM, Shape::kN}, lane_id);
    AccumulatorTileIterator dest_tile_it(ref_D, {Shape::kM, Shape::kN}, lane_id);

    // 定义线性缩放操作的函数对象
    cutlass::multiplies<typename FragmentIterator::Fragment> mul_source;
    cutlass::multiply_add<typename FragmentIterator::Fragment> mul_add_accumulator;

    // 遍历后处理组件，应用 alpha 和 beta 缩放
    CUTLASS_PRAGMA_UNROLL
    for (int idx = 0; idx < FragmentIterator::kIterations; ++idx) {

      // 定义累加器片段的存储空间
      typename FragmentIterator::Fragment accum_fragment;
      typename FragmentIterator::Fragment source_fragment;

      // 从累加器 tile 中选择一个片段
      accum_frag_it.load(accum_fragment);
      ++accum_frag_it;

      // 从共享内存加载对应的源数据片段（C 矩阵）
      source_tile_it.load(source_fragment);
      ++source_tile_it;

      // 计算线性组合：D = alpha * AB + beta * C
      source_fragment = mul_source(beta, source_fragment);
      accum_fragment = mul_add_accumulator(alpha, accum_fragment, source_fragment);

      // 将结果存储回共享内存
      dest_tile_it.store(accum_fragment);
      ++dest_tile_it;
    }
  }
};

} // namespace warp
} // namespace gemm
} // namespace cutlass

///////////////////////////////////////////////////////////////////////////////////////////////////

// 示例核函数：演示 Warp 对共享内存中矩阵的协同 GEMM 操作
// 这个核函数展示了完整的 Tensor Core GEMM 流程：
// 1. 从全局内存加载数据到共享内存
// 2. 执行 Warp 级矩阵乘法
// 3. 将结果写回全局内存
__global__ void kernel(
  double *D_gmem, 
  double alpha, 
  double const *A_gmem, 
  double const *B_gmem, 
  double beta,
  double const *C_gmem) {

  // 在共享内存中定义矩阵
  // 注意：这些矩阵的布局已经针对 Tensor Core 操作优化
  __shared__ double A[kM][kK];
  __shared__ double B[kN][kK];
  __shared__ double C[kM][kN];

  // 将数据从全局内存复制到共享内存
  // 只有线程 0 执行复制操作，避免冲突
  if (threadIdx.x == 0) {
    CUTLASS_PRAGMA_NO_UNROLL
    for (int m = 0; m < kM; ++m) {
      for (int k = 0; k < kK; ++k) {
        A[m][k] = A_gmem[m * kK + k];
      }
    }
    CUTLASS_PRAGMA_NO_UNROLL
    for (int n = 0; n < kN; ++n) {
      for (int k = 0; k < kK; ++k) {
        B[n][k] = B_gmem[n * kK + k];
      }
    }
    CUTLASS_PRAGMA_NO_UNROLL
    for (int m = 0; m < kM; ++m) {
      CUTLASS_PRAGMA_NO_UNROLL
      for (int n = 0; n < kN; ++n) {
        C[m][n] = C_gmem[m * kN + n];
      }
    }
  }

  __syncthreads();
  
  // 实例化 Warp 级矩阵乘法操作符
  // 参数说明：
  // - 指令形状 (8x8x4)：Tensor Core 的基本操作单元
  // - 整体形状 (kM, kN, kK)：完整的矩阵维度
  // - 数据类型：所有矩阵使用 double 精度
  // - 布局：A 行主序，B 列主序，C 行主序

  using GemmTensorOp = cutlass::gemm::warp::GemmTensorOp<
    cutlass::gemm::GemmShape<kM, kN, kK>,
    cutlass::gemm::GemmShape<8, 8, 4>,
    double,                             // A 矩阵元素类型
    cutlass::layout::RowMajor,          // A 矩阵布局（行主序）
    double,                             // B 矩阵元素类型
    cutlass::layout::ColumnMajor,       // B 矩阵布局（列主序）
    double,                             // C 矩阵元素类型
    cutlass::layout::RowMajor,          // C 矩阵布局（行主序）
    double                              // alpha 和 beta 标量类型
  >;

  // 实例化 GEMM 操作符
  GemmTensorOp gemm;

  // 执行 Warp 级 GEMM 操作
  // D = alpha * A * B + beta * C
  // threadIdx.x 作为 lane_id 传入，用于确定每个线程的职责
  gemm(
    alpha, 
    {&A[0][0], kK},
    {&B[0][0], kK},
    beta,
    {&C[0][0], kN},
    {&C[0][0], kN},
    threadIdx.x);

  __syncthreads();

  // 将结果从共享内存复制回全局内存
  // 只有线程 0 执行复制操作
  if (threadIdx.x == 0) {
    CUTLASS_PRAGMA_NO_UNROLL
    for (int m = 0; m < kM; ++m) {
      CUTLASS_PRAGMA_NO_UNROLL
      for (int n = 0; n < kN; ++n) {
        D_gmem[m * kN + n] = C[m][n];
      }
    }
  }
}

///////////////////////////////////////////////////////////////////////////////////////////////////

/// 程序入口点：规范化 Warp 级 GEMM 操作演示
int main(int argc, const char *arg[]) {

  bool notSupported = false;

  // 检查 CUDA 版本：CUTLASS 需要 CUDA 11 工具包来运行这些示例
  if (!(__CUDACC_VER_MAJOR__ >= 11)) {
    std::cerr << "NVIDIA Ampere Tensor Core operations must be compiled with CUDA 11.0 Toolkit or later." << std::endl;
    notSupported = true;
  }

  hipDeviceProp_t props;

  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (!((props.major * 10 + props.minor) >= 80)) {
    std::cerr << "This example requires compute capability at least 80."
              << std::endl;
    notSupported = true;
  }

  if (notSupported) {
    // 在不支持的平台上返回 0，使测试通过
    return 0;
  }

  cutlass::HostTensor<double, cutlass::layout::RowMajor> A({kM, kK});
  cutlass::HostTensor<double, cutlass::layout::ColumnMajor> B({kK, kN});
  cutlass::HostTensor<double, cutlass::layout::RowMajor> C({kM, kN});
  cutlass::HostTensor<double, cutlass::layout::RowMajor> D({kM, kN});

  uint64_t seed = 2020;
  double max = 8;
  double min = -8;

  cutlass::reference::host::TensorFillRandomUniform(
    A.host_view(),
    seed,
    max,
    min,
    0
  );

  cutlass::reference::host::TensorFillRandomUniform(
    B.host_view(),
    seed + 17,
    max,
    min,
    0
  );

  cutlass::reference::host::TensorFillRandomUniform(
    C.host_view(),
    seed + 31,
    max,
    min,
    0
  );

  A.sync_device();
  B.sync_device();
  C.sync_device();
  D.sync_device();

  dim3 grid(1,1);
  dim3 block(32, 1, 1);

  double alpha = 2.25;
  double beta = 1.24;

  kernel<<< grid, block >>>(
    D.device_data(),
    alpha,
    A.device_data(),
    B.device_data(),
    beta,
    C.device_data()
  );

  hipError_t result = hipDeviceSynchronize();
  if (result != hipSuccess) {
    std::cerr << "Failed to synchronize device after kernel launch." << std::endl;
    return -1;
  }

  D.sync_host();
  
  // 在主机端计算参考结果，用于验证 GPU 计算的正确性
  cutlass::HostTensor<double, cutlass::layout::RowMajor> D_ref({kM, kN}, false);

  cutlass::reference::host::GemmComplex(
    {kM, kN, kK},
    alpha,
    A.host_ref(),
    cutlass::ComplexTransform::kNone,
    B.host_ref(),
    cutlass::ComplexTransform::kNone,
    beta,
    C.host_ref(),
    D_ref.host_ref(),
    double()
  );

  // 验证 GPU 计算结果是否与参考结果匹配
  if (!cutlass::reference::host::TensorEquals(
    D.host_view(),
    D_ref.host_view())) {

    std::cerr 
      << "A =\n" << A.host_view() 
      << "\n\nB = \n" << B.host_view() 
      << "\n\nC = " << C.host_view() 
      << "\n\nRef =\n" << D_ref.host_view()
      << "\n\nD =\n" << D.host_view() << "\n\n";

    std::cerr << "Error - device results mismatch host reference." << std::endl;

    return -1;
  }

  std::cout << "Passed" << std::endl;

  return 0;
}

///////////////////////////////////////////////////////////////////////////////////////////////////
