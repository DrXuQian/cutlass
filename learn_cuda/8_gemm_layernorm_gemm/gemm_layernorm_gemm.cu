#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Warp-level reduction for sum
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Fused GEMM + LayerNorm + GEMM kernel
template<int TILE_SIZE>
__global__ void gemmLayerNormGemm(
    const float* A, const float* B1, const float* B2,
    const float* gamma, const float* beta,
    float* C,
    int M, int N1, int K1, int N2) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    __shared__ float intermediate[TILE_SIZE][TILE_SIZE];
    __shared__ float row_mean[TILE_SIZE];
    __shared__ float row_variance[TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    // Phase 1: First GEMM (A * B1)
    float sum1 = 0.0f;
    for (int k = 0; k < K1; k += TILE_SIZE) {
        if (row < M && k + threadIdx.x < K1) {
            sA[threadIdx.y][threadIdx.x] = A[row * K1 + k + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N1 && k + threadIdx.y < K1) {
            sB[threadIdx.y][threadIdx.x] = B1[(k + threadIdx.y) * N1 + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum1 += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Store first GEMM result
    intermediate[threadIdx.y][threadIdx.x] = sum1;
    __syncthreads();

    // Phase 2: LayerNorm on intermediate result
    // Compute row mean
    float val = (row < M && col < N1) ? sum1 : 0.0f;
    float row_sum = val;

    if (threadIdx.x < 32) {
        row_sum = warpReduceSum(row_sum);
    }

    if (threadIdx.x == 0) {
        row_mean[threadIdx.y] = row_sum / N1;
    }
    __syncthreads();

    float mean = row_mean[threadIdx.y];

    // Compute row variance
    float diff = (row < M && col < N1) ? (val - mean) : 0.0f;
    float diff_sq = diff * diff;

    if (threadIdx.x < 32) {
        diff_sq = warpReduceSum(diff_sq);
    }

    if (threadIdx.x == 0) {
        row_variance[threadIdx.y] = diff_sq / N1;
    }
    __syncthreads();

    float variance = row_variance[threadIdx.y];
    float std_dev = sqrtf(variance + 1e-5f);

    // Apply LayerNorm
    if (row < M && col < N1) {
        float normalized = (intermediate[threadIdx.y][threadIdx.x] - mean) / std_dev;
        // Apply affine transformation if gamma and beta are provided
        if (gamma && beta) {
            normalized = gamma[col] * normalized + beta[col];
        }
        intermediate[threadIdx.y][threadIdx.x] = normalized;
    }
    __syncthreads();

    // Phase 3: Second GEMM (normalized * B2)
    float sum2 = 0.0f;
    col = blockIdx.x * TILE_SIZE + threadIdx.x; // Reset col for second GEMM

    for (int k = 0; k < N1; k += TILE_SIZE) {
        // Load from intermediate result
        if (k / TILE_SIZE == blockIdx.x) {
            sA[threadIdx.y][threadIdx.x] = intermediate[threadIdx.y][threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N2 && k + threadIdx.y < N1) {
            sB[threadIdx.y][threadIdx.x] = B2[(k + threadIdx.y) * N2 + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum2 += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Write final result
    if (row < M && col < N2) {
        atomicAdd(&C[row * N2 + col], sum2);
    }
}

int main() {
    const int M = 32, K1 = 64, N1 = 64, N2 = 32;
    const int TILE_SIZE = 16;

    // Allocate matrices
    std::vector<float> h_A(M * K1);
    std::vector<float> h_B1(K1 * N1);
    std::vector<float> h_B2(N1 * N2);
    std::vector<float> h_gamma(N1, 1.0f);
    std::vector<float> h_beta(N1, 0.0f);
    std::vector<float> h_C(M * N2, 0.0f);

    // Initialize with small values
    for (int i = 0; i < M * K1; ++i) {
        h_A[i] = (rand() % 100) / 100.0f;
    }
    for (int i = 0; i < K1 * N1; ++i) {
        h_B1[i] = (rand() % 100) / 100.0f;
    }
    for (int i = 0; i < N1 * N2; ++i) {
        h_B2[i] = (rand() % 100) / 100.0f;
    }

    float *d_A, *d_B1, *d_B2, *d_gamma, *d_beta, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K1 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B1, K1 * N1 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B2, N1 * N2 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_gamma, N1 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_beta, N1 * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N2 * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), M * K1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B1, h_B1.data(), K1 * N1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B2, h_B2.data(), N1 * N2 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_gamma, h_gamma.data(), N1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_beta, h_beta.data(), N1 * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_C, 0, M * N2 * sizeof(float)));

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(
        (std::max(N1, N2) + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE
    );

    std::cout << "Running Fused GEMM + LayerNorm + GEMM..." << std::endl;
    std::cout << "Dimensions: A(" << M << "x" << K1 << ") * B1(" << K1 << "x" << N1
              << ") -> LayerNorm -> * B2(" << N1 << "x" << N2 << ")" << std::endl;

    gemmLayerNormGemm<TILE_SIZE><<<gridDim, blockDim>>>(
        d_A, d_B1, d_B2, d_gamma, d_beta, d_C,
        M, N1, K1, N2
    );
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, M * N2 * sizeof(float), hipMemcpyDeviceToHost));

    // Verify output shape and basic statistics
    float min_val = h_C[0], max_val = h_C[0], avg_val = 0.0f;
    for (int i = 0; i < M * N2; ++i) {
        min_val = std::min(min_val, h_C[i]);
        max_val = std::max(max_val, h_C[i]);
        avg_val += h_C[i];
    }
    avg_val /= (M * N2);

    std::cout << "Output statistics:" << std::endl;
    std::cout << "  Min: " << min_val << std::endl;
    std::cout << "  Max: " << max_val << std::endl;
    std::cout << "  Avg: " << avg_val << std::endl;
    std::cout << "  Shape: (" << M << ", " << N2 << ")" << std::endl;

    // Basic sanity check
    bool has_output = false;
    for (int i = 0; i < M * N2; ++i) {
        if (h_C[i] != 0.0f) {
            has_output = true;
            break;
        }
    }

    std::cout << "GEMM + LayerNorm + GEMM: " << (has_output ? "PASS" : "FAIL") << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B1));
    CHECK_CUDA(hipFree(d_B2));
    CHECK_CUDA(hipFree(d_gamma));
    CHECK_CUDA(hipFree(d_beta));
    CHECK_CUDA(hipFree(d_C));

    return 0;
}