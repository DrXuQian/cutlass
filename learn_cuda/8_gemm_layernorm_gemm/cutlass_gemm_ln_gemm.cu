#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Simple LayerNorm kernel
template <typename Element>
__global__ void applyLayerNorm(
    Element* matrix,
    const Element* gamma,
    const Element* beta,
    int M, int N) {

    int row = blockIdx.x;
    if (row >= M) return;

    // Compute mean
    float sum = 0.0f;
    for (int col = 0; col < N; ++col) {
        sum += float(matrix[row * N + col]);
    }
    float mean = sum / N;

    // Compute variance
    float variance_sum = 0.0f;
    for (int col = 0; col < N; ++col) {
        float diff = float(matrix[row * N + col]) - mean;
        variance_sum += diff * diff;
    }
    float variance = variance_sum / N;
    float std_dev = sqrtf(variance + 1e-5f);

    // Apply normalization with gamma and beta
    for (int col = 0; col < N; ++col) {
        float normalized = (float(matrix[row * N + col]) - mean) / std_dev;
        if (gamma && beta) {
            normalized = float(gamma[col]) * normalized + float(beta[col]);
        }
        matrix[row * N + col] = Element(normalized);
    }
}

int main() {
    const int M = 32, K1 = 64, N1 = 64, N2 = 32;

    // Define CUTLASS GEMM with TensorCore
    using ElementInput = cutlass::half_t;
    using ElementOutput = cutlass::half_t;
    using ElementAccumulator = float;
    using ElementCompute = float;

    using Gemm = cutlass::gemm::device::Gemm<
        ElementInput, cutlass::layout::RowMajor,
        ElementInput, cutlass::layout::RowMajor,
        ElementOutput, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm75,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 8>,
        cutlass::epilogue::thread::LinearCombination<
            ElementOutput,
            128 / cutlass::sizeof_bits<ElementOutput>::value,
            ElementAccumulator,
            ElementCompute
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        2
    >;

    // Allocate host matrices
    std::vector<cutlass::half_t> h_A(M * K1);
    std::vector<cutlass::half_t> h_B1(K1 * N1);
    std::vector<cutlass::half_t> h_B2(N1 * N2);
    std::vector<cutlass::half_t> h_gamma(N1, cutlass::half_t(1.0f));
    std::vector<cutlass::half_t> h_beta(N1, cutlass::half_t(0.0f));
    std::vector<cutlass::half_t> h_intermediate(M * N1, cutlass::half_t(0));
    std::vector<cutlass::half_t> h_output(M * N2, cutlass::half_t(0));

    // Initialize matrices
    for (int i = 0; i < M * K1; ++i) {
        h_A[i] = cutlass::half_t((rand() % 100) / 100.0f);
    }
    for (int i = 0; i < K1 * N1; ++i) {
        h_B1[i] = cutlass::half_t((rand() % 100) / 100.0f);
    }
    for (int i = 0; i < N1 * N2; ++i) {
        h_B2[i] = cutlass::half_t((rand() % 100) / 100.0f);
    }

    // Allocate device memory
    cutlass::half_t *d_A, *d_B1, *d_B2, *d_gamma, *d_beta;
    cutlass::half_t *d_intermediate, *d_output, *d_temp;

    CHECK_CUDA(hipMalloc(&d_A, M * K1 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_B1, K1 * N1 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_B2, N1 * N2 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_gamma, N1 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_beta, N1 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_intermediate, M * N1 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_output, M * N2 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMalloc(&d_temp, M * N1 * sizeof(cutlass::half_t)));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), M * K1 * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B1, h_B1.data(), K1 * N1 * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B2, h_B2.data(), N1 * N2 * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_gamma, h_gamma.data(), N1 * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_beta, h_beta.data(), N1 * sizeof(cutlass::half_t), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_intermediate, 0, M * N1 * sizeof(cutlass::half_t)));
    CHECK_CUDA(hipMemset(d_temp, 0, M * N1 * sizeof(cutlass::half_t)));

    std::cout << "Running CUTLASS TensorCore GEMM + LayerNorm + GEMM..." << std::endl;
    std::cout << "Dimensions: A(" << M << "x" << K1 << ") * B1(" << K1 << "x" << N1
              << ") -> LayerNorm -> * B2(" << N1 << "x" << N2 << ")" << std::endl;

    // Phase 1: First GEMM (A * B1)
    Gemm gemm_op;
    Gemm::Arguments args1(
        {M, N1, K1},
        {d_A, K1},
        {d_B1, N1},
        {d_temp, N1},
        {d_intermediate, N1},
        {ElementCompute(1.0f), ElementCompute(0.0f)}
    );

    cutlass::Status status = gemm_op(args1);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "First GEMM execution failed" << std::endl;
        return -1;
    }

    // Phase 2: Apply LayerNorm
    applyLayerNorm<<<M, 1>>>(d_intermediate, d_gamma, d_beta, M, N1);

    // Phase 3: Second GEMM (normalized * B2)
    Gemm::Arguments args2(
        {M, N2, N1},
        {d_intermediate, N1},
        {d_B2, N2},
        {d_temp, N2},
        {d_output, N2},
        {ElementCompute(1.0f), ElementCompute(0.0f)}
    );

    status = gemm_op(args2);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Second GEMM execution failed" << std::endl;
        return -1;
    }

    CHECK_CUDA(hipDeviceSynchronize());

    // Copy results back
    CHECK_CUDA(hipMemcpy(h_output.data(), d_output, M * N2 * sizeof(cutlass::half_t), hipMemcpyDeviceToHost));

    // Verify output statistics
    float min_val = float(h_output[0]), max_val = float(h_output[0]), avg_val = 0.0f;
    for (int i = 0; i < M * N2; ++i) {
        float val = float(h_output[i]);
        min_val = std::min(min_val, val);
        max_val = std::max(max_val, val);
        avg_val += val;
    }
    avg_val /= (M * N2);

    std::cout << "Output statistics:" << std::endl;
    std::cout << "  Min: " << min_val << std::endl;
    std::cout << "  Max: " << max_val << std::endl;
    std::cout << "  Avg: " << avg_val << std::endl;
    std::cout << "  Shape: (" << M << ", " << N2 << ")" << std::endl;

    // Basic sanity check
    bool has_output = false;
    for (int i = 0; i < M * N2; ++i) {
        if (float(h_output[i]) != 0.0f) {
            has_output = true;
            break;
        }
    }

    std::cout << "CUTLASS TensorCore GEMM + LayerNorm + GEMM: " << (has_output ? "PASS" : "FAIL") << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B1));
    CHECK_CUDA(hipFree(d_B2));
    CHECK_CUDA(hipFree(d_gamma));
    CHECK_CUDA(hipFree(d_beta));
    CHECK_CUDA(hipFree(d_intermediate));
    CHECK_CUDA(hipFree(d_output));
    CHECK_CUDA(hipFree(d_temp));

    return 0;
}