#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/reduction/device/reduce_split_k.h>
#include <cutlass/reduction/thread/reduction_operators.h>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Custom epilogue for GEMM with reduction
template <typename ElementC>
__global__ void reduceRows(
    ElementC const* matrix,
    ElementC* row_sums,
    int M, int N) {

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        ElementC sum = ElementC(0);
        for (int col = 0; col < N; ++col) {
            sum += matrix[row * N + col];
        }
        row_sums[row] = sum;
    }
}

int main() {
    const int M = 128, N = 128, K = 128;

    // Define CUTLASS GEMM with TensorCore
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;
    using ElementCompute = float;

    using Gemm = cutlass::gemm::device::Gemm<
        ElementA, cutlass::layout::RowMajor,
        ElementB, cutlass::layout::RowMajor,
        ElementC, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm75,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 8>,
        cutlass::epilogue::thread::LinearCombination<
            ElementC,
            128 / cutlass::sizeof_bits<ElementC>::value,
            ElementAccumulator,
            ElementCompute
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        2
    >;

    // Allocate host matrices
    std::vector<cutlass::half_t> h_A(M * K);
    std::vector<cutlass::half_t> h_B(K * N);
    std::vector<cutlass::half_t> h_C(M * N, cutlass::half_t(0));
    std::vector<cutlass::half_t> h_D(M * N, cutlass::half_t(0));
    std::vector<cutlass::half_t> h_row_sums(M, cutlass::half_t(0));

    // Initialize matrices
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = cutlass::half_t(1.0f);
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = cutlass::half_t(1.0f);
    }

    // Allocate device memory
    cutlass::half_t *d_A, *d_B, *d_C, *d_D, *d_row_sums;
    size_t size_A = M * K * sizeof(cutlass::half_t);
    size_t size_B = K * N * sizeof(cutlass::half_t);
    size_t size_C = M * N * sizeof(cutlass::half_t);
    size_t size_row_sums = M * sizeof(cutlass::half_t);

    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));
    CHECK_CUDA(hipMalloc(&d_D, size_C));
    CHECK_CUDA(hipMalloc(&d_row_sums, size_row_sums));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C.data(), size_C, hipMemcpyHostToDevice));

    // Phase 1: Run TensorCore GEMM
    Gemm gemm_op;
    Gemm::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D, N},
        {ElementCompute(1.0f), ElementCompute(0.0f)}
    );

    cutlass::Status status = gemm_op(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM execution failed" << std::endl;
        return -1;
    }

    // Phase 2: Fused row reduction
    dim3 blockDim(256);
    dim3 gridDim((M + blockDim.x - 1) / blockDim.x);
    reduceRows<<<gridDim, blockDim>>>(d_D, d_row_sums, M, N);

    CHECK_CUDA(hipDeviceSynchronize());

    // Copy results back
    CHECK_CUDA(hipMemcpy(h_D.data(), d_D, size_C, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_row_sums.data(), d_row_sums, size_row_sums, hipMemcpyDeviceToHost));

    // Verify GEMM result
    bool gemm_correct = true;
    float expected_gemm = float(K);
    for (int i = 0; i < 10; ++i) {
        float val = float(h_D[i]);
        if (std::abs(val - expected_gemm) > 1.0f) {
            gemm_correct = false;
            break;
        }
    }

    // Verify row sums
    bool sum_correct = true;
    float expected_sum = N * K;
    for (int i = 0; i < M; ++i) {
        float val = float(h_row_sums[i]);
        if (std::abs(val - expected_sum) > 10.0f) {
            sum_correct = false;
            break;
        }
    }

    std::cout << "CUTLASS TensorCore GEMM: " << (gemm_correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "GEMM expected: " << expected_gemm << ", Got: " << float(h_D[0]) << std::endl;

    std::cout << "Row Reduction: " << (sum_correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Row sum expected: " << expected_sum << ", Got: " << float(h_row_sums[0]) << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_D));
    CHECK_CUDA(hipFree(d_row_sums));

    return 0;
}