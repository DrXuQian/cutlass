#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Fused GEMM + Row Reduction kernel
template<int TILE_SIZE>
__global__ void gemmRowReduction(
    const float* A, const float* B,
    float* C, float* row_sums,
    int M, int N, int K) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_row_sums[TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Compute GEMM
    for (int k = 0; k < K; k += TILE_SIZE) {
        if (row < M && k + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && k + threadIdx.y < K) {
            sB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Store GEMM result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }

    // Initialize shared memory for row reduction
    if (threadIdx.x == 0) {
        tile_row_sums[threadIdx.y] = 0.0f;
    }
    __syncthreads();

    // Accumulate row sum within tile
    if (row < M && col < N) {
        atomicAdd(&tile_row_sums[threadIdx.y], sum);
    }
    __syncthreads();

    // Write row sums (one thread per row in tile)
    if (threadIdx.x == 0 && row < M) {
        atomicAdd(&row_sums[row], tile_row_sums[threadIdx.y]);
    }
}

// Fused GEMM + Column Maximum kernel
template<int TILE_SIZE>
__global__ void gemmColMax(
    const float* A, const float* B,
    float* C, float* col_max,
    int M, int N, int K) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_col_max[TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Compute GEMM
    for (int k = 0; k < K; k += TILE_SIZE) {
        if (row < M && k + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && k + threadIdx.y < K) {
            sB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Store GEMM result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }

    // Initialize shared memory for column max
    if (threadIdx.y == 0) {
        tile_col_max[threadIdx.x] = -INFINITY;
    }
    __syncthreads();

    // Find column maximum within tile
    if (row < M && col < N) {
        atomicMax(reinterpret_cast<int*>(&tile_col_max[threadIdx.x]),
                  __float_as_int(sum));
    }
    __syncthreads();

    // Write column max (one thread per column in tile)
    if (threadIdx.y == 0 && col < N) {
        atomicMax(reinterpret_cast<int*>(&col_max[col]),
                  __float_as_int(tile_col_max[threadIdx.x]));
    }
}

int main() {
    const int M = 128, N = 128, K = 128;
    const int TILE_SIZE = 16;

    // Allocate matrices
    std::vector<float> h_A(M * K, 1.0f);
    std::vector<float> h_B(K * N, 1.0f);
    std::vector<float> h_C(M * N, 0.0f);
    std::vector<float> h_row_sums(M, 0.0f);
    std::vector<float> h_col_max(N, -INFINITY);

    float *d_A, *d_B, *d_C, *d_row_sums, *d_col_max;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_row_sums, M * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_col_max, N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_row_sums, 0, M * sizeof(float)));

    // Initialize col_max with -infinity
    std::vector<float> neg_inf(N, -INFINITY);
    CHECK_CUDA(hipMemcpy(d_col_max, neg_inf.data(), N * sizeof(float), hipMemcpyHostToDevice));

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(
        (N + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE
    );

    // Test GEMM + Row Reduction
    std::cout << "Testing GEMM + Row Reduction..." << std::endl;
    gemmRowReduction<TILE_SIZE><<<gridDim, blockDim>>>(
        d_A, d_B, d_C, d_row_sums, M, N, K
    );
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_row_sums.data(), d_row_sums, M * sizeof(float), hipMemcpyDeviceToHost));

    // Verify row sums (each row sum should be N * K)
    bool row_correct = true;
    float expected_row_sum = N * K;
    for (int i = 0; i < M; ++i) {
        if (std::abs(h_row_sums[i] - expected_row_sum) > 1e-3) {
            row_correct = false;
            break;
        }
    }

    std::cout << "GEMM + Row Reduction: " << (row_correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Expected row sum: " << expected_row_sum << ", Got: " << h_row_sums[0] << std::endl;

    // Test GEMM + Column Max
    std::cout << "\nTesting GEMM + Column Max..." << std::endl;
    CHECK_CUDA(hipMemset(d_C, 0, M * N * sizeof(float)));

    gemmColMax<TILE_SIZE><<<gridDim, blockDim>>>(
        d_A, d_B, d_C, d_col_max, M, N, K
    );
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_col_max.data(), d_col_max, N * sizeof(float), hipMemcpyDeviceToHost));

    // Verify column max (each should be K)
    bool col_correct = true;
    float expected_col_max = K;
    for (int i = 0; i < N; ++i) {
        if (std::abs(h_col_max[i] - expected_col_max) > 1e-3) {
            col_correct = false;
            break;
        }
    }

    std::cout << "GEMM + Column Max: " << (col_correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Expected column max: " << expected_col_max << ", Got: " << h_col_max[0] << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_row_sums));
    CHECK_CUDA(hipFree(d_col_max));

    return 0;
}