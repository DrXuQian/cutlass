#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Fused GEMM + ReLU kernel
template<int TILE_SIZE>
__global__ void gemmReLU(
    const float* A, const float* B, float* C,
    int M, int N, int K,
    float alpha, float beta) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Compute GEMM
    for (int k = 0; k < K; k += TILE_SIZE) {
        // Load tiles
        if (row < M && k + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && k + threadIdx.y < K) {
            sB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute dot product
        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Apply scaling and bias, then ReLU activation
    if (row < M && col < N) {
        float result = alpha * sum + beta * C[row * N + col];
        // Fused ReLU activation
        C[row * N + col] = fmaxf(result, 0.0f);
    }
}

// Fused GEMM + GEMM kernel (two consecutive GEMMs)
template<int TILE_SIZE>
__global__ void fusedDoubleGemm(
    const float* A, const float* B1, const float* B2, float* C,
    int M, int N, int K1, int K2) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    __shared__ float intermediate[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    int tid_y = threadIdx.y;
    int tid_x = threadIdx.x;

    // First GEMM: A * B1 -> intermediate
    float sum1 = 0.0f;
    for (int k = 0; k < K1; k += TILE_SIZE) {
        if (row < M && k + tid_x < K1) {
            sA[tid_y][tid_x] = A[row * K1 + k + tid_x];
        } else {
            sA[tid_y][tid_x] = 0.0f;
        }

        if (col < K2 && k + tid_y < K1) {
            sB[tid_y][tid_x] = B1[(k + tid_y) * K2 + col];
        } else {
            sB[tid_y][tid_x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum1 += sA[tid_y][i] * sB[i][tid_x];
        }
        __syncthreads();
    }

    // Store intermediate result in shared memory
    intermediate[tid_y][tid_x] = sum1;
    __syncthreads();

    // Second GEMM: intermediate * B2 -> C
    float sum2 = 0.0f;
    for (int k = 0; k < K2; k += TILE_SIZE) {
        // Reuse intermediate result
        if (k == 0) {
            sA[tid_y][tid_x] = intermediate[tid_y][tid_x];
        } else {
            sA[tid_y][tid_x] = 0.0f;
        }

        if (col < N && k + tid_y < K2) {
            sB[tid_y][tid_x] = B2[(k + tid_y) * N + col];
        } else {
            sB[tid_y][tid_x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum2 += sA[tid_y][i] * sB[i][tid_x];
        }
        __syncthreads();
    }

    // Write final result
    if (row < M && col < N) {
        C[row * N + col] = sum2;
    }
}

int main() {
    const int M = 256, N = 256, K = 256;
    const int TILE_SIZE = 16;

    size_t size = M * N * sizeof(float);

    // Allocate matrices
    std::vector<float> h_A(M * K, 1.0f);
    std::vector<float> h_B(K * N, 2.0f);
    std::vector<float> h_C(M * N, 0.0f);

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, size));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_C, 0, size));

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(
        (N + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE
    );

    // Test GEMM + ReLU fusion
    std::cout << "Testing Fused GEMM + ReLU..." << std::endl;
    gemmReLU<TILE_SIZE><<<gridDim, blockDim>>>(
        d_A, d_B, d_C, M, N, K, 1.0f, 0.0f
    );
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, size, hipMemcpyDeviceToHost));

    // Verify (should be max(2*K, 0) = 512)
    bool correct = true;
    float expected = 2.0f * K;
    for (int i = 0; i < 10; ++i) {
        if (std::abs(h_C[i] - expected) > 1e-3) {
            correct = false;
            break;
        }
    }

    std::cout << "GEMM + ReLU: " << (correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Expected: " << expected << ", Got: " << h_C[0] << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    return 0;
}