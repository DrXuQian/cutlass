#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/epilogue/thread/linear_combination_relu.h>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

int main() {
    const int M = 256, N = 256, K = 256;

    // Define CUTLASS GEMM with ReLU fusion using TensorCore
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;
    using ElementCompute = float;

    // GEMM with fused ReLU epilogue
    using GemmReLU = cutlass::gemm::device::Gemm<
        ElementA, cutlass::layout::RowMajor,
        ElementB, cutlass::layout::RowMajor,
        ElementC, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm75,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 8>,
        cutlass::epilogue::thread::LinearCombinationRelu<
            ElementC,
            128 / cutlass::sizeof_bits<ElementC>::value,
            ElementAccumulator,
            ElementCompute
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        2
    >;

    // Allocate host matrices
    std::vector<cutlass::half_t> h_A(M * K);
    std::vector<cutlass::half_t> h_B(K * N);
    std::vector<cutlass::half_t> h_C(M * N, cutlass::half_t(0));
    std::vector<cutlass::half_t> h_D(M * N, cutlass::half_t(0));

    // Initialize matrices
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = cutlass::half_t(2.0f);
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = cutlass::half_t(1.0f);
    }

    // Allocate device memory
    cutlass::half_t *d_A, *d_B, *d_C, *d_D;
    size_t size_A = M * K * sizeof(cutlass::half_t);
    size_t size_B = K * N * sizeof(cutlass::half_t);
    size_t size_C = M * N * sizeof(cutlass::half_t);

    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));
    CHECK_CUDA(hipMalloc(&d_D, size_C));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C.data(), size_C, hipMemcpyHostToDevice));

    // Create GEMM+ReLU instance
    GemmReLU gemm_relu_op;

    // Configure arguments
    GemmReLU::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D, N},
        {ElementCompute(1.0f), ElementCompute(0.0f)}
    );

    // Run GEMM+ReLU
    cutlass::Status status = gemm_relu_op(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM+ReLU execution failed" << std::endl;
        return -1;
    }

    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result back
    CHECK_CUDA(hipMemcpy(h_D.data(), d_D, size_C, hipMemcpyDeviceToHost));

    // Verify result (should be max(2*K, 0) = 512)
    bool correct = true;
    float expected = 2.0f * K;
    for (int i = 0; i < 10; ++i) {
        float val = float(h_D[i]);
        if (std::abs(val - expected) > 1.0f) {
            correct = false;
            break;
        }
    }

    std::cout << "CUTLASS TensorCore GEMM+ReLU: " << (correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Expected: " << expected << ", Got: " << float(h_D[0]) << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_D));

    return 0;
}