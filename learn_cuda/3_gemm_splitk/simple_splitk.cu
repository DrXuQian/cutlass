#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Simple Split-K GEMM kernel
template<int TILE_SIZE, int K_SPLIT>
__global__ void splitKGemm(
    const float* A, const float* B, float* C,
    int M, int N, int K) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    int split_id = blockIdx.z;

    // Calculate K range for this split
    int k_per_split = (K + K_SPLIT - 1) / K_SPLIT;
    int k_start = split_id * k_per_split;
    int k_end = min(k_start + k_per_split, K);

    float sum = 0.0f;

    // Process K tiles for this split
    for (int k = k_start; k < k_end; k += TILE_SIZE) {
        // Load tiles into shared memory
        if (row < M && k + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && k + threadIdx.y < K) {
            sB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute partial products
        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Atomic add to accumulate results from different splits
    if (row < M && col < N) {
        atomicAdd(&C[row * N + col], sum);
    }
}

int main() {
    const int M = 512, N = 512, K = 2048;
    const int K_SPLIT = 4;
    const int TILE_SIZE = 16;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    // Allocate and initialize matrices
    std::vector<float> h_A(M * K, 1.0f);
    std::vector<float> h_B(K * N, 1.0f);
    std::vector<float> h_C(M * N, 0.0f);

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_C, 0, size_C));

    // Launch kernel
    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(
        (N + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE,
        K_SPLIT
    );

    splitKGemm<TILE_SIZE, K_SPLIT><<<gridDim, blockDim>>>(
        d_A, d_B, d_C, M, N, K
    );

    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result back
    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, size_C, hipMemcpyDeviceToHost));

    // Verify result (all elements should be K)
    bool correct = true;
    for (int i = 0; i < M * N; ++i) {
        if (std::abs(h_C[i] - K) > 1e-3) {
            correct = false;
            break;
        }
    }

    std::cout << "Split-K GEMM: " << (correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Expected: " << K << ", Got: " << h_C[0] << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    return 0;
}