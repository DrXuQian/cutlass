#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm_splitk_parallel.h>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

int main() {
    const int M = 512, N = 512, K = 2048;
    const int split_k_slices = 4;

    // Define CUTLASS GEMM with Split-K and TensorCore
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;
    using ElementCompute = float;

    using Gemm = cutlass::gemm::device::GemmSplitKParallel<
        ElementA, cutlass::layout::RowMajor,
        ElementB, cutlass::layout::RowMajor,
        ElementC, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm75,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 8>,
        cutlass::epilogue::thread::LinearCombination<
            ElementC,
            128 / cutlass::sizeof_bits<ElementC>::value,
            ElementAccumulator,
            ElementCompute
        >
    >;

    // Allocate host matrices
    std::vector<cutlass::half_t> h_A(M * K);
    std::vector<cutlass::half_t> h_B(K * N);
    std::vector<cutlass::half_t> h_C(M * N, cutlass::half_t(0));
    std::vector<cutlass::half_t> h_D(M * N, cutlass::half_t(0));

    // Initialize matrices
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = cutlass::half_t(1.0f);
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = cutlass::half_t(1.0f);
    }

    // Allocate device memory
    cutlass::half_t *d_A, *d_B, *d_C, *d_D;
    size_t size_A = M * K * sizeof(cutlass::half_t);
    size_t size_B = K * N * sizeof(cutlass::half_t);
    size_t size_C = M * N * sizeof(cutlass::half_t);

    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));
    CHECK_CUDA(hipMalloc(&d_D, size_C));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C.data(), size_C, hipMemcpyHostToDevice));

    // Create GEMM instance
    Gemm gemm_op;

    // Configure problem and arguments
    Gemm::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D, N},
        {ElementCompute(1.0f), ElementCompute(0.0f)},
        split_k_slices
    );

    // Query workspace size
    size_t workspace_size = gemm_op.get_workspace_size(args);
    void* workspace = nullptr;
    if (workspace_size > 0) {
        CHECK_CUDA(hipMalloc(&workspace, workspace_size));
    }

    // Initialize and run
    cutlass::Status status = gemm_op.initialize(args, workspace);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM initialization failed" << std::endl;
        return -1;
    }

    status = gemm_op();
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM execution failed" << std::endl;
        return -1;
    }

    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result back
    CHECK_CUDA(hipMemcpy(h_D.data(), d_D, size_C, hipMemcpyDeviceToHost));

    // Verify result
    bool correct = true;
    float expected = float(K);
    for (int i = 0; i < M * N; ++i) {
        float val = float(h_D[i]);
        if (std::abs(val - expected) > 1.0f) {
            correct = false;
            break;
        }
    }

    std::cout << "CUTLASS Split-K TensorCore GEMM: " << (correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Expected: " << expected << ", Got: " << float(h_D[0]) << std::endl;
    std::cout << "Split-K slices: " << split_k_slices << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_D));
    if (workspace) {
        CHECK_CUDA(hipFree(workspace));
    }

    return 0;
}