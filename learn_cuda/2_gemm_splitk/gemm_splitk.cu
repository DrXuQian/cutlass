#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/gemm/device/gemm_splitk_parallel.h>
#include <cutlass/reduction/device/reduce_split_k.h>
#include <cutlass/reduction/thread/reduction_operators.h>

#include "../common/matrix_utils.h"
#include "../common/cpu_gemm.h"
#include "../common/cuda_timer.h"
#include "../common/cuda_utils.h"

// Helper function to print Split-K explanation
void printSplitKExplanation() {
    std::cout << "\n=== Split-K GEMM Explanation ===" << std::endl;
    std::cout << "Split-K divides the K dimension into multiple chunks:" << std::endl;
    std::cout << "- Regular GEMM: C = A(M×K) × B(K×N)" << std::endl;
    std::cout << "- Split-K GEMM: K is split into P parts" << std::endl;
    std::cout << "  - Each part computes: C_p = A(M×K/P) × B(K/P×N)" << std::endl;
    std::cout << "  - Final result: C = sum(C_p) for p=0 to P-1" << std::endl;
    std::cout << "Benefits:" << std::endl;
    std::cout << "  - Better parallelism for large K" << std::endl;
    std::cout << "  - Improved load balancing" << std::endl;
    std::cout << "  - Reduced shared memory pressure" << std::endl;
    std::cout << "================================\n" << std::endl;
}

int main(int argc, char** argv) {
    // Matrix dimensions - using large K for Split-K benefit
    int M = 1024;
    int N = 1024;
    int K = 8192;  // Large K dimension
    float alpha = 1.0f;
    float beta = 0.0f;

    // Split-K configuration
    int split_k_slices = 8;  // Number of K partitions

    // Allow command line arguments
    if (argc >= 4) {
        M = std::atoi(argv[1]);
        N = std::atoi(argv[2]);
        K = std::atoi(argv[3]);
    }
    if (argc >= 5) {
        split_k_slices = std::atoi(argv[4]);
    }

    std::cout << "CUTLASS Split-K GEMM Example" << std::endl;
    std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K << std::endl;
    std::cout << "Split-K slices: " << split_k_slices << std::endl;
    std::cout << "Each slice processes K/" << split_k_slices << " = "
              << K/split_k_slices << " elements" << std::endl;

    printSplitKExplanation();

    // Print device info
    cuda_utils::printDeviceInfo();

    // Allocate host memory
    float *h_A = cuda_utils::allocateHostMatrix<float>(M, K);
    float *h_B = cuda_utils::allocateHostMatrix<float>(K, N);
    float *h_C = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D_regular = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D_splitk = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D_ref = cuda_utils::allocateHostMatrix<float>(M, N);

    // Initialize matrices
    cuda_utils::initializeRandomMatrix(h_A, M, K, -1.0f, 1.0f, 42);
    cuda_utils::initializeRandomMatrix(h_B, K, N, -1.0f, 1.0f, 43);
    cuda_utils::initializeConstantMatrix(h_C, M, N, 0.0f);

    // Copy for reference
    for (int i = 0; i < M * N; ++i) {
        h_D_ref[i] = h_C[i];
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_D_regular, *d_D_splitk;
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_A, M, K));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_B, K, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_C, M, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_D_regular, M, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_D_splitk, M, N));

    // Copy to device
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_A, h_A, M, K));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_B, h_B, K, N));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_C, h_C, M, N));

    // Define CUTLASS GEMM types
    using RowMajor = cutlass::layout::RowMajor;

    // Regular GEMM (no Split-K)
    using CutlassGemmRegular = cutlass::gemm::device::Gemm<
        float, RowMajor,
        float, RowMajor,
        float, RowMajor,
        float,
        cutlass::arch::OpClassSimt,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 128, 8>,   // ThreadblockShape
        cutlass::gemm::GemmShape<32, 64, 8>,     // WarpShape
        cutlass::gemm::GemmShape<1, 1, 1>,       // InstructionShape
        cutlass::epilogue::thread::LinearCombination<
            float, 1, float, float
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        3  // Stages
    >;

    // Split-K GEMM
    using CutlassGemmSplitK = cutlass::gemm::device::GemmSplitKParallel<
        float, RowMajor,
        float, RowMajor,
        float, RowMajor,
        float,
        cutlass::arch::OpClassSimt,
        cutlass::arch::Sm80,
        cutlass::gemm::GemmShape<128, 128, 8>,   // ThreadblockShape - K must be 8 for Sm80
        cutlass::gemm::GemmShape<32, 64, 8>,     // WarpShape
        cutlass::gemm::GemmShape<1, 1, 1>,       // InstructionShape
        cutlass::epilogue::thread::LinearCombination<
            float, 1, float, float
        >
    >;

    cuda_utils::CudaTimer timer;
    const int num_iterations = 10;

    // ========================================
    // 1. Regular GEMM (baseline)
    // ========================================
    std::cout << "\n1. Regular GEMM (baseline):" << std::endl;

    CutlassGemmRegular gemm_regular;
    CutlassGemmRegular::Arguments args_regular(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D_regular, N},
        {alpha, beta}
    );

    // Warmup
    cutlass::Status status = gemm_regular(args_regular);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Regular GEMM failed!" << std::endl;
        return -1;
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Benchmark
    timer.start();
    for (int i = 0; i < num_iterations; ++i) {
        status = gemm_regular(args_regular);
    }
    CHECK_CUDA(hipDeviceSynchronize());
    timer.stop();

    float regular_time = timer.getElapsedTime() / num_iterations;
    double regular_gflops = cuda_utils::computeGFLOPS(M, N, K, regular_time);

    std::cout << "  Time: " << regular_time << " ms" << std::endl;
    std::cout << "  Performance: " << regular_gflops << " GFLOPS" << std::endl;

    // ========================================
    // 2. Split-K GEMM
    // ========================================
    std::cout << "\n2. Split-K GEMM (" << split_k_slices << " slices):" << std::endl;

    CutlassGemmSplitK gemm_splitk;
    CutlassGemmSplitK::Arguments args_splitk(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D_splitk, N},
        {alpha, beta},
        split_k_slices  // Split-K slices
    );

    // Warmup
    status = gemm_splitk(args_splitk);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "Split-K GEMM failed!" << std::endl;
        return -1;
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Benchmark
    timer.reset();
    timer.start();
    for (int i = 0; i < num_iterations; ++i) {
        status = gemm_splitk(args_splitk);
    }
    CHECK_CUDA(hipDeviceSynchronize());
    timer.stop();

    float splitk_time = timer.getElapsedTime() / num_iterations;
    double splitk_gflops = cuda_utils::computeGFLOPS(M, N, K, splitk_time);

    std::cout << "  Time: " << splitk_time << " ms" << std::endl;
    std::cout << "  Performance: " << splitk_gflops << " GFLOPS" << std::endl;
    std::cout << "  Speedup over regular: " << regular_time / splitk_time << "x" << std::endl;

    // ========================================
    // 3. Try different Split-K configurations
    // ========================================
    std::cout << "\n3. Performance with different Split-K slices:" << std::endl;
    std::cout << "Slices | Time (ms) | GFLOPS | Speedup" << std::endl;
    std::cout << "-------|-----------|--------|--------" << std::endl;

    std::vector<int> slice_counts = {1, 2, 4, 8, 16, 32};
    float best_time = regular_time;
    int best_slices = 1;

    for (int slices : slice_counts) {
        if (K % slices != 0) continue;  // Skip if K is not divisible

        CutlassGemmSplitK::Arguments args_test(
            {M, N, K},
            {d_A, K}, {d_B, N}, {d_C, N}, {d_D_splitk, N},
            {alpha, beta},
            slices
        );

        // Warmup
        gemm_splitk(args_test);
        CHECK_CUDA(hipDeviceSynchronize());

        // Benchmark
        timer.reset();
        timer.start();
        for (int i = 0; i < num_iterations; ++i) {
            gemm_splitk(args_test);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        timer.stop();

        float time = timer.getElapsedTime() / num_iterations;
        double gflops = cuda_utils::computeGFLOPS(M, N, K, time);
        float speedup = regular_time / time;

        std::cout << std::setw(6) << slices << " | "
                  << std::setw(9) << std::setprecision(4) << time << " | "
                  << std::setw(6) << std::setprecision(0) << gflops << " | "
                  << std::setw(6) << std::setprecision(3) << speedup << "x" << std::endl;

        if (time < best_time) {
            best_time = time;
            best_slices = slices;
        }
    }

    std::cout << "\nBest configuration: " << best_slices << " slices" << std::endl;

    // ========================================
    // 4. CPU Reference
    // ========================================
    std::cout << "\n4. CPU Reference:" << std::endl;
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gemm::gemm_cpu(M, N, K, alpha, h_A, K, h_B, N, beta, h_D_ref, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_diff = cpu_end - cpu_start;
    double cpu_time = cpu_diff.count() * 1000.0;
    double cpu_gflops = cuda_utils::computeGFLOPS(M, N, K, cpu_time);

    std::cout << "  Time: " << cpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << cpu_gflops << " GFLOPS" << std::endl;

    // ========================================
    // 5. Verification
    // ========================================
    std::cout << "\n=== Verification ===" << std::endl;

    // Copy results back
    CHECK_CUDA(cuda_utils::copyDeviceToHost(h_D_regular, d_D_regular, M, N));
    CHECK_CUDA(cuda_utils::copyDeviceToHost(h_D_splitk, d_D_splitk, M, N));

    bool regular_passed = cuda_utils::compareMatrices(h_D_regular, h_D_ref, M, N, 1e-3f, false);
    bool splitk_passed = cuda_utils::compareMatrices(h_D_splitk, h_D_ref, M, N, 1e-3f, false);

    std::cout << "Regular GEMM: " << (regular_passed ? "✓ PASSED" : "✗ FAILED") << std::endl;
    std::cout << "Split-K GEMM: " << (splitk_passed ? "✓ PASSED" : "✗ FAILED") << std::endl;

    // ========================================
    // 6. Performance Summary
    // ========================================
    std::cout << "\n=== Performance Summary ===" << std::endl;
    std::cout << "Configuration          | Time (ms) | GFLOPS  | vs CPU" << std::endl;
    std::cout << "-----------------------|-----------|---------|--------" << std::endl;
    std::cout << "CPU Reference          | " << std::setw(9) << cpu_time
              << " | " << std::setw(7) << std::setprecision(1) << std::fixed << cpu_gflops
              << " | 1.00x" << std::endl;
    std::cout << "Regular GEMM           | " << std::setw(9) << std::setprecision(3) << regular_time
              << " | " << std::setw(7) << std::setprecision(1) << regular_gflops
              << " | " << std::setprecision(1) << cpu_time/regular_time << "x" << std::endl;
    std::cout << "Split-K GEMM (best)    | " << std::setw(9) << std::setprecision(3) << best_time
              << " | " << std::setw(7) << std::setprecision(1) << cuda_utils::computeGFLOPS(M, N, K, best_time)
              << " | " << std::setprecision(1) << cpu_time/best_time << "x" << std::endl;

    // Split-K analysis
    std::cout << "\n=== Split-K Analysis ===" << std::endl;
    std::cout << "K dimension: " << K << std::endl;
    std::cout << "Optimal slices: " << best_slices << std::endl;
    std::cout << "Elements per slice: " << K/best_slices << std::endl;
    std::cout << "Split-K speedup: " << std::setprecision(2) << regular_time/best_time << "x over regular GEMM" << std::endl;

    if (best_slices > 1) {
        std::cout << "\n✓ Split-K provides performance benefit for this configuration!" << std::endl;
        std::cout << "  Large K dimension (" << K << ") benefits from parallel reduction." << std::endl;
    } else {
        std::cout << "\n✗ Regular GEMM is optimal for this configuration." << std::endl;
        std::cout << "  Consider increasing K dimension for Split-K benefits." << std::endl;
    }

    // Cleanup
    cuda_utils::freeHostMatrix(h_A);
    cuda_utils::freeHostMatrix(h_B);
    cuda_utils::freeHostMatrix(h_C);
    cuda_utils::freeHostMatrix(h_D_regular);
    cuda_utils::freeHostMatrix(h_D_splitk);
    cuda_utils::freeHostMatrix(h_D_ref);
    cuda_utils::freeDeviceMatrix(d_A);
    cuda_utils::freeDeviceMatrix(d_B);
    cuda_utils::freeDeviceMatrix(d_C);
    cuda_utils::freeDeviceMatrix(d_D_regular);
    cuda_utils::freeDeviceMatrix(d_D_splitk);

    return (regular_passed && splitk_passed) ? 0 : -1;
}