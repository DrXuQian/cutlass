#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include "../common/matrix_utils.h"
#include "../common/cpu_gemm.h"
#include "../common/cuda_timer.h"
#include "../common/cuda_utils.h"

// Simple kernel demonstrating Split-K concept
__global__ void manual_splitk_gemm(
    int M, int N, int K, int K_split,
    float alpha, const float* A, const float* B,
    float beta, float* C, float* workspace) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int split_id = blockIdx.z;  // Which K slice this block handles

    if (row < M && col < N && split_id < K_split) {
        // Calculate K range for this split
        int k_per_split = K / K_split;
        int k_start = split_id * k_per_split;
        int k_end = (split_id == K_split - 1) ? K : k_start + k_per_split;

        // Compute partial sum for this K slice
        float sum = 0.0f;
        for (int k = k_start; k < k_end; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }

        // Store partial result in workspace
        workspace[split_id * M * N + row * N + col] = sum;
    }
}

// Reduction kernel to sum partial results
__global__ void reduce_splitk_results(
    int M, int N, int K_split,
    float alpha, float beta,
    const float* workspace, const float* C_in, float* C_out) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;

        // Sum all K-split partial results
        for (int split = 0; split < K_split; ++split) {
            sum += workspace[split * M * N + row * N + col];
        }

        // Apply alpha/beta and write final result
        C_out[row * N + col] = alpha * sum + beta * C_in[row * N + col];
    }
}

int main() {
    // Matrix dimensions
    const int M = 512;
    const int N = 512;
    const int K = 4096;  // Large K for Split-K benefit
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Split-K configurations to test
    std::vector<int> split_configs = {1, 2, 4, 8};

    std::cout << "Split-K GEMM Demonstration" << std::endl;
    std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K << std::endl;
    std::cout << "\n=== Split-K Concept ===" << std::endl;
    std::cout << "Instead of each thread block computing the full K reduction," << std::endl;
    std::cout << "Split-K divides K into chunks processed by different blocks," << std::endl;
    std::cout << "then reduces the partial results.\n" << std::endl;

    // Allocate host memory
    float *h_A = cuda_utils::allocateHostMatrix<float>(M, K);
    float *h_B = cuda_utils::allocateHostMatrix<float>(K, N);
    float *h_C = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D_ref = cuda_utils::allocateHostMatrix<float>(M, N);

    // Initialize matrices
    cuda_utils::initializeRandomMatrix(h_A, M, K, -1.0f, 1.0f, 42);
    cuda_utils::initializeRandomMatrix(h_B, K, N, -1.0f, 1.0f, 43);
    cuda_utils::initializeConstantMatrix(h_C, M, N, 0.0f);
    cuda_utils::initializeConstantMatrix(h_D_ref, M, N, 0.0f);

    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_D;
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_A, M, K));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_B, K, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_C, M, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_D, M, N));

    // Copy to device
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_A, h_A, M, K));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_B, h_B, K, N));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_C, h_C, M, N));

    // CPU reference
    cpu_gemm::gemm_cpu(M, N, K, alpha, h_A, K, h_B, N, beta, h_D_ref, N);

    // Test different Split-K configurations
    cuda_utils::CudaTimer timer;
    dim3 block(16, 16);
    dim3 grid((N + block.x - 1) / block.x, (M + block.y - 1) / block.y);

    std::cout << "K_split | Time (ms) | Speedup | Verified" << std::endl;
    std::cout << "--------|-----------|---------|----------" << std::endl;

    float baseline_time = 0;

    for (int K_split : split_configs) {
        // Allocate workspace for partial results
        float* d_workspace;
        CHECK_CUDA(hipMalloc(&d_workspace, sizeof(float) * K_split * M * N));

        // Set grid for Split-K (using Z dimension for K splits)
        dim3 splitk_grid(grid.x, grid.y, K_split);

        // Warmup
        manual_splitk_gemm<<<splitk_grid, block>>>(
            M, N, K, K_split, alpha, d_A, d_B, beta, d_C, d_workspace);
        reduce_splitk_results<<<grid, block>>>(
            M, N, K_split, alpha, beta, d_workspace, d_C, d_D);
        CHECK_CUDA(hipDeviceSynchronize());

        // Benchmark
        const int iterations = 100;
        timer.reset();
        timer.start();
        for (int i = 0; i < iterations; ++i) {
            manual_splitk_gemm<<<splitk_grid, block>>>(
                M, N, K, K_split, alpha, d_A, d_B, beta, d_C, d_workspace);
            reduce_splitk_results<<<grid, block>>>(
                M, N, K_split, alpha, beta, d_workspace, d_C, d_D);
        }
        CHECK_CUDA(hipDeviceSynchronize());
        timer.stop();

        float time = timer.getElapsedTime() / iterations;
        if (K_split == 1) baseline_time = time;
        float speedup = baseline_time / time;

        // Verify
        CHECK_CUDA(cuda_utils::copyDeviceToHost(h_D, d_D, M, N));
        bool passed = cuda_utils::compareMatrices(h_D, h_D_ref, M, N, 1e-3f, false);

        std::cout << std::setw(7) << K_split << " | "
                  << std::setw(9) << std::setprecision(4) << time << " | "
                  << std::setw(7) << std::setprecision(3) << speedup << "x | "
                  << (passed ? "✓ PASS" : "✗ FAIL") << std::endl;

        hipFree(d_workspace);
    }

    // CUTLASS comparison
    std::cout << "\n=== CUTLASS GEMM (for comparison) ===" << std::endl;

    using RowMajor = cutlass::layout::RowMajor;
    using CutlassGemm = cutlass::gemm::device::Gemm<
        float, RowMajor,
        float, RowMajor,
        float, RowMajor
    >;

    CutlassGemm gemm_op;
    CutlassGemm::Arguments args(
        {M, N, K},
        {d_A, K}, {d_B, N}, {d_C, N}, {d_D, N},
        {alpha, beta}
    );

    // Warmup
    gemm_op(args);
    CHECK_CUDA(hipDeviceSynchronize());

    // Benchmark
    timer.reset();
    timer.start();
    for (int i = 0; i < 100; ++i) {
        gemm_op(args);
    }
    CHECK_CUDA(hipDeviceSynchronize());
    timer.stop();

    float cutlass_time = timer.getElapsedTime() / 100;
    double cutlass_gflops = cuda_utils::computeGFLOPS(M, N, K, cutlass_time);

    std::cout << "CUTLASS GEMM: " << cutlass_time << " ms ("
              << cutlass_gflops << " GFLOPS)" << std::endl;

    // Analysis
    std::cout << "\n=== Analysis ===" << std::endl;
    std::cout << "1. Split-K reduces memory pressure by processing smaller K chunks" << std::endl;
    std::cout << "2. Multiple blocks can work on the same output tile" << std::endl;
    std::cout << "3. Reduction overhead must be considered" << std::endl;
    std::cout << "4. Most beneficial when K >> M, N" << std::endl;

    std::cout << "\nKey insights:" << std::endl;
    std::cout << "- K=" << K << " is " << (K > M ? "larger" : "smaller") << " than M=" << M << std::endl;
    std::cout << "- Each K-split processes " << K/8 << " elements (for K_split=8)" << std::endl;
    std::cout << "- Workspace size: " << 8 * M * N * sizeof(float) / (1024.0 * 1024.0)
              << " MB for K_split=8" << std::endl;

    // Cleanup
    cuda_utils::freeHostMatrix(h_A);
    cuda_utils::freeHostMatrix(h_B);
    cuda_utils::freeHostMatrix(h_C);
    cuda_utils::freeHostMatrix(h_D);
    cuda_utils::freeHostMatrix(h_D_ref);
    cuda_utils::freeDeviceMatrix(d_A);
    cuda_utils::freeDeviceMatrix(d_B);
    cuda_utils::freeDeviceMatrix(d_C);
    cuda_utils::freeDeviceMatrix(d_D);

    return 0;
}