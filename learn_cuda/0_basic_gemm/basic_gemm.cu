#include <iostream>
#include <sstream>
#include <vector>
#include <hip/hip_runtime.h>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>

#include "../common/matrix_utils.h"
#include "../common/cpu_gemm.h"
#include "../common/cuda_timer.h"
#include "../common/cuda_utils.h"

int main() {
    // 定义矩阵尺寸
    int M = 512, N = 512, K = 512;
    float alpha = 1.0f, beta = 0.0f;

    std::cout << "CUTLASS Basic GEMM Example" << std::endl;
    std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K << std::endl;
    std::cout << "alpha=" << alpha << ", beta=" << beta << std::endl;

    // 打印设备信息
    cuda_utils::printDeviceInfo();

    // 分配主机内存
    float *h_A = cuda_utils::allocateHostMatrix<float>(M, K);
    float *h_B = cuda_utils::allocateHostMatrix<float>(K, N);
    float *h_C = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D = cuda_utils::allocateHostMatrix<float>(M, N);  // CUTLASS output
    float *h_D_ref = cuda_utils::allocateHostMatrix<float>(M, N);  // CPU reference

    // 初始化矩阵
    cuda_utils::initializeRandomMatrix(h_A, M, K, -1.0f, 1.0f, 42);
    cuda_utils::initializeRandomMatrix(h_B, K, N, -1.0f, 1.0f, 43);
    cuda_utils::initializeRandomMatrix(h_C, M, N, -1.0f, 1.0f, 44);

    // 复制C到D_ref用于CPU计算
    for (int i = 0; i < M * N; ++i) {
        h_D_ref[i] = h_C[i];
    }

    // 打印输入矩阵样本
    cuda_utils::printMatrix(h_A, M, K, "Matrix A (sample)", 5, 5);
    cuda_utils::printMatrix(h_B, K, N, "Matrix B (sample)", 5, 5);
    cuda_utils::printMatrix(h_C, M, N, "Matrix C (sample)", 5, 5);

    // 分配设备内存
    float *d_A, *d_B, *d_C, *d_D;
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_A, M, K));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_B, K, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_C, M, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_D, M, N));

    // 复制数据到设备
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_A, h_A, M, K));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_B, h_B, K, N));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_C, h_C, M, N));

    // 使用列主序布局定义GEMM类型
    using ColumnMajor = cutlass::layout::ColumnMajor;
    using RowMajor = cutlass::layout::RowMajor;

    // 注意：我们的数据是行主序，所以需要使用RowMajor布局
    using CutlassGemm = cutlass::gemm::device::Gemm<
        float,        // A矩阵数据类型
        RowMajor,     // A矩阵布局
        float,        // B矩阵数据类型
        RowMajor,     // B矩阵布局
        float,        // C矩阵数据类型
        RowMajor      // C矩阵布局
    >;

    // 创建GEMM操作符
    CutlassGemm gemm_operator;

    // 构造CUTLASS GEMM参数对象
    CutlassGemm::Arguments args(
        {M, N, K},          // GEMM问题维度
        {d_A, K},           // A矩阵张量引用 (pointer, leading dimension)
        {d_B, N},           // B矩阵张量引用
        {d_C, N},           // C矩阵张量引用
        {d_D, N},           // 目标矩阵D
        {alpha, beta}       // 标量参数
    );

    // 创建计时器
    cuda_utils::CudaTimer timer;

    // 预热
    cutlass::Status status = gemm_operator(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "CUTLASS GEMM failed during warmup!" << std::endl;
        return -1;
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // 基准测试
    const int num_iterations = 10;
    timer.start();
    for (int i = 0; i < num_iterations; ++i) {
        status = gemm_operator(args);
        if (status != cutlass::Status::kSuccess) {
            std::cerr << "CUTLASS GEMM failed!" << std::endl;
            return -1;
        }
    }
    CHECK_CUDA(hipDeviceSynchronize());
    timer.stop();

    float gpu_time = timer.getElapsedTime() / num_iterations;
    double gpu_gflops = cuda_utils::computeGFLOPS(M, N, K, gpu_time);

    std::cout << "\nCUTLASS Performance:" << std::endl;
    std::cout << "  Time: " << gpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << gpu_gflops << " GFLOPS" << std::endl;

    // 复制结果回主机
    CHECK_CUDA(cuda_utils::copyDeviceToHost(h_D, d_D, M, N));

    // CPU验证
    std::cout << "\nRunning CPU verification..." << std::endl;
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gemm::gemm_cpu(M, N, K, alpha, h_A, K, h_B, N, beta, h_D_ref, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_diff = cpu_end - cpu_start;
    double cpu_time = cpu_diff.count() * 1000.0;  // Convert to ms
    double cpu_gflops = cuda_utils::computeGFLOPS(M, N, K, cpu_time);

    std::cout << "CPU Performance:" << std::endl;
    std::cout << "  Time: " << cpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << cpu_gflops << " GFLOPS" << std::endl;

    // 验证结果
    bool passed = cuda_utils::compareMatrices(h_D, h_D_ref, M, N, 1e-3f, true);

    if (passed) {
        std::cout << "\n✓ Verification PASSED!" << std::endl;
    } else {
        std::cout << "\n✗ Verification FAILED!" << std::endl;
    }

    // 打印加速比
    double speedup = cpu_time / gpu_time;
    std::cout << "\nSpeedup: " << speedup << "x" << std::endl;

    // 打印输出矩阵样本
    cuda_utils::printMatrix(h_D, M, N, "Matrix D = alpha*A*B + beta*C (sample)", 5, 5);

    // 打印内存使用情况
    std::cout << "\nMemory usage:" << std::endl;
    cuda_utils::printMemoryUsage();

    // 清理内存
    cuda_utils::freeHostMatrix(h_A);
    cuda_utils::freeHostMatrix(h_B);
    cuda_utils::freeHostMatrix(h_C);
    cuda_utils::freeHostMatrix(h_D);
    cuda_utils::freeHostMatrix(h_D_ref);
    cuda_utils::freeDeviceMatrix(d_A);
    cuda_utils::freeDeviceMatrix(d_B);
    cuda_utils::freeDeviceMatrix(d_C);
    cuda_utils::freeDeviceMatrix(d_D);

    return passed ? 0 : -1;
}