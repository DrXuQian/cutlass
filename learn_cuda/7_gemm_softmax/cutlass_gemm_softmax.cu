#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Simple softmax kernel
template <typename Element>
__global__ void applySoftmax(
    Element* matrix,
    int M, int N) {

    int row = blockIdx.x;
    if (row >= M) return;

    // Find max in row
    float max_val = -INFINITY;
    for (int col = 0; col < N; ++col) {
        float val = float(matrix[row * N + col]);
        max_val = fmaxf(max_val, val);
    }

    // Compute exp and sum
    float sum = 0.0f;
    for (int col = 0; col < N; ++col) {
        float val = float(matrix[row * N + col]);
        float exp_val = expf(val - max_val);
        matrix[row * N + col] = Element(exp_val);
        sum += exp_val;
    }

    // Normalize
    for (int col = 0; col < N; ++col) {
        float val = float(matrix[row * N + col]);
        matrix[row * N + col] = Element(val / sum);
    }
}

int main() {
    const int M = 64, N = 64, K = 64;

    // Define CUTLASS GEMM with TensorCore
    using ElementA = cutlass::half_t;
    using ElementB = cutlass::half_t;
    using ElementC = cutlass::half_t;
    using ElementAccumulator = float;
    using ElementCompute = float;

    using Gemm = cutlass::gemm::device::Gemm<
        ElementA, cutlass::layout::RowMajor,
        ElementB, cutlass::layout::RowMajor,
        ElementC, cutlass::layout::RowMajor,
        ElementAccumulator,
        cutlass::arch::OpClassTensorOp,
        cutlass::arch::Sm75,
        cutlass::gemm::GemmShape<128, 128, 32>,
        cutlass::gemm::GemmShape<64, 64, 32>,
        cutlass::gemm::GemmShape<16, 8, 8>,
        cutlass::epilogue::thread::LinearCombination<
            ElementC,
            128 / cutlass::sizeof_bits<ElementC>::value,
            ElementAccumulator,
            ElementCompute
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        2
    >;

    // Allocate host matrices
    std::vector<cutlass::half_t> h_A(M * K);
    std::vector<cutlass::half_t> h_B(K * N);
    std::vector<cutlass::half_t> h_C(M * N, cutlass::half_t(0));
    std::vector<cutlass::half_t> h_D(M * N, cutlass::half_t(0));

    // Initialize matrices with small values
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = cutlass::half_t((rand() % 100) / 100.0f);
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = cutlass::half_t((rand() % 100) / 100.0f);
    }

    // Allocate device memory
    cutlass::half_t *d_A, *d_B, *d_C, *d_D;
    size_t size_A = M * K * sizeof(cutlass::half_t);
    size_t size_B = K * N * sizeof(cutlass::half_t);
    size_t size_C = M * N * sizeof(cutlass::half_t);

    CHECK_CUDA(hipMalloc(&d_A, size_A));
    CHECK_CUDA(hipMalloc(&d_B, size_B));
    CHECK_CUDA(hipMalloc(&d_C, size_C));
    CHECK_CUDA(hipMalloc(&d_D, size_C));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), size_A, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), size_B, hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_C, h_C.data(), size_C, hipMemcpyHostToDevice));

    // Phase 1: TensorCore GEMM
    Gemm gemm_op;
    Gemm::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D, N},
        {ElementCompute(1.0f), ElementCompute(0.0f)}
    );

    cutlass::Status status = gemm_op(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM execution failed" << std::endl;
        return -1;
    }

    // Phase 2: Apply softmax
    applySoftmax<<<M, 1>>>(d_D, M, N);

    CHECK_CUDA(hipDeviceSynchronize());

    // Copy result back
    CHECK_CUDA(hipMemcpy(h_D.data(), d_D, size_C, hipMemcpyDeviceToHost));

    // Verify softmax properties
    bool correct = true;
    for (int i = 0; i < M; ++i) {
        float row_sum = 0.0f;
        for (int j = 0; j < N; ++j) {
            float val = float(h_D[i * N + j]);
            row_sum += val;
            // Check values are in [0, 1]
            if (val < 0.0f || val > 1.0f) {
                correct = false;
                break;
            }
        }
        // Check row sums to 1
        if (std::abs(row_sum - 1.0f) > 0.01f) {
            correct = false;
            std::cout << "Row " << i << " sum: " << row_sum << std::endl;
            break;
        }
    }

    std::cout << "CUTLASS TensorCore GEMM + Softmax: " << (correct ? "PASS" : "FAIL") << std::endl;

    float first_row_sum = 0.0f;
    for (int j = 0; j < N; ++j) {
        first_row_sum += float(h_D[j]);
    }
    std::cout << "First row sum: " << first_row_sum << " (should be ~1.0)" << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_D));

    return 0;
}