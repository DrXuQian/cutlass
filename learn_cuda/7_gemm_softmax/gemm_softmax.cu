#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(error) << std::endl; \
        exit(1); \
    } \
} while(0)

// Warp-level reduction for max
__device__ float warpReduceMax(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val = fmaxf(val, __shfl_down_sync(0xffffffff, val, offset));
    }
    return val;
}

// Warp-level reduction for sum
__device__ float warpReduceSum(float val) {
    for (int offset = 16; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset);
    }
    return val;
}

// Fused GEMM + Softmax kernel
template<int TILE_SIZE>
__global__ void gemmSoftmax(
    const float* A, const float* B, float* C,
    int M, int N, int K) {

    __shared__ float sA[TILE_SIZE][TILE_SIZE];
    __shared__ float sB[TILE_SIZE][TILE_SIZE];
    __shared__ float row_max[TILE_SIZE];
    __shared__ float row_sum[TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    // Phase 1: Compute GEMM
    for (int k = 0; k < K; k += TILE_SIZE) {
        if (row < M && k + threadIdx.x < K) {
            sA[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if (col < N && k + threadIdx.y < K) {
            sB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += sA[threadIdx.y][i] * sB[i][threadIdx.x];
        }

        __syncthreads();
    }

    // Store intermediate GEMM result
    if (row < M && col < N) {
        C[row * N + col] = sum;
    }

    // Phase 2: Compute row-wise softmax
    // Step 1: Find row maximum
    float max_val = (row < M && col < N) ? sum : -INFINITY;
    if (threadIdx.x < 32) {
        max_val = warpReduceMax(max_val);
    }

    if (threadIdx.x == 0) {
        row_max[threadIdx.y] = max_val;
    }
    __syncthreads();

    // Broadcast max to all threads in row
    float row_max_val = row_max[threadIdx.y];

    // Step 2: Compute exp(x - max) and sum
    float exp_val = (row < M && col < N) ? expf(sum - row_max_val) : 0.0f;

    float sum_exp = exp_val;
    if (threadIdx.x < 32) {
        sum_exp = warpReduceSum(sum_exp);
    }

    if (threadIdx.x == 0) {
        row_sum[threadIdx.y] = sum_exp;
    }
    __syncthreads();

    // Step 3: Normalize
    if (row < M && col < N) {
        C[row * N + col] = exp_val / row_sum[threadIdx.y];
    }
}

// Simple row-wise softmax for verification
void cpuRowSoftmax(float* data, int M, int N) {
    for (int i = 0; i < M; ++i) {
        // Find max
        float max_val = -INFINITY;
        for (int j = 0; j < N; ++j) {
            max_val = std::max(max_val, data[i * N + j]);
        }

        // Compute exp and sum
        float sum = 0.0f;
        for (int j = 0; j < N; ++j) {
            data[i * N + j] = std::exp(data[i * N + j] - max_val);
            sum += data[i * N + j];
        }

        // Normalize
        for (int j = 0; j < N; ++j) {
            data[i * N + j] /= sum;
        }
    }
}

int main() {
    const int M = 64, N = 64, K = 64;
    const int TILE_SIZE = 32;

    // Allocate matrices
    std::vector<float> h_A(M * K);
    std::vector<float> h_B(K * N);
    std::vector<float> h_C(M * N, 0.0f);
    std::vector<float> h_C_ref(M * N, 0.0f);

    // Initialize with small values to avoid overflow
    for (int i = 0; i < M * K; ++i) {
        h_A[i] = (rand() % 100) / 100.0f;
    }
    for (int i = 0; i < K * N; ++i) {
        h_B[i] = (rand() % 100) / 100.0f;
    }

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, M * K * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, K * N * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, M * N * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice));

    dim3 blockDim(TILE_SIZE, TILE_SIZE);
    dim3 gridDim(
        (N + TILE_SIZE - 1) / TILE_SIZE,
        (M + TILE_SIZE - 1) / TILE_SIZE
    );

    // Run fused GEMM + Softmax
    std::cout << "Running Fused GEMM + Softmax..." << std::endl;
    gemmSoftmax<TILE_SIZE><<<gridDim, blockDim>>>(
        d_A, d_B, d_C, M, N, K
    );
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(h_C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost));

    // Compute reference on CPU
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float sum = 0.0f;
            for (int k = 0; k < K; ++k) {
                sum += h_A[i * K + k] * h_B[k * N + j];
            }
            h_C_ref[i * N + j] = sum;
        }
    }
    cpuRowSoftmax(h_C_ref.data(), M, N);

    // Verify softmax properties
    bool correct = true;
    for (int i = 0; i < M; ++i) {
        float row_sum = 0.0f;
        for (int j = 0; j < N; ++j) {
            row_sum += h_C[i * N + j];
            // Check that values are in [0, 1]
            if (h_C[i * N + j] < 0.0f || h_C[i * N + j] > 1.0f) {
                correct = false;
                break;
            }
        }
        // Check that row sums to 1
        if (std::abs(row_sum - 1.0f) > 1e-3) {
            correct = false;
            std::cout << "Row " << i << " sum: " << row_sum << std::endl;
            break;
        }
    }

    // Compare with reference
    float max_error = 0.0f;
    for (int i = 0; i < M * N; ++i) {
        max_error = std::max(max_error, std::abs(h_C[i] - h_C_ref[i]));
    }

    std::cout << "GEMM + Softmax: " << (correct ? "PASS" : "FAIL") << std::endl;
    std::cout << "Max error vs reference: " << max_error << std::endl;
    std::cout << "First row sum: ";
    float first_row_sum = 0.0f;
    for (int j = 0; j < N; ++j) {
        first_row_sum += h_C[j];
    }
    std::cout << first_row_sum << " (should be ~1.0)" << std::endl;

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));

    return 0;
}