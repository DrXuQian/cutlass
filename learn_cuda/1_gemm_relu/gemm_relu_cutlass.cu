#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/epilogue/thread/linear_combination_relu.h>

#include "../common/matrix_utils.h"
#include "../common/cpu_gemm.h"
#include "../common/cuda_timer.h"
#include "../common/cuda_utils.h"

// CPU reference for GEMM + ReLU
void cpu_gemm_relu_ref(int M, int N, int K,
                       float alpha, const float* A, int lda,
                       const float* B, int ldb,
                       float beta, float* C, int ldc) {
    cpu_gemm::gemm_cpu(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);
    for (int i = 0; i < M * N; ++i) {
        C[i] = fmaxf(0.0f, C[i]);
    }
}

int main() {
    // Matrix dimensions
    const int M = 1024;
    const int N = 1024;
    const int K = 1024;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    std::cout << "CUTLASS GEMM + ReLU Fusion Example" << std::endl;
    std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K << std::endl;
    std::cout << "Operation: C = ReLU(alpha * A * B + beta * C)" << std::endl;
    std::cout << "alpha=" << alpha << ", beta=" << beta << "\n" << std::endl;

    // Allocate host memory
    float *h_A = cuda_utils::allocateHostMatrix<float>(M, K);
    float *h_B = cuda_utils::allocateHostMatrix<float>(K, N);
    float *h_C = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D_ref = cuda_utils::allocateHostMatrix<float>(M, N);

    // Initialize matrices
    cuda_utils::initializeRandomMatrix(h_A, M, K, -2.0f, 2.0f, 42);
    cuda_utils::initializeRandomMatrix(h_B, K, N, -2.0f, 2.0f, 43);
    cuda_utils::initializeConstantMatrix(h_C, M, N, 0.0f);

    // Copy C for reference
    for (int i = 0; i < M * N; ++i) {
        h_D_ref[i] = h_C[i];
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_D;
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_A, M, K));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_B, K, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_C, M, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_D, M, N));

    // Copy to device
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_A, h_A, M, K));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_B, h_B, K, N));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_C, h_C, M, N));

    // Define CUTLASS GEMM with ReLU epilogue
    using RowMajor = cutlass::layout::RowMajor;

    // GEMM with ReLU fusion in epilogue
    using CutlassGemmRelu = cutlass::gemm::device::Gemm<
        float,                                          // ElementA
        RowMajor,                                       // LayoutA
        float,                                          // ElementB
        RowMajor,                                       // LayoutB
        float,                                          // ElementC
        RowMajor,                                       // LayoutC
        float,                                          // ElementAccumulator
        cutlass::arch::OpClassSimt,                    // OpClass
        cutlass::arch::Sm80,                           // ArchTag
        cutlass::gemm::GemmShape<128, 128, 8>,        // ThreadblockShape
        cutlass::gemm::GemmShape<32, 64, 8>,          // WarpShape
        cutlass::gemm::GemmShape<1, 1, 1>,            // InstructionShape
        cutlass::epilogue::thread::LinearCombinationRelu< // Epilogue with ReLU
            float,                                      // ElementOutput
            1,                                          // ElementsPerAccess
            float,                                      // ElementAccumulator
            float                                       // ElementCompute
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, // Swizzle
        3                                              // Stages
    >;

    CutlassGemmRelu gemm_relu_op;

    // Setup arguments
    CutlassGemmRelu::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D, N},
        {alpha, beta}
    );

    // Execute CUTLASS GEMM + ReLU
    std::cout << "Running CUTLASS GEMM with Fused ReLU..." << std::endl;

    cuda_utils::CudaTimer timer;
    const int num_iterations = 10;

    // Warmup
    cutlass::Status status = gemm_relu_op(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "CUTLASS GEMM+ReLU failed!" << std::endl;
        return -1;
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Benchmark
    timer.start();
    for (int i = 0; i < num_iterations; ++i) {
        status = gemm_relu_op(args);
        if (status != cutlass::Status::kSuccess) {
            std::cerr << "CUTLASS GEMM+ReLU failed!" << std::endl;
            return -1;
        }
    }
    CHECK_CUDA(hipDeviceSynchronize());
    timer.stop();

    float gpu_time = timer.getElapsedTime() / num_iterations;
    double gpu_gflops = cuda_utils::computeGFLOPS(M, N, K, gpu_time);

    std::cout << "CUTLASS Performance:" << std::endl;
    std::cout << "  Time: " << gpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << gpu_gflops << " GFLOPS" << std::endl;

    // Copy result back
    CHECK_CUDA(cuda_utils::copyDeviceToHost(h_D, d_D, M, N));

    // CPU reference
    std::cout << "\nRunning CPU reference..." << std::endl;
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gemm_relu_ref(M, N, K, alpha, h_A, K, h_B, N, beta, h_D_ref, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_diff = cpu_end - cpu_start;
    double cpu_time = cpu_diff.count() * 1000.0;
    double cpu_gflops = cuda_utils::computeGFLOPS(M, N, K, cpu_time);

    std::cout << "CPU Performance:" << std::endl;
    std::cout << "  Time: " << cpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << cpu_gflops << " GFLOPS" << std::endl;

    // Verification
    std::cout << "\nVerifying results..." << std::endl;
    bool passed = cuda_utils::compareMatrices(h_D, h_D_ref, M, N, 1e-3f, true);

    if (passed) {
        std::cout << "✓ Verification PASSED!" << std::endl;
    } else {
        std::cout << "✗ Verification FAILED!" << std::endl;
    }

    // Count ReLU activations
    int relu_count = 0;
    for (int i = 0; i < M * N; ++i) {
        if (h_D_ref[i] > 0) relu_count++;
    }

    std::cout << "\nReLU activation rate: " << (100.0f * relu_count / (M * N))
              << "% (" << relu_count << "/" << (M * N) << " values > 0)" << std::endl;

    std::cout << "\nSpeedup: " << cpu_time / gpu_time << "x" << std::endl;

    // Print output sample
    cuda_utils::printMatrix(h_D, M, N, "\nOutput Matrix (after ReLU, sample)", 5, 5);

    // Cleanup
    cuda_utils::freeHostMatrix(h_A);
    cuda_utils::freeHostMatrix(h_B);
    cuda_utils::freeHostMatrix(h_C);
    cuda_utils::freeHostMatrix(h_D);
    cuda_utils::freeHostMatrix(h_D_ref);
    cuda_utils::freeDeviceMatrix(d_A);
    cuda_utils::freeDeviceMatrix(d_B);
    cuda_utils::freeDeviceMatrix(d_C);
    cuda_utils::freeDeviceMatrix(d_D);

    return passed ? 0 : -1;
}