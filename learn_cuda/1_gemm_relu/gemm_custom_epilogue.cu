#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/epilogue/thread/linear_combination.h>
#include <cutlass/epilogue/thread/activation.h>

#include "../common/matrix_utils.h"
#include "../common/cpu_gemm.h"
#include "../common/cuda_timer.h"
#include "../common/cuda_utils.h"

// 自定义 ReLU 激活函数
template <typename T>
struct CustomReLU {
    CUTLASS_HOST_DEVICE
    T operator()(T const &value) const {
        return value > T(0) ? value : T(0);
    }
};

// 自定义 Epilogue Functor
template <
    typename ElementOutput_,
    int Count,
    typename ElementAccumulator_ = ElementOutput_,
    typename ElementCompute_ = ElementOutput_
>
class CustomLinearCombinationRelu {
public:
    using ElementOutput = ElementOutput_;
    using ElementAccumulator = ElementAccumulator_;
    using ElementCompute = ElementCompute_;
    using ElementC = ElementOutput_;

    static int const kCount = Count;

    using FragmentOutput = cutlass::Array<ElementOutput, kCount>;
    using FragmentAccumulator = cutlass::Array<ElementAccumulator, kCount>;
    using FragmentCompute = cutlass::Array<ElementCompute, kCount>;
    using FragmentC = cutlass::Array<ElementC, kCount>;

    struct Params {
        ElementCompute alpha;
        ElementCompute beta;

        CUTLASS_HOST_DEVICE
        Params() : alpha(1), beta(0) {}

        CUTLASS_HOST_DEVICE
        Params(ElementCompute alpha, ElementCompute beta)
            : alpha(alpha), beta(beta) {}
    };

private:
    ElementCompute alpha_;
    ElementCompute beta_;

public:
    CUTLASS_HOST_DEVICE
    CustomLinearCombinationRelu(Params const &params)
        : alpha_(params.alpha), beta_(params.beta) {}

    CUTLASS_HOST_DEVICE
    bool is_source_needed() const {
        return beta_ != ElementCompute(0);
    }

    CUTLASS_HOST_DEVICE
    void set_k_partition(int k_partition, int k_partition_count) {}

    CUTLASS_HOST_DEVICE
    FragmentOutput operator()(
        FragmentAccumulator const &accumulator,
        FragmentC const &source) const {

        FragmentOutput output;
        CustomReLU<ElementCompute> relu_op;

        CUTLASS_PRAGMA_UNROLL
        for (int i = 0; i < kCount; ++i) {
            // 计算 alpha * accumulator + beta * source
            ElementCompute compute_result =
                alpha_ * ElementCompute(accumulator[i]) +
                beta_ * ElementCompute(source[i]);

            // 应用 ReLU: max(0, x)
            output[i] = ElementOutput(relu_op(compute_result));
        }

        return output;
    }

    CUTLASS_HOST_DEVICE
    FragmentOutput operator()(FragmentAccumulator const &accumulator) const {
        FragmentOutput output;
        CustomReLU<ElementCompute> relu_op;

        CUTLASS_PRAGMA_UNROLL
        for (int i = 0; i < kCount; ++i) {
            // 计算 alpha * accumulator
            ElementCompute compute_result = alpha_ * ElementCompute(accumulator[i]);

            // 应用 ReLU: max(0, x)
            output[i] = ElementOutput(relu_op(compute_result));
        }

        return output;
    }
};

// CPU reference
void cpu_gemm_relu_ref(int M, int N, int K,
                       float alpha, const float* A, int lda,
                       const float* B, int ldb,
                       float beta, float* C, int ldc) {
    cpu_gemm::gemm_cpu(M, N, K, alpha, A, lda, B, ldb, beta, C, ldc);
    for (int i = 0; i < M * N; ++i) {
        C[i] = fmaxf(0.0f, C[i]);
    }
}

int main() {
    const int M = 1024;
    const int N = 1024;
    const int K = 1024;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    std::cout << "CUTLASS GEMM with Custom ReLU Epilogue" << std::endl;
    std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K << std::endl;
    std::cout << "Operation: C = ReLU(alpha * A * B + beta * C)" << std::endl;
    std::cout << "alpha=" << alpha << ", beta=" << beta << "\n" << std::endl;

    // Allocate host memory
    float *h_A = cuda_utils::allocateHostMatrix<float>(M, K);
    float *h_B = cuda_utils::allocateHostMatrix<float>(K, N);
    float *h_C = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_D_ref = cuda_utils::allocateHostMatrix<float>(M, N);

    // Initialize matrices
    cuda_utils::initializeRandomMatrix(h_A, M, K, -2.0f, 2.0f, 42);
    cuda_utils::initializeRandomMatrix(h_B, K, N, -2.0f, 2.0f, 43);
    cuda_utils::initializeConstantMatrix(h_C, M, N, 0.0f);

    for (int i = 0; i < M * N; ++i) {
        h_D_ref[i] = h_C[i];
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C, *d_D;
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_A, M, K));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_B, K, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_C, M, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_D, M, N));

    // Copy to device
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_A, h_A, M, K));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_B, h_B, K, N));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_C, h_C, M, N));

    // Define CUTLASS GEMM with custom epilogue
    using RowMajor = cutlass::layout::RowMajor;

    // 使用自定义的 epilogue functor
    using CutlassGemmCustom = cutlass::gemm::device::Gemm<
        float,                                          // ElementA
        RowMajor,                                       // LayoutA
        float,                                          // ElementB
        RowMajor,                                       // LayoutB
        float,                                          // ElementC
        RowMajor,                                       // LayoutC
        float,                                          // ElementAccumulator
        cutlass::arch::OpClassSimt,                    // OpClass
        cutlass::arch::Sm80,                           // ArchTag
        cutlass::gemm::GemmShape<128, 128, 8>,        // ThreadblockShape
        cutlass::gemm::GemmShape<32, 64, 8>,          // WarpShape
        cutlass::gemm::GemmShape<1, 1, 1>,            // InstructionShape
        CustomLinearCombinationRelu<                   // 自定义 Epilogue
            float,                                      // ElementOutput
            1,                                          // ElementsPerAccess
            float,                                      // ElementAccumulator
            float                                       // ElementCompute
        >,
        cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>,
        3                                              // Stages
    >;

    CutlassGemmCustom gemm_op;

    // Setup arguments
    CutlassGemmCustom::Arguments args(
        {M, N, K},
        {d_A, K},
        {d_B, N},
        {d_C, N},
        {d_D, N},
        {alpha, beta}
    );

    std::cout << "Running CUTLASS GEMM with Custom ReLU Epilogue..." << std::endl;

    cuda_utils::CudaTimer timer;
    const int num_iterations = 10;

    // Warmup
    cutlass::Status status = gemm_op(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "CUTLASS GEMM failed! Error: " << cutlass::cutlassGetStatusString(status) << std::endl;
        return -1;
    }
    CHECK_CUDA(hipDeviceSynchronize());

    // Benchmark
    timer.start();
    for (int i = 0; i < num_iterations; ++i) {
        status = gemm_op(args);
        if (status != cutlass::Status::kSuccess) {
            std::cerr << "CUTLASS GEMM failed!" << std::endl;
            return -1;
        }
    }
    CHECK_CUDA(hipDeviceSynchronize());
    timer.stop();

    float gpu_time = timer.getElapsedTime() / num_iterations;
    double gpu_gflops = cuda_utils::computeGFLOPS(M, N, K, gpu_time);

    std::cout << "CUTLASS with Custom Epilogue Performance:" << std::endl;
    std::cout << "  Time: " << gpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << gpu_gflops << " GFLOPS" << std::endl;

    // Copy result back
    CHECK_CUDA(cuda_utils::copyDeviceToHost(h_D, d_D, M, N));

    // CPU reference
    std::cout << "\nRunning CPU reference..." << std::endl;
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gemm_relu_ref(M, N, K, alpha, h_A, K, h_B, N, beta, h_D_ref, N);
    auto cpu_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cpu_diff = cpu_end - cpu_start;
    double cpu_time = cpu_diff.count() * 1000.0;

    std::cout << "CPU Performance:" << std::endl;
    std::cout << "  Time: " << cpu_time << " ms" << std::endl;

    // Verification
    std::cout << "\nVerifying results..." << std::endl;
    bool passed = cuda_utils::compareMatrices(h_D, h_D_ref, M, N, 1e-3f, true);

    if (passed) {
        std::cout << "✓ Verification PASSED!" << std::endl;
    } else {
        std::cout << "✗ Verification FAILED!" << std::endl;
    }

    // Show custom epilogue details
    std::cout << "\n=== Custom Epilogue Implementation ===" << std::endl;
    std::cout << "自定义 Epilogue 实现了以下操作：" << std::endl;
    std::cout << "1. 计算线性组合: result = alpha * accumulator + beta * source" << std::endl;
    std::cout << "2. 应用 ReLU 激活: output = max(0, result)" << std::endl;
    std::cout << "3. 所有操作在一个融合的 epilogue 中完成，避免额外内存访问" << std::endl;

    // Count ReLU activations
    int relu_count = 0;
    int zero_count = 0;
    for (int i = 0; i < M * N; ++i) {
        if (h_D_ref[i] > 0) {
            relu_count++;
        } else {
            zero_count++;
        }
    }

    std::cout << "\nReLU Statistics:" << std::endl;
    std::cout << "  Active values (> 0): " << relu_count << " ("
              << (100.0f * relu_count / (M * N)) << "%)" << std::endl;
    std::cout << "  Zeroed values (≤ 0): " << zero_count << " ("
              << (100.0f * zero_count / (M * N)) << "%)" << std::endl;

    std::cout << "\nSpeedup: " << cpu_time / gpu_time << "x" << std::endl;

    // Print sample output
    std::cout << "\nOutput sample (first 5x5):" << std::endl;
    for (int i = 0; i < std::min(5, M); ++i) {
        for (int j = 0; j < std::min(5, N); ++j) {
            std::cout << std::setw(8) << std::setprecision(2) << h_D[i * N + j] << " ";
        }
        std::cout << "..." << std::endl;
    }

    // Cleanup
    cuda_utils::freeHostMatrix(h_A);
    cuda_utils::freeHostMatrix(h_B);
    cuda_utils::freeHostMatrix(h_C);
    cuda_utils::freeHostMatrix(h_D);
    cuda_utils::freeHostMatrix(h_D_ref);
    cuda_utils::freeDeviceMatrix(d_A);
    cuda_utils::freeDeviceMatrix(d_B);
    cuda_utils::freeDeviceMatrix(d_C);
    cuda_utils::freeDeviceMatrix(d_D);

    return passed ? 0 : -1;
}