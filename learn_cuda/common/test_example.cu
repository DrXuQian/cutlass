#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "matrix_utils.h"
#include "cpu_gemm.h"
#include "cuda_timer.h"
#include "cuda_utils.h"

// Simple CUDA kernel for GEMM
__global__ void simple_gemm_kernel(int M, int N, int K,
                                   float alpha, const float* A,
                                   const float* B, float beta, float* C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = alpha * sum + beta * C[row * N + col];
    }
}

int main() {
    // Matrix dimensions
    const int M = 512;
    const int N = 512;
    const int K = 512;
    const float alpha = 1.0f;
    const float beta = 0.0f;

    std::cout << "Testing CUDA common utilities" << std::endl;
    std::cout << "Matrix dimensions: M=" << M << ", N=" << N << ", K=" << K << std::endl;

    // Print device info
    cuda_utils::printDeviceInfo();

    // Allocate host matrices
    float *h_A = cuda_utils::allocateHostMatrix<float>(M, K);
    float *h_B = cuda_utils::allocateHostMatrix<float>(K, N);
    float *h_C = cuda_utils::allocateHostMatrix<float>(M, N);
    float *h_C_ref = cuda_utils::allocateHostMatrix<float>(M, N);

    // Initialize matrices with random values
    cuda_utils::initializeRandomMatrix(h_A, M, K, -1.0f, 1.0f);
    cuda_utils::initializeRandomMatrix(h_B, K, N, -1.0f, 1.0f);
    cuda_utils::initializeConstantMatrix(h_C, M, N, 0.0f);
    cuda_utils::initializeConstantMatrix(h_C_ref, M, N, 0.0f);

    // Print sample of input matrices
    cuda_utils::printMatrix(h_A, M, K, "Matrix A (sample)", 5, 5);
    cuda_utils::printMatrix(h_B, K, N, "Matrix B (sample)", 5, 5);

    // Allocate device matrices
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_A, M, K));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_B, K, N));
    CHECK_CUDA(cuda_utils::allocateDeviceMatrix(&d_C, M, N));

    // Copy matrices to device
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_A, h_A, M, K));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_B, h_B, K, N));
    CHECK_CUDA(cuda_utils::copyHostToDevice(d_C, h_C, M, N));

    // Configure kernel launch parameters
    dim3 block_size = cuda_utils::calculate2DBlockSize(N, M);
    dim3 grid_size = cuda_utils::calculate2DGridSize(N, M, block_size);

    std::cout << "\nKernel configuration:" << std::endl;
    std::cout << "  Block size: (" << block_size.x << ", " << block_size.y << ")" << std::endl;
    std::cout << "  Grid size: (" << grid_size.x << ", " << grid_size.y << ")" << std::endl;

    // Create timer
    cuda_utils::CudaTimer timer;

    // Warm up
    simple_gemm_kernel<<<grid_size, block_size>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    cuda_utils::syncAndCheck();

    // Benchmark GPU kernel
    const int num_iterations = 10;
    timer.start();
    for (int i = 0; i < num_iterations; ++i) {
        simple_gemm_kernel<<<grid_size, block_size>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    }
    cuda_utils::syncAndCheck();
    timer.stop();

    float gpu_time = timer.getElapsedTime() / num_iterations;
    double gpu_gflops = cuda_utils::computeGFLOPS(M, N, K, gpu_time);

    std::cout << "\nGPU Performance:" << std::endl;
    std::cout << "  Time: " << gpu_time << " ms" << std::endl;
    std::cout << "  Performance: " << gpu_gflops << " GFLOPS" << std::endl;

    // Copy result back to host
    CHECK_CUDA(cuda_utils::copyDeviceToHost(h_C, d_C, M, N));

    // CPU verification
    std::cout << "\nRunning CPU verification..." << std::endl;
    double cpu_time = cpu_gemm::benchmark_cpu_gemm(M, N, K, alpha, h_A, M,
                                                   h_B, N, beta, h_C_ref, N, 1);

    // Verify results
    bool passed = cpu_gemm::verify_gemm(M, N, K, alpha, h_A, M,
                                        h_B, N, beta, h_C, N, 1e-3f, true);

    if (passed) {
        std::cout << "\nVerification PASSED!" << std::endl;
    } else {
        std::cout << "\nVerification FAILED!" << std::endl;
    }

    // Print speedup
    double speedup = cpu_time / gpu_time;
    std::cout << "\nSpeedup: " << speedup << "x" << std::endl;

    // Print memory usage
    std::cout << "\nMemory usage:" << std::endl;
    cuda_utils::printMemoryUsage();

    // Clean up
    cuda_utils::freeHostMatrix(h_A);
    cuda_utils::freeHostMatrix(h_B);
    cuda_utils::freeHostMatrix(h_C);
    cuda_utils::freeHostMatrix(h_C_ref);
    cuda_utils::freeDeviceMatrix(d_A);
    cuda_utils::freeDeviceMatrix(d_B);
    cuda_utils::freeDeviceMatrix(d_C);

    return passed ? 0 : 1;
}